
#include <hip/hip_runtime.h>

extern "C" __global__ void reduce0(double *g_idata, double *g_odata, unsigned int n)
{
	extern __shared__ double sdata[];

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    sdata[tid] = (i < n) ? g_idata[i] : 0;
    
    __syncthreads();

    // do reduction in shared mem
    for(unsigned int s=1; s < blockDim.x; s *= 2) {
        // modulo arithmetic is slow!
        if ((tid % (2*s)) == 0) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

extern "C" __global__ void reduce1(double *g_idata, double *g_odata, unsigned int n)
{
	extern __shared__ double sdata[];

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    sdata[tid] = (i < n) ? g_idata[i] : 0;
    
    __syncthreads();

    // do reduction in shared mem
    for(unsigned int s=1; s < blockDim.x; s *= 2) 
    {
        int index = 2 * s * tid;

        if (index < blockDim.x) 
        {
            sdata[index] += sdata[index + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

extern "C" __global__ void reduce2(double *g_idata, double *g_odata, unsigned int n)
{
	extern __shared__ double sdata[];

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    sdata[tid] = (i < n) ? g_idata[i] : 0;
    
    __syncthreads();

    // do reduction in shared mem
    for(unsigned int s=blockDim.x/2; s>0; s>>=1) 
    {
        if (tid < s) 
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

extern "C" __global__ void reduce3(double *g_idata, double *g_odata, unsigned int n)
{
	extern __shared__ double sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

    double mySum = (i < n) ? g_idata[i] : 0;
    if (i + blockDim.x < n) 
        mySum += g_idata[i+blockDim.x];  

    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    for(unsigned int s=blockDim.x/2; s>0; s>>=1) 
    {
        if (tid < s) 
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem 
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

extern "C" __global__ void reduce4(double *g_idata, double *g_odata, unsigned int n, unsigned int blockSize)
{
	extern __shared__ double sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

    double mySum = (i < n) ? g_idata[i] : 0;
    if (i + blockSize < n) 
        mySum += g_idata[i+blockSize];  

    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    for(unsigned int s=blockDim.x/2; s>32; s>>=1) 
    {
        if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile double *smem = sdata;
        if (blockSize >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; }
        if (blockSize >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; }
        if (blockSize >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; }
        if (blockSize >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; }
        if (blockSize >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; }
        if (blockSize >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; }
    }

    // write result for this block to global mem 
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

extern "C" __global__ void reduce5(double *g_idata, double *g_odata, unsigned int n, unsigned int blockSize)
{
	extern __shared__ double sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;

    double mySum = (i < n) ? g_idata[i] : 0;
    if (i + blockSize < n) 
        mySum += g_idata[i+blockSize];  

    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }
    
    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile double* smem = sdata;
        if (blockSize >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; }
        if (blockSize >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; }
        if (blockSize >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; }
        if (blockSize >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; }
        if (blockSize >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; }
        if (blockSize >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; }
    }
    
    // write result for this block to global mem 
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

extern "C" __global__ void reduce6(double *g_idata, double *g_odata, unsigned int n, unsigned int blockSize, int nIsPow2)
{
	extern __shared__ double sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    
    double mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {         
        mySum += g_idata[i];
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) 
            mySum += g_idata[i+blockSize];  
        i += gridSize;
    } 

    // each thread puts its local sum into shared memory 
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }
    
    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile double* smem = sdata;
        if (blockSize >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; }
        if (blockSize >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; }
        if (blockSize >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; }
        if (blockSize >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; }
        if (blockSize >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; }
        if (blockSize >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; }
    }
    
    // write result for this block to global mem 
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}
