
#include <hip/hip_runtime.h>
#define __nv50_blockIdx_x blockIdx.x
#define __nv50_blockIdx_y blockIdx.y
#define __nv50_blockIdx_z blockIdx.z
#define __nv50_threadIdx_x threadIdx.x
#define __nv50_threadIdx_y threadIdx.y
#define __nv50_threadIdx_z threadIdx.z
#define __nv50_blockdim_x blockDim.x
#define __nv50_blockdim_y blockDim.y
#define __nv50_blockdim_z blockDim.z
#define __nv50_griddim_x gridDim.x
#define __nv50_griddim_y gridDim.y
#define __nv50_griddim_z gridDim.z

extern "C" __global__ void __accrg_main_1_1(
  double * X,
  long long N)
{
  
  int i;
  int __acc_tmp_0;
  
  i = __nv50_blockIdx_x * __nv50_blockdim_x;
  i = __nv50_threadIdx_x + i;
  __acc_tmp_0 = __nv50_blockdim_x * __nv50_griddim_x;
  while(N > (long long) i)
  {
    if(i >= (int)(0LL))
    {
      //(*X0)[(long long) i] = (*X0)[(long long) i] * 2.0;
      *(X+(long long) i) = *(X +(long long) i) * 2.0;
    }
    i = __acc_tmp_0 + i;
  }
} /* __accrg_main_1_1 */

