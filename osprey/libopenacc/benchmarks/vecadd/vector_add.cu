
#include <hip/hip_runtime.h>

extern "C" __global__ void vector_addition(double* A, double* B, double* C, int N)
{
	int tx = threadIdx.x + blockIdx.x * blockDim.x;
	if(tx < N)
		C[tx] = A[tx] + B[tx];
}
