#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__x_solve_347__5ivlez75(hmpprt::u64* __hmpp_vla_sizes__rhsX_1, hmpprt::s32 nz2_1, double* rhs, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j, hmpprt::s32* __hmpp_addr__k_1, double* rhsX_1)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__x_solve_347__5ivlez75_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u64>  __hmpp_vla_sizes__rhsX_2, hmpprt::s32 nz2_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs_2, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32* __hmpp_addr__k, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhsX_2)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__x_solve_347__5ivlez75_parallel_region_1 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__x_solve_347__5ivlez75_parallel_region_1(hmpprt::s32 nz2, double* rhs_1, hmpprt::u64* __hmpp_vla_sizes__rhsX, double* rhsX);
#endif // __HIPCC__




# 5 "<preprocessor>"

#ifdef __HIPCC__

extern "C" __global__ void __hmpp_acc_region__x_solve_347__5ivlez75_parallel_region_1(hmpprt::s32 nz2, double* rhs_1, hmpprt::u64* __hmpp_vla_sizes__rhsX, double* rhsX)
{
 # 7 "<preprocessor>"
 hmpprt::s32 iter_per_gang_1;
 # 7 "<preprocessor>"
 hmpprt::s32 first_gang_iter_1;
 # 7 "<preprocessor>"
 hmpprt::s32 last_gang_iter_1;
 # 7 "<preprocessor>"
 iter_per_gang_1 = 1 + nz2 / 192;
 # 7 "<preprocessor>"
 first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
 # 7 "<preprocessor>"
 last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < nz2 ? (first_gang_iter_1 + iter_per_gang_1 - 1) : nz2);
 # 7 "<preprocessor>"
 hmpprt::s32 k_1;
 # 348 "x_solve.c"
 # 348 "x_solve.c"
 for (k_1 = first_gang_iter_1 ; k_1 <= last_gang_iter_1 ; k_1 = k_1 + 1)
 {
  # 8 "<preprocessor>"
  hmpprt::s32 j_1;
  # 350 "x_solve.c"
  # 350 "x_solve.c"
  for (j_1 = (hmpprt::gr_btidx()) ; j_1 <= 36 ; j_1 = j_1 + (hmpprt::gr_btnumx()))
  {
   # 9 "<preprocessor>"
   hmpprt::s32 i_2;
   # 352 "x_solve.c"
   # 352 "x_solve.c"
   for (i_2 = 0 ; i_2 <= 36 ; i_2 = i_2 + 1)
   {
    # 353 "x_solve.c"
    *(rhs_1 + i_2 + 37uLL * (j_1 + 37uLL * k_1)) = *(rhsX + j_1 + *(__hmpp_vla_sizes__rhsX + 3) * (i_2 + *(__hmpp_vla_sizes__rhsX + 2) * k_1));
    # 354 "x_solve.c"
    *(rhs_1 + i_2 + 37uLL * (j_1 + 37uLL * (k_1 + 36uLL))) = *(rhsX + j_1 + *(__hmpp_vla_sizes__rhsX + 3) * (i_2 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + *(__hmpp_vla_sizes__rhsX + 1))));
    # 355 "x_solve.c"
    *(rhs_1 + i_2 + 37uLL * (j_1 + 37uLL * (k_1 + 72uLL))) = *(rhsX + j_1 + *(__hmpp_vla_sizes__rhsX + 3) * (i_2 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + *(__hmpp_vla_sizes__rhsX + 1) * 2)));
    # 356 "x_solve.c"
    *(rhs_1 + i_2 + 37uLL * (j_1 + 37uLL * (k_1 + 108uLL))) = *(rhsX + j_1 + *(__hmpp_vla_sizes__rhsX + 3) * (i_2 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + *(__hmpp_vla_sizes__rhsX + 1) * 3)));
    # 357 "x_solve.c"
    *(rhs_1 + i_2 + 37uLL * (j_1 + 37uLL * (k_1 + 144uLL))) = *(rhsX + j_1 + *(__hmpp_vla_sizes__rhsX + 3) * (i_2 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + *(__hmpp_vla_sizes__rhsX + 1) * 4)));
   }
   # 5 "<preprocessor>"
  }
  # 5 "<preprocessor>"
 }
 # 5 "<preprocessor>"
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__x_solve_347__5ivlez75_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u64>  __hmpp_vla_sizes__rhsX_2, hmpprt::s32 nz2_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs_2, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32* __hmpp_addr__k, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhsX_2)
{
 # 7 "<preprocessor>"
 hmpprt::s32 k;
 # 8 "<preprocessor>"
 hmpprt::s32 j;
 # 8 "<preprocessor>"
 j = *__hmpp_addr__j_1;
 # 9 "<preprocessor>"
 hmpprt::s32 i_1;
 # 9 "<preprocessor>"
 i_1 = *__hmpp_addr__i_1;
 # 34 "<preprocessor>"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(1);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nz2_2), "nz2");
  __hmppcg_call.addLocalParameter(&rhs_2, 8, "rhs_1");
  __hmppcg_call.addLocalParameter(&__hmpp_vla_sizes__rhsX_2, 8, "__hmpp_vla_sizes__rhsX");
  __hmppcg_call.addLocalParameter(&rhsX_2, 8, "rhsX");
  __hmppcg_call.launch(__hmpp_acc_region__x_solve_347__5ivlez75_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 34 "<preprocessor>"
 *__hmpp_addr__i_1 = i_1;
 # 35 "<preprocessor>"
 *__hmpp_addr__j_1 = j;
 # 36 "<preprocessor>"
 *__hmpp_addr__k = k;
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__x_solve_347__5ivlez75(hmpprt::u64* __hmpp_vla_sizes__rhsX_1, hmpprt::s32 nz2_1, double* rhs, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j, hmpprt::s32* __hmpp_addr__k_1, double* rhsX_1)
{
 # 1 "<preprocessor>"
 (__hmpp_acc_region__x_solve_347__5ivlez75_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u64> (__hmpp_vla_sizes__rhsX_1), nz2_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (rhs), __hmpp_addr__i, __hmpp_addr__j, __hmpp_addr__k_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (rhsX_1)));
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      __hmpp_acc_region__x_solve_347__5ivlez75_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "__hmpp_acc_region__x_solve_347__5ivlez75_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("__hmpp_acc_region__x_solve_347__5ivlez75", "prototype __hmpp_acc_region__x_solve_347__5ivlez75(__hmpp_vla_sizes__rhsX: ^cudaglob u64, nz2: s32, rhs: ^cudaglob double, __hmpp_addr__i: ^host s32, __hmpp_addr__j: ^host s32, __hmpp_addr__k: ^host s32, rhsX: ^cudaglob double)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete __hmpp_acc_region__x_solve_347__5ivlez75_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
