#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__compute_rhs_98__7xoelmjh(hmpprt::s32 nx2, hmpprt::s32 ny2, hmpprt::s32 nz2, double c1, double c2_2, double con43, double* u, double* us_1, double* vs_2, double* ws_2, double* qs_2, double* rho_i, double* square, double* rhs_2, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32* __hmpp_addr__k_1)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__compute_rhs_98__7xoelmjh_internal_1(hmpprt::s32 nx2_2, hmpprt::s32 ny2_2, hmpprt::s32 nz2_2, double c1_2, double c2, double con43_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  u_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  us_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  vs, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  ws, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  qs, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rho_i_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  square_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j, hmpprt::s32* __hmpp_addr__k)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__compute_rhs_98__7xoelmjh_parallel_region_1 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__compute_rhs_98__7xoelmjh_parallel_region_1(double c1_1, double c2_1, double con43_1, double dx1tx1_1, double dx2tx1_1, double dx3tx1_1, double dx4tx1, double dx5tx1_1, hmpprt::s32 nx2_1, hmpprt::s32 ny2_1, hmpprt::s32 nz2_1, double* qs_1, double* rho_i_1, double* rhs_1, double* square_1, double tx2_1, double* u_1, double* us, double* vs_1, double* ws_1, double xxcon2_1, double xxcon3_1, double xxcon4, double xxcon5);
#endif // __HIPCC__




# 5 "<preprocessor>"

#ifdef __HIPCC__

extern "C" __global__ void __hmpp_acc_region__compute_rhs_98__7xoelmjh_parallel_region_1(double c1_1, double c2_1, double con43_1, double dx1tx1_1, double dx2tx1_1, double dx3tx1_1, double dx4tx1, double dx5tx1_1, hmpprt::s32 nx2_1, hmpprt::s32 ny2_1, hmpprt::s32 nz2_1, double* qs_1, double* rho_i_1, double* rhs_1, double* square_1, double tx2_1, double* u_1, double* us, double* vs_1, double* ws_1, double xxcon2_1, double xxcon3_1, double xxcon4, double xxcon5)
{
 # 10 "<preprocessor>"
 hmpprt::s32 iter_per_gang_1;
 # 10 "<preprocessor>"
 hmpprt::s32 first_gang_iter_1;
 # 10 "<preprocessor>"
 hmpprt::s32 last_gang_iter_1;
 # 10 "<preprocessor>"
 iter_per_gang_1 = 1 + (nz2_1 - 1) / 192;
 # 10 "<preprocessor>"
 first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
 # 10 "<preprocessor>"
 last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < (nz2_1 - 1) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : (nz2_1 - 1));
 # 10 "<preprocessor>"
 hmpprt::s32 k_1;
 # 99 "rhs.c"
 # 99 "rhs.c"
 for (k_1 = first_gang_iter_1 ; k_1 <= last_gang_iter_1 ; k_1 = k_1 + 1)
 {
  # 11 "<preprocessor>"
  hmpprt::s32 end_2;
  # 11 "<preprocessor>"
  hmpprt::s32 j_1;
  # 101 "rhs.c"
  # 101 "rhs.c"
  # 101 "rhs.c"
  for (j_1 = (hmpprt::gr_btidx()), end_2 = ny2_1 - 1 ; j_1 <= end_2 ; j_1 = j_1 + (hmpprt::gr_btnumx()))
  {
   # 12 "<preprocessor>"
   hmpprt::s32 end_1;
   # 12 "<preprocessor>"
   hmpprt::s32 i_2;
   # 103 "rhs.c"
   # 103 "rhs.c"
   # 103 "rhs.c"
   for (i_2 = 0, end_1 = nx2_1 - 1 ; i_2 <= end_1 ; i_2 = i_2 + 1)
   {
    # 104 "rhs.c"
    double up1_1;
    # 104 "rhs.c"
    double uijk_1;
    # 104 "rhs.c"
    double um1_1;
    # 104 "rhs.c"
    uijk_1 = *(us + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1)));
    # 105 "rhs.c"
    up1_1 = *(us + (i_2 + 2) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1)));
    # 106 "rhs.c"
    um1_1 = *(us + i_2 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1)));
    # 110 "rhs.c"
    *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) = *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) + dx1tx1_1 * (*(u_1 + (i_2 + 2) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) - (double) 2.0 * *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) + *(u_1 + i_2 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1)))) - tx2_1 * (*(u_1 + (i_2 + 2) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 37))) - *(u_1 + i_2 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 37))));
    # 117 "rhs.c"
    *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 37))) = *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 37))) + dx2tx1_1 * (*(u_1 + (i_2 + 2) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 37))) - (double) 2.0 * *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 37))) + *(u_1 + i_2 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 37)))) + xxcon2_1 * con43_1 * (up1_1 - (double) 2.0 * uijk_1 + um1_1) - tx2_1 * (*(u_1 + (i_2 + 2) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 37))) * up1_1 - *(u_1 + i_2 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 37))) * um1_1 + (*(u_1 + (i_2 + 2) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 145))) - *(square_1 + (i_2 + 2) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) - *(u_1 + i_2 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 145))) + *(square_1 + i_2 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1)))) * c2_1);
    # 122 "rhs.c"
    *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 73))) = *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 73))) + dx3tx1_1 * (*(u_1 + (i_2 + 2) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 73))) - (double) 2.0 * *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 73))) + *(u_1 + i_2 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 73)))) + xxcon2_1 * (*(vs_1 + (i_2 + 2) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) - (double) 2.0 * *(vs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) + *(vs_1 + i_2 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1)))) - tx2_1 * (*(u_1 + (i_2 + 2) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 73))) * up1_1 - *(u_1 + i_2 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 73))) * um1_1);
    # 127 "rhs.c"
    *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 109))) = *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 109))) + dx4tx1 * (*(u_1 + (i_2 + 2) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 109))) - (double) 2.0 * *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 109))) + *(u_1 + i_2 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 109)))) + xxcon2_1 * (*(ws_1 + (i_2 + 2) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) - (double) 2.0 * *(ws_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) + *(ws_1 + i_2 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1)))) - tx2_1 * (*(u_1 + (i_2 + 2) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 109))) * up1_1 - *(u_1 + i_2 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 109))) * um1_1);
    # 137 "rhs.c"
    *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 145))) = *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 145))) + dx5tx1_1 * (*(u_1 + (i_2 + 2) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 145))) - (double) 2.0 * *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 145))) + *(u_1 + i_2 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 145)))) + xxcon3_1 * (*(qs_1 + (i_2 + 2) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) - (double) 2.0 * *(qs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) + *(qs_1 + i_2 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1)))) + xxcon4 * (up1_1 * up1_1 - (double) 2.0 * uijk_1 * uijk_1 + um1_1 * um1_1) + xxcon5 * (*(u_1 + (i_2 + 2) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 145))) * *(rho_i_1 + (i_2 + 2) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) - (double) 2.0 * *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 145))) * *(rho_i_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) + *(u_1 + i_2 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 145))) * *(rho_i_1 + i_2 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1)))) - tx2_1 * ((c1_1 * *(u_1 + (i_2 + 2) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 145))) - c2_1 * *(square_1 + (i_2 + 2) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1)))) * up1_1 - (c1_1 * *(u_1 + i_2 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 145))) - c2_1 * *(square_1 + i_2 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1)))) * um1_1);
   }
   # 5 "<preprocessor>"
  }
  # 5 "<preprocessor>"
 }
 # 5 "<preprocessor>"
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__compute_rhs_98__7xoelmjh_internal_1(hmpprt::s32 nx2_2, hmpprt::s32 ny2_2, hmpprt::s32 nz2_2, double c1_2, double c2, double con43_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  u_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  us_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  vs, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  ws, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  qs, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rho_i_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  square_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j, hmpprt::s32* __hmpp_addr__k)
{
 # 10 "<preprocessor>"
 hmpprt::s32 k;
 # 11 "<preprocessor>"
 hmpprt::s32 j;
 # 11 "<preprocessor>"
 j = *__hmpp_addr__j;
 # 12 "<preprocessor>"
 hmpprt::s32 i_1;
 # 12 "<preprocessor>"
 i_1 = *__hmpp_addr__i_1;
 # 13 "<preprocessor>"
 double dx5tx1;
 # 14 "<preprocessor>"
 double dx4tx1_1;
 # 15 "<preprocessor>"
 double dx3tx1;
 # 16 "<preprocessor>"
 double dx2tx1;
 # 17 "<preprocessor>"
 double dx1tx1;
 # 18 "<preprocessor>"
 double xxcon5_1;
 # 19 "<preprocessor>"
 double xxcon4_1;
 # 20 "<preprocessor>"
 double xxcon3;
 # 21 "<preprocessor>"
 double xxcon2;
 # 22 "<preprocessor>"
 double tx2;
 # 76 "<preprocessor>"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(1);
  __hmppcg_call.addLocalParameter(&c1_2, 8, "c1_1");
  __hmppcg_call.addLocalParameter(&c2, 8, "c2_1");
  __hmppcg_call.addLocalParameter(&con43_2, 8, "con43_1");
  __hmppcg_call.addLocalParameter(&dx1tx1, 8, "dx1tx1_1");
  __hmppcg_call.addLocalParameter(&dx2tx1, 8, "dx2tx1_1");
  __hmppcg_call.addLocalParameter(&dx3tx1, 8, "dx3tx1_1");
  __hmppcg_call.addLocalParameter(&dx4tx1_1, 8, "dx4tx1");
  __hmppcg_call.addLocalParameter(&dx5tx1, 8, "dx5tx1_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nx2_2), "nx2_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (ny2_2), "ny2_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nz2_2), "nz2_1");
  __hmppcg_call.addLocalParameter(&qs, 8, "qs_1");
  __hmppcg_call.addLocalParameter(&rho_i_2, 8, "rho_i_1");
  __hmppcg_call.addLocalParameter(&rhs, 8, "rhs_1");
  __hmppcg_call.addLocalParameter(&square_2, 8, "square_1");
  __hmppcg_call.addLocalParameter(&tx2, 8, "tx2_1");
  __hmppcg_call.addLocalParameter(&u_2, 8, "u_1");
  __hmppcg_call.addLocalParameter(&us_2, 8, "us");
  __hmppcg_call.addLocalParameter(&vs, 8, "vs_1");
  __hmppcg_call.addLocalParameter(&ws, 8, "ws_1");
  __hmppcg_call.addLocalParameter(&xxcon2, 8, "xxcon2_1");
  __hmppcg_call.addLocalParameter(&xxcon3, 8, "xxcon3_1");
  __hmppcg_call.addLocalParameter(&xxcon4_1, 8, "xxcon4");
  __hmppcg_call.addLocalParameter(&xxcon5_1, 8, "xxcon5");
  __hmppcg_call.launch(__hmpp_acc_region__compute_rhs_98__7xoelmjh_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 76 "<preprocessor>"
 *__hmpp_addr__i_1 = i_1;
 # 77 "<preprocessor>"
 *__hmpp_addr__j = j;
 # 78 "<preprocessor>"
 *__hmpp_addr__k = k;
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__compute_rhs_98__7xoelmjh(hmpprt::s32 nx2, hmpprt::s32 ny2, hmpprt::s32 nz2, double c1, double c2_2, double con43, double* u, double* us_1, double* vs_2, double* ws_2, double* qs_2, double* rho_i, double* square, double* rhs_2, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32* __hmpp_addr__k_1)
{
 # 1 "<preprocessor>"
 (__hmpp_acc_region__compute_rhs_98__7xoelmjh_internal_1(nx2, ny2, nz2, c1, c2_2, con43, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (u), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (us_1), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (vs_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (ws_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (qs_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (rho_i), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (square), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (rhs_2), __hmpp_addr__i, __hmpp_addr__j_1, __hmpp_addr__k_1));
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      __hmpp_acc_region__compute_rhs_98__7xoelmjh_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "__hmpp_acc_region__compute_rhs_98__7xoelmjh_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("__hmpp_acc_region__compute_rhs_98__7xoelmjh", "prototype __hmpp_acc_region__compute_rhs_98__7xoelmjh(nx2: s32, ny2: s32, nz2: s32, c1: double, c2: double, con43: double, u: ^cudaglob double, us: ^cudaglob double, vs: ^cudaglob double, ws: ^cudaglob double, qs: ^cudaglob double, rho_i: ^cudaglob double, square: ^cudaglob double, rhs: ^cudaglob double, __hmpp_addr__i: ^host s32, __hmpp_addr__j: ^host s32, __hmpp_addr__k: ^host s32)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete __hmpp_acc_region__compute_rhs_98__7xoelmjh_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
