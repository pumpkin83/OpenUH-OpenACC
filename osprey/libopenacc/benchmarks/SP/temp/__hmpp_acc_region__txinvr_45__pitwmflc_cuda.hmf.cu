#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__txinvr_45__pitwmflc()
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__txinvr_45__pitwmflc_internal_1()
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__txinvr_45__pitwmflc_parallel_region_1 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__txinvr_45__pitwmflc_parallel_region_1(hmpprt::s32 nx2_1, hmpprt::s32 ny2, hmpprt::s32 nz2_1);
#endif // __HIPCC__




# 5 "<preprocessor>"

#ifdef __HIPCC__

extern "C" __global__ void __hmpp_acc_region__txinvr_45__pitwmflc_parallel_region_1(hmpprt::s32 nx2_1, hmpprt::s32 ny2, hmpprt::s32 nz2_1)
{
 # 21 "<preprocessor>"
 hmpprt::s32 iter_per_gang_1;
 # 21 "<preprocessor>"
 hmpprt::s32 first_gang_iter_1;
 # 21 "<preprocessor>"
 hmpprt::s32 last_gang_iter_1;
 # 21 "<preprocessor>"
 iter_per_gang_1 = 1 + (nz2_1 - 1) / 192;
 # 21 "<preprocessor>"
 first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
 # 21 "<preprocessor>"
 last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < (nz2_1 - 1) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : (nz2_1 - 1));
 # 21 "<preprocessor>"
 hmpprt::s32 k_1;
 # 47 "txinvr.c"
 # 47 "txinvr.c"
 for (k_1 = first_gang_iter_1 ; k_1 <= last_gang_iter_1 ; k_1 = k_1 + 1)
 {
  # 22 "<preprocessor>"
  hmpprt::s32 end_2;
  # 22 "<preprocessor>"
  hmpprt::s32 j_1;
  # 48 "txinvr.c"
  # 48 "txinvr.c"
  # 48 "txinvr.c"
  for (j_1 = (hmpprt::gr_btidy()), end_2 = ny2 - 1 ; j_1 <= end_2 ; j_1 = j_1 + (hmpprt::gr_btnumy()))
  {
   # 23 "<preprocessor>"
   hmpprt::s32 end_1;
   # 23 "<preprocessor>"
   hmpprt::s32 i_2;
   # 49 "txinvr.c"
   # 49 "txinvr.c"
   # 49 "txinvr.c"
   for (i_2 = (hmpprt::gr_btidx()), end_1 = nx2_1 - 1 ; i_2 <= end_1 ; i_2 = i_2 + (hmpprt::gr_btnumx()))
   {
    }
   # 5 "<preprocessor>"
  }
  # 5 "<preprocessor>"
 }
 # 5 "<preprocessor>"
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__txinvr_45__pitwmflc_internal_1()
{
 # 33 "<preprocessor>"
 hmpprt::s32 nz2;
 # 34 "<preprocessor>"
 hmpprt::s32 ny2_1;
 # 35 "<preprocessor>"
 hmpprt::s32 nx2;
 # 5 "<preprocessor>"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(8LL);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nx2), "nx2_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (ny2_1), "ny2");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nz2), "nz2_1");
  __hmppcg_call.launch(__hmpp_acc_region__txinvr_45__pitwmflc_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__txinvr_45__pitwmflc()
{
 # 1 "<preprocessor>"
 (__hmpp_acc_region__txinvr_45__pitwmflc_internal_1());
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      __hmpp_acc_region__txinvr_45__pitwmflc_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "__hmpp_acc_region__txinvr_45__pitwmflc_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("__hmpp_acc_region__txinvr_45__pitwmflc", "prototype __hmpp_acc_region__txinvr_45__pitwmflc()");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete __hmpp_acc_region__txinvr_45__pitwmflc_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
