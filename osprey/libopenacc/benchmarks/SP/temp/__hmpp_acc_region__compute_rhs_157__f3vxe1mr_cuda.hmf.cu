#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__compute_rhs_157__f3vxe1mr(hmpprt::s32 ny2_1, hmpprt::s32 nz2_1, double dssp_2, double* u_1, double* rhs_1, hmpprt::s32 i_2, hmpprt::s32* __hmpp_addr__j, hmpprt::s32* __hmpp_addr__k_1, hmpprt::s32* __hmpp_addr__m)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__compute_rhs_157__f3vxe1mr_internal_1(hmpprt::s32 ny2_2, hmpprt::s32 nz2_2, double dssp_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  u_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs_2, hmpprt::s32 i_3, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32* __hmpp_addr__k, hmpprt::s32* __hmpp_addr__m_1)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__compute_rhs_157__f3vxe1mr_parallel_region_1 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__compute_rhs_157__f3vxe1mr_parallel_region_1(double dssp, hmpprt::s32 i_1, hmpprt::s32* m_2, hmpprt::s32 ny2, hmpprt::s32 nz2, double* rhs, double* u);
#endif // __HIPCC__




# 5 "<preprocessor>"

#ifdef __HIPCC__

extern "C" __global__ void __hmpp_acc_region__compute_rhs_157__f3vxe1mr_parallel_region_1(double dssp, hmpprt::s32 i_1, hmpprt::s32* m_2, hmpprt::s32 ny2, hmpprt::s32 nz2, double* rhs, double* u)
{
 # 8 "<preprocessor>"
 hmpprt::s32 iter_per_gang_1;
 # 8 "<preprocessor>"
 hmpprt::s32 first_gang_iter_1;
 # 8 "<preprocessor>"
 hmpprt::s32 last_gang_iter_1;
 # 8 "<preprocessor>"
 iter_per_gang_1 = 1 + (nz2 - 1) / 192;
 # 8 "<preprocessor>"
 first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
 # 8 "<preprocessor>"
 last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < (nz2 - 1) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : (nz2 - 1));
 # 8 "<preprocessor>"
 hmpprt::s32 k_1;
 # 158 "rhs.c"
 # 158 "rhs.c"
 for (k_1 = first_gang_iter_1 ; k_1 <= last_gang_iter_1 ; k_1 = k_1 + 1)
 {
  # 9 "<preprocessor>"
  hmpprt::s32 end_2;
  # 9 "<preprocessor>"
  hmpprt::s32 j_1;
  # 160 "rhs.c"
  # 160 "rhs.c"
  # 160 "rhs.c"
  for (j_1 = (hmpprt::gr_btidx()), end_2 = ny2 - 1 ; j_1 <= end_2 ; j_1 = j_1 + (hmpprt::gr_btnumx()))
  {
   # 7 "<preprocessor>"
   hmpprt::s32 m_1;
   # 161 "rhs.c"
   # 161 "rhs.c"
   for (m_1 = 0 ; m_1 <= 4 ; m_1 = m_1 + 1)
   {
    # 164 "rhs.c"
    *(rhs + i_1 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1 + 36uLL * m_1))) = *(rhs + i_1 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1 + 36uLL * m_1))) - dssp * ((double) -4.0 * *(u + (i_1 - 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1 + 36uLL * m_1))) + (double) 6.0 * *(u + i_1 + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1 + 36uLL * m_1))) - (double) 4.0 * *(u + (i_1 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1 + 36uLL * m_1))) + *(u + (i_1 + 2) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1 + 36uLL * m_1))));
   }
   # 5 "<preprocessor>"
   # 5 "<preprocessor>"
   *m_2 = 5;
  }
  # 5 "<preprocessor>"
 }
 # 5 "<preprocessor>"
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__compute_rhs_157__f3vxe1mr_internal_1(hmpprt::s32 ny2_2, hmpprt::s32 nz2_2, double dssp_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  u_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs_2, hmpprt::s32 i_3, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32* __hmpp_addr__k, hmpprt::s32* __hmpp_addr__m_1)
{
 # 7 "<preprocessor>"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  m;
 # 7 "<preprocessor>"
 hmpprt::Context::getInstance()->allocate((void **) (&m), hmpprt::MS_CUDA_GLOB, 4);
 # 7 "<preprocessor>"
 *m = *__hmpp_addr__m_1;
 # 8 "<preprocessor>"
 hmpprt::s32 k;
 # 9 "<preprocessor>"
 hmpprt::s32 j;
 # 9 "<preprocessor>"
 j = *__hmpp_addr__j_1;
 # 28 "<preprocessor>"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(1);
  __hmppcg_call.addLocalParameter(&dssp_1, 8, "dssp");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (i_3), "i_1");
  __hmppcg_call.addLocalParameter(&m, 8, "m_2");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (ny2_2), "ny2");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nz2_2), "nz2");
  __hmppcg_call.addLocalParameter(&rhs_2, 8, "rhs");
  __hmppcg_call.addLocalParameter(&u_2, 8, "u");
  __hmppcg_call.launch(__hmpp_acc_region__compute_rhs_157__f3vxe1mr_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 28 "<preprocessor>"
 *__hmpp_addr__j_1 = j;
 # 29 "<preprocessor>"
 *__hmpp_addr__k = k;
 # 30 "<preprocessor>"
 *__hmpp_addr__m_1 = *m;
 # 5 "<preprocessor>"
 hmpprt::Context::getInstance()->free((void **) (&m));
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__compute_rhs_157__f3vxe1mr(hmpprt::s32 ny2_1, hmpprt::s32 nz2_1, double dssp_2, double* u_1, double* rhs_1, hmpprt::s32 i_2, hmpprt::s32* __hmpp_addr__j, hmpprt::s32* __hmpp_addr__k_1, hmpprt::s32* __hmpp_addr__m)
{
 # 1 "<preprocessor>"
 (__hmpp_acc_region__compute_rhs_157__f3vxe1mr_internal_1(ny2_1, nz2_1, dssp_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (u_1), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (rhs_1), i_2, __hmpp_addr__j, __hmpp_addr__k_1, __hmpp_addr__m));
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      __hmpp_acc_region__compute_rhs_157__f3vxe1mr_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "__hmpp_acc_region__compute_rhs_157__f3vxe1mr_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("__hmpp_acc_region__compute_rhs_157__f3vxe1mr", "prototype __hmpp_acc_region__compute_rhs_157__f3vxe1mr(ny2: s32, nz2: s32, dssp: double, u: ^cudaglob double, rhs: ^cudaglob double, i: s32, __hmpp_addr__j: ^host s32, __hmpp_addr__k: ^host s32, __hmpp_addr__m: ^host s32)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete __hmpp_acc_region__compute_rhs_157__f3vxe1mr_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
