#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__compute_rhs_81__5ypcgwah(hmpprt::s32* grid_points, double* rhs_1, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j, hmpprt::s32* __hmpp_addr__k, hmpprt::s32* __hmpp_addr__m_1)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__compute_rhs_81__5ypcgwah_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  grid_points_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs_2, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32* __hmpp_addr__k_1, hmpprt::s32* __hmpp_addr__m)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__compute_rhs_81__5ypcgwah_parallel_region_1 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__compute_rhs_81__5ypcgwah_parallel_region_1(double* forcing, hmpprt::s32* grid_points_1, hmpprt::s32* m_2, double* rhs);
#endif // __HIPCC__




# 5 "<preprocessor>"

#ifdef __HIPCC__

extern "C" __global__ void __hmpp_acc_region__compute_rhs_81__5ypcgwah_parallel_region_1(double* forcing, hmpprt::s32* grid_points_1, hmpprt::s32* m_2, double* rhs)
{
 # 8 "<preprocessor>"
 hmpprt::s32 iter_per_gang_1;
 # 8 "<preprocessor>"
 hmpprt::s32 first_gang_iter_1;
 # 8 "<preprocessor>"
 hmpprt::s32 last_gang_iter_1;
 # 8 "<preprocessor>"
 iter_per_gang_1 = 1 + (*(grid_points_1 + 2) - 1) / 192;
 # 8 "<preprocessor>"
 first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
 # 8 "<preprocessor>"
 last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < (*(grid_points_1 + 2) - 1) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : (*(grid_points_1 + 2) - 1));
 # 8 "<preprocessor>"
 hmpprt::s32 k_1;
 # 82 "rhs.c"
 # 82 "rhs.c"
 for (k_1 = first_gang_iter_1 ; k_1 <= last_gang_iter_1 ; k_1 = k_1 + 1)
 {
  # 9 "<preprocessor>"
  hmpprt::s32 end_3;
  # 9 "<preprocessor>"
  hmpprt::s32 j_1;
  # 84 "rhs.c"
  # 84 "rhs.c"
  # 84 "rhs.c"
  for (j_1 = (hmpprt::gr_btidx()), end_3 = *(grid_points_1 + 1) - 1 ; j_1 <= end_3 ; j_1 = j_1 + (hmpprt::gr_btnumx()))
  {
   # 10 "<preprocessor>"
   hmpprt::s32 end_2;
   # 10 "<preprocessor>"
   hmpprt::s32 i_2;
   # 86 "rhs.c"
   # 86 "rhs.c"
   # 86 "rhs.c"
   for (i_2 = 0, end_2 = *grid_points_1 - 1 ; i_2 <= end_2 ; i_2 = i_2 + 1)
   {
    # 7 "<preprocessor>"
    hmpprt::s32 m_1;
    # 87 "rhs.c"
    # 87 "rhs.c"
    for (m_1 = 0 ; m_1 <= 4 ; m_1 = m_1 + 1)
    {
     # 88 "rhs.c"
     *(rhs + i_2 + 37uLL * (j_1 + 37uLL * (k_1 + 36uLL * m_1))) = *(forcing + i_2 + 37uLL * (k_1 + 36uLL * (m_1 + 5uLL * j_1)));
    }
    # 5 "<preprocessor>"
    # 5 "<preprocessor>"
    *m_2 = 5;
   }
   # 5 "<preprocessor>"
  }
  # 5 "<preprocessor>"
 }
 # 5 "<preprocessor>"
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__compute_rhs_81__5ypcgwah_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  grid_points_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs_2, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32* __hmpp_addr__k_1, hmpprt::s32* __hmpp_addr__m)
{
 # 7 "<preprocessor>"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  m;
 # 7 "<preprocessor>"
 hmpprt::Context::getInstance()->allocate((void **) (&m), hmpprt::MS_CUDA_GLOB, 4);
 # 7 "<preprocessor>"
 *m = *__hmpp_addr__m;
 # 8 "<preprocessor>"
 hmpprt::s32 k;
 # 9 "<preprocessor>"
 hmpprt::s32 j;
 # 9 "<preprocessor>"
 j = *__hmpp_addr__j_1;
 # 10 "<preprocessor>"
 hmpprt::s32 i_1;
 # 10 "<preprocessor>"
 i_1 = *__hmpp_addr__i;
 # 11 "<preprocessor>"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  forcing_1;
 # 34 "<preprocessor>"
 hmpprt::Context::getInstance()->allocate((void **) (&forcing_1), hmpprt::MS_CUDA_GLOB, 1971360uLL);
 # 34 "<preprocessor>"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(1);
  __hmppcg_call.addLocalParameter(&forcing_1, 8, "forcing");
  __hmppcg_call.addLocalParameter(&grid_points_2, 8, "grid_points_1");
  __hmppcg_call.addLocalParameter(&m, 8, "m_2");
  __hmppcg_call.addLocalParameter(&rhs_2, 8, "rhs");
  __hmppcg_call.launch(__hmpp_acc_region__compute_rhs_81__5ypcgwah_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 34 "<preprocessor>"
 *__hmpp_addr__i = i_1;
 # 35 "<preprocessor>"
 *__hmpp_addr__j_1 = j;
 # 36 "<preprocessor>"
 *__hmpp_addr__k_1 = k;
 # 37 "<preprocessor>"
 *__hmpp_addr__m = *m;
 # 5 "<preprocessor>"
 hmpprt::Context::getInstance()->free((void **) (&m));
 # 5 "<preprocessor>"
 hmpprt::Context::getInstance()->free((void **) (&forcing_1));
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__compute_rhs_81__5ypcgwah(hmpprt::s32* grid_points, double* rhs_1, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j, hmpprt::s32* __hmpp_addr__k, hmpprt::s32* __hmpp_addr__m_1)
{
 # 1 "<preprocessor>"
 (__hmpp_acc_region__compute_rhs_81__5ypcgwah_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32> (grid_points), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (rhs_1), __hmpp_addr__i_1, __hmpp_addr__j, __hmpp_addr__k, __hmpp_addr__m_1));
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      __hmpp_acc_region__compute_rhs_81__5ypcgwah_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "__hmpp_acc_region__compute_rhs_81__5ypcgwah_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("__hmpp_acc_region__compute_rhs_81__5ypcgwah", "prototype __hmpp_acc_region__compute_rhs_81__5ypcgwah(grid_points: ^cudaglob s32, rhs: ^cudaglob double, __hmpp_addr__i: ^host s32, __hmpp_addr__j: ^host s32, __hmpp_addr__k: ^host s32, __hmpp_addr__m: ^host s32)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete __hmpp_acc_region__compute_rhs_81__5ypcgwah_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
