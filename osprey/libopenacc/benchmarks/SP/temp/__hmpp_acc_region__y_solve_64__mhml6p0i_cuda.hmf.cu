#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__y_solve_64__mhml6p0i(hmpprt::s32 ni_2, hmpprt::s32 nj_1, hmpprt::s32 gp0_2, hmpprt::s32 gp1, hmpprt::s32 gp2_2)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__y_solve_64__mhml6p0i_internal_1(hmpprt::s32 ni, hmpprt::s32 nj, hmpprt::s32 gp0, hmpprt::s32 gp1_2, hmpprt::s32 gp2)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__y_solve_64__mhml6p0i_parallel_region_1 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__y_solve_64__mhml6p0i_parallel_region_1(hmpprt::s32 gp0_1, hmpprt::s32 gp1_1, hmpprt::s32 gp2_1, hmpprt::s32 ni_1);
#endif // __HIPCC__




# 5 "<preprocessor>"

#ifdef __HIPCC__

extern "C" __global__ void __hmpp_acc_region__y_solve_64__mhml6p0i_parallel_region_1(hmpprt::s32 gp0_1, hmpprt::s32 gp1_1, hmpprt::s32 gp2_1, hmpprt::s32 ni_1)
{
 # 17 "<preprocessor>"
 hmpprt::s32 iter_per_gang_1;
 # 17 "<preprocessor>"
 hmpprt::s32 first_gang_iter_1;
 # 17 "<preprocessor>"
 hmpprt::s32 last_gang_iter_1;
 # 17 "<preprocessor>"
 iter_per_gang_1 = 1 + (gp2_1 - 3) / 192;
 # 17 "<preprocessor>"
 first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
 # 17 "<preprocessor>"
 last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < (gp2_1 - 3) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : (gp2_1 - 3));
 # 17 "<preprocessor>"
 hmpprt::s32 k_1;
 # 66 "y_solve.c"
 # 66 "y_solve.c"
 for (k_1 = first_gang_iter_1 ; k_1 <= last_gang_iter_1 ; k_1 = k_1 + 1)
 {
  # 19 "<preprocessor>"
  hmpprt::s32 end_2;
  # 19 "<preprocessor>"
  hmpprt::s32 i_2;
  # 68 "y_solve.c"
  # 68 "y_solve.c"
  # 68 "y_solve.c"
  for (i_2 = (hmpprt::gr_btidy()), end_2 = ni_1 - 1 ; i_2 <= end_2 ; i_2 = i_2 + (hmpprt::gr_btnumy()))
  {
   # 14 "<preprocessor>"
   hmpprt::s32 m_1;
   # 69 "y_solve.c"
   # 69 "y_solve.c"
   for (m_1 = (hmpprt::gr_btidx()) ; m_1 <= 4 ; m_1 = m_1 + (hmpprt::gr_btnumx()))
   {
    }
   # 19 "<preprocessor>"
  }
  # 19 "<preprocessor>"
  # 19 "<preprocessor>"
  hmpprt::s32 end_5;
  # 19 "<preprocessor>"
  hmpprt::s32 i_3;
  # 92 "y_solve.c"
  # 92 "y_solve.c"
  # 92 "y_solve.c"
  for (i_3 = (hmpprt::gr_btidy()), end_5 = gp0_1 - 3 ; i_3 <= end_5 ; i_3 = i_3 + (hmpprt::gr_btnumy()))
  {
   # 18 "<preprocessor>"
   hmpprt::s32 end_3;
   # 18 "<preprocessor>"
   hmpprt::s32 j_1;
   # 93 "y_solve.c"
   # 93 "y_solve.c"
   # 93 "y_solve.c"
   for (j_1 = (hmpprt::gr_btidx()), end_3 = gp1_1 - 1 ; j_1 <= end_3 ; j_1 = j_1 + (hmpprt::gr_btnumx()))
   {
    }
   # 18 "<preprocessor>"
   # 18 "<preprocessor>"
   hmpprt::s32 end_4;
   # 18 "<preprocessor>"
   hmpprt::s32 j_2;
   # 99 "y_solve.c"
   # 99 "y_solve.c"
   # 99 "y_solve.c"
   for (j_2 = (hmpprt::gr_btidx()), end_4 = gp1_1 - 3 ; j_2 <= end_4 ; j_2 = j_2 + (hmpprt::gr_btnumx()))
   {
    }
   # 19 "<preprocessor>"
  }
  # 19 "<preprocessor>"
  # 19 "<preprocessor>"
  hmpprt::s32 end_6;
  # 19 "<preprocessor>"
  hmpprt::s32 i_4;
  # 114 "y_solve.c"
  # 114 "y_solve.c"
  # 114 "y_solve.c"
  for (i_4 = (hmpprt::gr_btidy()), end_6 = gp0_1 - 3 ; i_4 <= end_6 ; i_4 = i_4 + (hmpprt::gr_btnumy()))
  {
   }
  # 18 "<preprocessor>"
  # 18 "<preprocessor>"
  hmpprt::s32 end_8;
  # 18 "<preprocessor>"
  hmpprt::s32 j_3;
  # 126 "y_solve.c"
  # 126 "y_solve.c"
  # 126 "y_solve.c"
  for (j_3 = (hmpprt::gr_btidy()), end_8 = gp1_1 - 7 ; j_3 <= end_8 ; j_3 = j_3 + (hmpprt::gr_btnumy()))
  {
   # 19 "<preprocessor>"
   hmpprt::s32 end_7;
   # 19 "<preprocessor>"
   hmpprt::s32 i_5;
   # 127 "y_solve.c"
   # 127 "y_solve.c"
   # 127 "y_solve.c"
   for (i_5 = (hmpprt::gr_btidx()), end_7 = gp0_1 - 3 ; i_5 <= end_7 ; i_5 = i_5 + (hmpprt::gr_btnumx()))
   {
    }
   # 19 "<preprocessor>"
  }
  # 19 "<preprocessor>"
  # 19 "<preprocessor>"
  hmpprt::s32 end_9;
  # 19 "<preprocessor>"
  hmpprt::s32 i_6;
  # 136 "y_solve.c"
  # 136 "y_solve.c"
  # 136 "y_solve.c"
  for (i_6 = (hmpprt::gr_btidy()), end_9 = gp0_1 - 3 ; i_6 <= end_9 ; i_6 = i_6 + (hmpprt::gr_btnumy()))
  {
   }
  # 18 "<preprocessor>"
  # 18 "<preprocessor>"
  hmpprt::s32 end_11;
  # 18 "<preprocessor>"
  hmpprt::s32 j_4;
  # 151 "y_solve.c"
  # 151 "y_solve.c"
  # 151 "y_solve.c"
  for (j_4 = (hmpprt::gr_btidy()), end_11 = gp1_1 - 3 ; j_4 <= end_11 ; j_4 = j_4 + (hmpprt::gr_btnumy()))
  {
   # 19 "<preprocessor>"
   hmpprt::s32 end_10;
   # 19 "<preprocessor>"
   hmpprt::s32 i_7;
   # 152 "y_solve.c"
   # 152 "y_solve.c"
   # 152 "y_solve.c"
   for (i_7 = (hmpprt::gr_btidx()), end_10 = gp0_1 - 3 ; i_7 <= end_10 ; i_7 = i_7 + (hmpprt::gr_btnumx()))
   {
    }
   # 18 "<preprocessor>"
  }
  # 18 "<preprocessor>"
  # 18 "<preprocessor>"
  hmpprt::s32 end_16;
  # 18 "<preprocessor>"
  hmpprt::s32 j_5;
  # 170 "y_solve.c"
  # 170 "y_solve.c"
  # 170 "y_solve.c"
  for (j_5 = (hmpprt::gr_btidy()), end_16 = gp1_1 - 3 ; j_5 <= end_16 ; j_5 = j_5 + (hmpprt::gr_btnumy()))
  {
   # 19 "<preprocessor>"
   hmpprt::s32 end_15;
   # 19 "<preprocessor>"
   hmpprt::s32 i_8;
   # 173 "y_solve.c"
   # 173 "y_solve.c"
   # 173 "y_solve.c"
   for (i_8 = (hmpprt::gr_btidx()), end_15 = gp0_1 - 3 ; i_8 <= end_15 ; i_8 = i_8 + (hmpprt::gr_btnumx()))
   {
    # 14 "<preprocessor>"
    hmpprt::s32 m_2;
    # 177 "y_solve.c"
    # 177 "y_solve.c"
    for (m_2 = 0 ; m_2 <= 2 ; m_2 = m_2 + 1)
    {
     }
    # 14 "<preprocessor>"
    # 14 "<preprocessor>"
    hmpprt::s32 m_3;
    # 182 "y_solve.c"
    # 182 "y_solve.c"
    for (m_3 = 0 ; m_3 <= 2 ; m_3 = m_3 + 1)
    {
     }
    # 14 "<preprocessor>"
    # 14 "<preprocessor>"
    hmpprt::s32 m_4;
    # 187 "y_solve.c"
    # 187 "y_solve.c"
    for (m_4 = 0 ; m_4 <= 2 ; m_4 = m_4 + 1)
    {
     }
    # 19 "<preprocessor>"
   }
   # 19 "<preprocessor>"
  }
  # 19 "<preprocessor>"
  # 19 "<preprocessor>"
  hmpprt::s32 end_20;
  # 19 "<preprocessor>"
  hmpprt::s32 i_9;
  # 200 "y_solve.c"
  # 200 "y_solve.c"
  # 200 "y_solve.c"
  for (i_9 = (hmpprt::gr_btidy()), end_20 = gp0_1 - 3 ; i_9 <= end_20 ; i_9 = i_9 + (hmpprt::gr_btnumy()))
  {
   # 14 "<preprocessor>"
   hmpprt::s32 m_5;
   # 204 "y_solve.c"
   # 204 "y_solve.c"
   for (m_5 = (hmpprt::gr_btidx()) ; m_5 <= 2 ; m_5 = m_5 + (hmpprt::gr_btnumx()))
   {
    }
   # 14 "<preprocessor>"
   # 14 "<preprocessor>"
   hmpprt::s32 m_6;
   # 209 "y_solve.c"
   # 209 "y_solve.c"
   for (m_6 = (hmpprt::gr_btidx()) ; m_6 <= 2 ; m_6 = m_6 + (hmpprt::gr_btnumx()))
   {
    }
   # 14 "<preprocessor>"
   # 14 "<preprocessor>"
   hmpprt::s32 m_7;
   # 216 "y_solve.c"
   # 216 "y_solve.c"
   for (m_7 = (hmpprt::gr_btidx()) ; m_7 <= 2 ; m_7 = m_7 + (hmpprt::gr_btnumx()))
   {
    }
   # 18 "<preprocessor>"
  }
  # 18 "<preprocessor>"
  # 18 "<preprocessor>"
  hmpprt::s32 end_22;
  # 18 "<preprocessor>"
  hmpprt::s32 j_6;
  # 224 "y_solve.c"
  # 224 "y_solve.c"
  # 224 "y_solve.c"
  for (j_6 = (hmpprt::gr_btidy()), end_22 = gp1_1 - 3 ; j_6 <= end_22 ; j_6 = j_6 + (hmpprt::gr_btnumy()))
  {
   # 19 "<preprocessor>"
   hmpprt::s32 end_21;
   # 19 "<preprocessor>"
   hmpprt::s32 i_10;
   # 227 "y_solve.c"
   # 227 "y_solve.c"
   # 227 "y_solve.c"
   for (i_10 = (hmpprt::gr_btidx()), end_21 = gp0_1 - 3 ; i_10 <= end_21 ; i_10 = i_10 + (hmpprt::gr_btnumx()))
   {
    }
   # 19 "<preprocessor>"
  }
  # 19 "<preprocessor>"
  # 19 "<preprocessor>"
  hmpprt::s32 end_23;
  # 19 "<preprocessor>"
  hmpprt::s32 i_11;
  # 259 "y_solve.c"
  # 259 "y_solve.c"
  # 259 "y_solve.c"
  for (i_11 = (hmpprt::gr_btidy()), end_23 = gp0_1 - 3 ; i_11 <= end_23 ; i_11 = i_11 + (hmpprt::gr_btnumy()))
  {
   }
  # 19 "<preprocessor>"
  # 19 "<preprocessor>"
  hmpprt::s32 end_25;
  # 19 "<preprocessor>"
  hmpprt::s32 i_12;
  # 291 "y_solve.c"
  # 291 "y_solve.c"
  # 291 "y_solve.c"
  for (i_12 = (hmpprt::gr_btidy()), end_25 = gp0_1 - 3 ; i_12 <= end_25 ; i_12 = i_12 + (hmpprt::gr_btnumy()))
  {
   # 14 "<preprocessor>"
   hmpprt::s32 m_8;
   # 292 "y_solve.c"
   # 292 "y_solve.c"
   for (m_8 = (hmpprt::gr_btidx()) ; m_8 <= 2 ; m_8 = m_8 + (hmpprt::gr_btnumx()))
   {
    }
   # 18 "<preprocessor>"
  }
  # 18 "<preprocessor>"
  # 18 "<preprocessor>"
  hmpprt::s32 end_28;
  # 18 "<preprocessor>"
  hmpprt::s32 j_7;
  # 303 "y_solve.c"
  # 303 "y_solve.c"
  # 303 "y_solve.c"
  for (j_7 = (hmpprt::gr_btidy()), end_28 = gp1_1 - 3 ; j_7 <= end_28 ; j_7 = j_7 + (hmpprt::gr_btnumy()))
  {
   # 19 "<preprocessor>"
   hmpprt::s32 end_27;
   # 19 "<preprocessor>"
   hmpprt::s32 i_13;
   # 306 "y_solve.c"
   # 306 "y_solve.c"
   # 306 "y_solve.c"
   for (i_13 = (hmpprt::gr_btidx()), end_27 = gp0_1 - 3 ; i_13 <= end_27 ; i_13 = i_13 + (hmpprt::gr_btnumx()))
   {
    # 14 "<preprocessor>"
    hmpprt::s32 m_9;
    # 307 "y_solve.c"
    # 307 "y_solve.c"
    for (m_9 = 0 ; m_9 <= 2 ; m_9 = m_9 + 1)
    {
     }
    # 5 "<preprocessor>"
   }
   # 5 "<preprocessor>"
  }
  # 5 "<preprocessor>"
 }
 # 5 "<preprocessor>"
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__y_solve_64__mhml6p0i_internal_1(hmpprt::s32 ni, hmpprt::s32 nj, hmpprt::s32 gp0, hmpprt::s32 gp1_2, hmpprt::s32 gp2)
{
 # 5 "<preprocessor>"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(8LL);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (gp0), "gp0_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (gp1_2), "gp1_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (gp2), "gp2_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (ni), "ni_1");
  __hmppcg_call.launch(__hmpp_acc_region__y_solve_64__mhml6p0i_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__y_solve_64__mhml6p0i(hmpprt::s32 ni_2, hmpprt::s32 nj_1, hmpprt::s32 gp0_2, hmpprt::s32 gp1, hmpprt::s32 gp2_2)
{
 # 1 "<preprocessor>"
 (__hmpp_acc_region__y_solve_64__mhml6p0i_internal_1(ni_2, nj_1, gp0_2, gp1, gp2_2));
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      __hmpp_acc_region__y_solve_64__mhml6p0i_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "__hmpp_acc_region__y_solve_64__mhml6p0i_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("__hmpp_acc_region__y_solve_64__mhml6p0i", "prototype __hmpp_acc_region__y_solve_64__mhml6p0i(ni: s32, nj: s32, gp0: s32, gp1: s32, gp2: s32)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete __hmpp_acc_region__y_solve_64__mhml6p0i_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
