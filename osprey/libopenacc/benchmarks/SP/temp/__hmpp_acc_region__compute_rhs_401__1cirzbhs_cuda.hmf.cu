#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__compute_rhs_401__1cirzbhs(hmpprt::s32 nx2_1, hmpprt::s32 ny2, hmpprt::s32 nz2_1, double dssp_1, double* u_2, double* rhs_2, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j, hmpprt::s32* __hmpp_addr__k, hmpprt::s32* __hmpp_addr__m)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__compute_rhs_401__1cirzbhs_internal_1(hmpprt::s32 nx2_2, hmpprt::s32 ny2_2, hmpprt::s32 nz2_2, double dssp_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  u_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs_1, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32* __hmpp_addr__k_1, hmpprt::s32* __hmpp_addr__m_1)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__compute_rhs_401__1cirzbhs_parallel_region_1 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__compute_rhs_401__1cirzbhs_parallel_region_1(double dssp, hmpprt::s32* m, hmpprt::s32 nx2, hmpprt::s32 ny2_1, hmpprt::s32 nz2, double* rhs, double* u);
#endif // __HIPCC__




# 5 "<preprocessor>"

#ifdef __HIPCC__

extern "C" __global__ void __hmpp_acc_region__compute_rhs_401__1cirzbhs_parallel_region_1(double dssp, hmpprt::s32* m, hmpprt::s32 nx2, hmpprt::s32 ny2_1, hmpprt::s32 nz2, double* rhs, double* u)
{
 # 8 "<preprocessor>"
 hmpprt::s32 iter_per_gang_1;
 # 8 "<preprocessor>"
 hmpprt::s32 first_gang_iter_1;
 # 8 "<preprocessor>"
 hmpprt::s32 last_gang_iter_1;
 # 8 "<preprocessor>"
 iter_per_gang_1 = 1 + (nz2 - 5) / 192;
 # 8 "<preprocessor>"
 first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
 # 8 "<preprocessor>"
 last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < (nz2 - 5) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : (nz2 - 5));
 # 8 "<preprocessor>"
 hmpprt::s32 k_1;
 # 402 "rhs.c"
 # 402 "rhs.c"
 for (k_1 = first_gang_iter_1 ; k_1 <= last_gang_iter_1 ; k_1 = k_1 + 1)
 {
  # 9 "<preprocessor>"
  hmpprt::s32 end_3;
  # 9 "<preprocessor>"
  hmpprt::s32 j_1;
  # 404 "rhs.c"
  # 404 "rhs.c"
  # 404 "rhs.c"
  for (j_1 = (hmpprt::gr_btidx()), end_3 = ny2_1 - 1 ; j_1 <= end_3 ; j_1 = j_1 + (hmpprt::gr_btnumx()))
  {
   # 10 "<preprocessor>"
   hmpprt::s32 end_2;
   # 10 "<preprocessor>"
   hmpprt::s32 i_2;
   # 406 "rhs.c"
   # 406 "rhs.c"
   # 406 "rhs.c"
   for (i_2 = 0, end_2 = nx2 - 1 ; i_2 <= end_2 ; i_2 = i_2 + 1)
   {
    # 7 "<preprocessor>"
    hmpprt::s32 m_1;
    # 407 "rhs.c"
    # 407 "rhs.c"
    for (m_1 = 0 ; m_1 <= 4 ; m_1 = m_1 + 1)
    {
     # 411 "rhs.c"
     *(rhs + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 3 + 36uLL * m_1))) = *(rhs + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 3 + 36uLL * m_1))) - dssp * (*(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1 + 36uLL * m_1))) - (double) 4.0 * *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 2 + 36uLL * m_1))) + (double) 6.0 * *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 3 + 36uLL * m_1))) - (double) 4.0 * *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 4 + 36uLL * m_1))) + *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 5 + 36uLL * m_1))));
    }
    # 5 "<preprocessor>"
    # 5 "<preprocessor>"
    *m = 5;
   }
   # 5 "<preprocessor>"
  }
  # 5 "<preprocessor>"
 }
 # 5 "<preprocessor>"
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__compute_rhs_401__1cirzbhs_internal_1(hmpprt::s32 nx2_2, hmpprt::s32 ny2_2, hmpprt::s32 nz2_2, double dssp_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  u_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs_1, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32* __hmpp_addr__k_1, hmpprt::s32* __hmpp_addr__m_1)
{
 # 7 "<preprocessor>"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  m_2;
 # 7 "<preprocessor>"
 hmpprt::Context::getInstance()->allocate((void **) (&m_2), hmpprt::MS_CUDA_GLOB, 4);
 # 7 "<preprocessor>"
 *m_2 = *__hmpp_addr__m_1;
 # 8 "<preprocessor>"
 hmpprt::s32 k;
 # 9 "<preprocessor>"
 hmpprt::s32 j;
 # 9 "<preprocessor>"
 j = *__hmpp_addr__j_1;
 # 10 "<preprocessor>"
 hmpprt::s32 i_1;
 # 10 "<preprocessor>"
 i_1 = *__hmpp_addr__i;
 # 36 "<preprocessor>"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(1);
  __hmppcg_call.addLocalParameter(&dssp_2, 8, "dssp");
  __hmppcg_call.addLocalParameter(&m_2, 8, "m");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nx2_2), "nx2");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (ny2_2), "ny2_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nz2_2), "nz2");
  __hmppcg_call.addLocalParameter(&rhs_1, 8, "rhs");
  __hmppcg_call.addLocalParameter(&u_1, 8, "u");
  __hmppcg_call.launch(__hmpp_acc_region__compute_rhs_401__1cirzbhs_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 36 "<preprocessor>"
 *__hmpp_addr__i = i_1;
 # 37 "<preprocessor>"
 *__hmpp_addr__j_1 = j;
 # 38 "<preprocessor>"
 *__hmpp_addr__k_1 = k;
 # 39 "<preprocessor>"
 *__hmpp_addr__m_1 = *m_2;
 # 5 "<preprocessor>"
 hmpprt::Context::getInstance()->free((void **) (&m_2));
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__compute_rhs_401__1cirzbhs(hmpprt::s32 nx2_1, hmpprt::s32 ny2, hmpprt::s32 nz2_1, double dssp_1, double* u_2, double* rhs_2, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j, hmpprt::s32* __hmpp_addr__k, hmpprt::s32* __hmpp_addr__m)
{
 # 1 "<preprocessor>"
 (__hmpp_acc_region__compute_rhs_401__1cirzbhs_internal_1(nx2_1, ny2, nz2_1, dssp_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (u_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (rhs_2), __hmpp_addr__i_1, __hmpp_addr__j, __hmpp_addr__k, __hmpp_addr__m));
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      __hmpp_acc_region__compute_rhs_401__1cirzbhs_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "__hmpp_acc_region__compute_rhs_401__1cirzbhs_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("__hmpp_acc_region__compute_rhs_401__1cirzbhs", "prototype __hmpp_acc_region__compute_rhs_401__1cirzbhs(nx2: s32, ny2: s32, nz2: s32, dssp: double, u: ^cudaglob double, rhs: ^cudaglob double, __hmpp_addr__i: ^host s32, __hmpp_addr__j: ^host s32, __hmpp_addr__k: ^host s32, __hmpp_addr__m: ^host s32)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete __hmpp_acc_region__compute_rhs_401__1cirzbhs_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
