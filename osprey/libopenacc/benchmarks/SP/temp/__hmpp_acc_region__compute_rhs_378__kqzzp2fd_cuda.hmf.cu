#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__compute_rhs_378__kqzzp2fd(hmpprt::s32 nx2_2, hmpprt::s32 ny2_2, double dssp_2, double* u_2, double* rhs_1, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j, hmpprt::s32 k_2, hmpprt::s32* __hmpp_addr__m)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__compute_rhs_378__kqzzp2fd_internal_1(hmpprt::s32 nx2_1, hmpprt::s32 ny2_1, double dssp, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  u_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs_2, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32 k, hmpprt::s32* __hmpp_addr__m_1)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__compute_rhs_378__kqzzp2fd_parallel_region_1 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__compute_rhs_378__kqzzp2fd_parallel_region_1(double dssp_1, hmpprt::s32* i_1, hmpprt::s32 k_1, hmpprt::s32* m, hmpprt::s32 nx2, hmpprt::s32 ny2, double* rhs, double* u);
#endif // __HIPCC__




# 5 "<preprocessor>"

#ifdef __HIPCC__

extern "C" __global__ void __hmpp_acc_region__compute_rhs_378__kqzzp2fd_parallel_region_1(double dssp_1, hmpprt::s32* i_1, hmpprt::s32 k_1, hmpprt::s32* m, hmpprt::s32 nx2, hmpprt::s32 ny2, double* rhs, double* u)
{
 # 8 "<preprocessor>"
 hmpprt::s32 iter_per_gang_1;
 # 8 "<preprocessor>"
 hmpprt::s32 first_gang_iter_1;
 # 8 "<preprocessor>"
 hmpprt::s32 last_gang_iter_1;
 # 8 "<preprocessor>"
 iter_per_gang_1 = ((1 + (ny2 - 1) / 192) > 8LL ? (1 + (ny2 - 1) / 192) : 8LL);
 # 8 "<preprocessor>"
 first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
 # 8 "<preprocessor>"
 last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < (ny2 - 1) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : (ny2 - 1));
 # 8 "<preprocessor>"
 hmpprt::s32 j_1;
 # 379 "rhs.c"
 # 379 "rhs.c"
 for (j_1 = first_gang_iter_1 + (hmpprt::gr_btidy()) ; j_1 <= last_gang_iter_1 ; j_1 = j_1 + (hmpprt::gr_btnumy()))
 {
  # 5 "<preprocessor>"
  if (1 <= nx2)
  {
   # 9 "<preprocessor>"
   hmpprt::s32 end_2;
   # 9 "<preprocessor>"
   hmpprt::s32 i_2;
   # 380 "rhs.c"
   # 380 "rhs.c"
   # 380 "rhs.c"
   for (i_2 = 0, end_2 = nx2 - 1 ; i_2 <= end_2 ; i_2 = i_2 + 1)
   {
    # 7 "<preprocessor>"
    hmpprt::s32 m_1;
    # 381 "rhs.c"
    # 381 "rhs.c"
    for (m_1 = (hmpprt::gr_btidx()) ; m_1 <= 4 ; m_1 = m_1 + (hmpprt::gr_btnumx()))
    {
     # 383 "rhs.c"
     *(rhs + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 36uLL * m_1))) = *(rhs + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 36uLL * m_1))) - dssp_1 * ((double) 5.0 * *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 36uLL * m_1))) - (double) 4.0 * *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1 + 36uLL * m_1))) + *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 2 + 36uLL * m_1))));
    }
    # 5 "<preprocessor>"
    # 5 "<preprocessor>"
    *m = 5;
   }
   # 5 "<preprocessor>"
   # 5 "<preprocessor>"
   *i_1 = nx2 + 1;
  }
  else
  {
   # 5 "<preprocessor>"
   *i_1 = 1;
  }
 }
 # 5 "<preprocessor>"
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__compute_rhs_378__kqzzp2fd_internal_1(hmpprt::s32 nx2_1, hmpprt::s32 ny2_1, double dssp, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  u_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs_2, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32 k, hmpprt::s32* __hmpp_addr__m_1)
{
 # 7 "<preprocessor>"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  m_2;
 # 7 "<preprocessor>"
 hmpprt::Context::getInstance()->allocate((void **) (&m_2), hmpprt::MS_CUDA_GLOB, 4);
 # 7 "<preprocessor>"
 *m_2 = *__hmpp_addr__m_1;
 # 8 "<preprocessor>"
 hmpprt::s32 j;
 # 9 "<preprocessor>"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  i_3;
 # 9 "<preprocessor>"
 hmpprt::Context::getInstance()->allocate((void **) (&i_3), hmpprt::MS_CUDA_GLOB, 4);
 # 9 "<preprocessor>"
 *i_3 = *__hmpp_addr__i_1;
 # 23 "<preprocessor>"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(8LL);
  __hmppcg_call.addLocalParameter(&dssp, 8, "dssp_1");
  __hmppcg_call.addLocalParameter(&i_3, 8, "i_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (k), "k_1");
  __hmppcg_call.addLocalParameter(&m_2, 8, "m");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nx2_1), "nx2");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (ny2_1), "ny2");
  __hmppcg_call.addLocalParameter(&rhs_2, 8, "rhs");
  __hmppcg_call.addLocalParameter(&u_1, 8, "u");
  __hmppcg_call.launch(__hmpp_acc_region__compute_rhs_378__kqzzp2fd_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 23 "<preprocessor>"
 *__hmpp_addr__i_1 = *i_3;
 # 24 "<preprocessor>"
 *__hmpp_addr__j_1 = j;
 # 25 "<preprocessor>"
 *__hmpp_addr__m_1 = *m_2;
 # 5 "<preprocessor>"
 hmpprt::Context::getInstance()->free((void **) (&m_2));
 # 5 "<preprocessor>"
 hmpprt::Context::getInstance()->free((void **) (&i_3));
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__compute_rhs_378__kqzzp2fd(hmpprt::s32 nx2_2, hmpprt::s32 ny2_2, double dssp_2, double* u_2, double* rhs_1, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j, hmpprt::s32 k_2, hmpprt::s32* __hmpp_addr__m)
{
 # 1 "<preprocessor>"
 (__hmpp_acc_region__compute_rhs_378__kqzzp2fd_internal_1(nx2_2, ny2_2, dssp_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (u_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (rhs_1), __hmpp_addr__i, __hmpp_addr__j, k_2, __hmpp_addr__m));
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      __hmpp_acc_region__compute_rhs_378__kqzzp2fd_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "__hmpp_acc_region__compute_rhs_378__kqzzp2fd_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("__hmpp_acc_region__compute_rhs_378__kqzzp2fd", "prototype __hmpp_acc_region__compute_rhs_378__kqzzp2fd(nx2: s32, ny2: s32, dssp: double, u: ^cudaglob double, rhs: ^cudaglob double, __hmpp_addr__i: ^host s32, __hmpp_addr__j: ^host s32, k: s32, __hmpp_addr__m: ^host s32)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete __hmpp_acc_region__compute_rhs_378__kqzzp2fd_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
