#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__z_solve_65__uxet4axu(hmpprt::s32 nx2_4, hmpprt::s32 nz2_3, hmpprt::s32 nj_1, hmpprt::s32 ni_2, hmpprt::s32 gp2_2)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__z_solve_65__uxet4axu_internal_1(hmpprt::s32 nx2, hmpprt::s32 nz2, hmpprt::s32 nj, hmpprt::s32 ni, hmpprt::s32 gp2)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_4 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_4(hmpprt::s32 nx2_1, hmpprt::s32 ny2_3, hmpprt::s32 nz2_1);
#endif // __HIPCC__




# 5 "<preprocessor>"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_3 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_3(hmpprt::s32 gp2_1, hmpprt::s32 nx2_2, hmpprt::s32 ny2_2);
#endif // __HIPCC__




# 5 "<preprocessor>"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_2 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_2(hmpprt::s32 ni_1, hmpprt::s32 ny2_1);
#endif // __HIPCC__




# 5 "<preprocessor>"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_1 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_1(hmpprt::s32 nx2_3, hmpprt::s32 ny2_4, hmpprt::s32 nz2_2);
#endif // __HIPCC__




# 5 "<preprocessor>"

#ifdef __HIPCC__

extern "C" __global__ void __hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_1(hmpprt::s32 nx2_3, hmpprt::s32 ny2_4, hmpprt::s32 nz2_2)
{
 # 18 "<preprocessor>"
 hmpprt::s32 iter_per_gang_1;
 # 18 "<preprocessor>"
 hmpprt::s32 first_gang_iter_1;
 # 18 "<preprocessor>"
 hmpprt::s32 last_gang_iter_1;
 # 18 "<preprocessor>"
 iter_per_gang_1 = ((1 + (ny2_4 - 1) / 192) > 8LL ? (1 + (ny2_4 - 1) / 192) : 8LL);
 # 18 "<preprocessor>"
 first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
 # 18 "<preprocessor>"
 last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < (ny2_4 - 1) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : (ny2_4 - 1));
 # 18 "<preprocessor>"
 hmpprt::s32 j_3;
 # 116 "z_solve.c"
 # 116 "z_solve.c"
 for (j_3 = first_gang_iter_1 + (hmpprt::gr_btidy()) ; j_3 <= last_gang_iter_1 ; j_3 = j_3 + (hmpprt::gr_btnumy()))
 {
  # 19 "<preprocessor>"
  hmpprt::s32 end_1;
  # 19 "<preprocessor>"
  hmpprt::s32 i_4;
  # 120 "z_solve.c"
  # 120 "z_solve.c"
  # 120 "z_solve.c"
  for (i_4 = (hmpprt::gr_btidx()), end_1 = nx2_3 - 1 ; i_4 <= end_1 ; i_4 = i_4 + (hmpprt::gr_btnumx()))
  {
   }
  # 17 "<preprocessor>"
  # 17 "<preprocessor>"
  hmpprt::s32 end_3;
  # 17 "<preprocessor>"
  hmpprt::s32 k_3;
  # 133 "z_solve.c"
  # 133 "z_solve.c"
  # 133 "z_solve.c"
  for (k_3 = (hmpprt::gr_btidx()), end_3 = nz2_2 - 5 ; k_3 <= end_3 ; k_3 = k_3 + (hmpprt::gr_btnumx()))
  {
   # 19 "<preprocessor>"
   hmpprt::s32 end_2;
   # 19 "<preprocessor>"
   hmpprt::s32 i_5;
   # 134 "z_solve.c"
   # 134 "z_solve.c"
   # 134 "z_solve.c"
   for (i_5 = 0, end_2 = nx2_3 - 1 ; i_5 <= end_2 ; i_5 = i_5 + 1)
   {
    }
   # 19 "<preprocessor>"
  }
  # 19 "<preprocessor>"
  # 19 "<preprocessor>"
  hmpprt::s32 end_4;
  # 19 "<preprocessor>"
  hmpprt::s32 i_6;
  # 143 "z_solve.c"
  # 143 "z_solve.c"
  # 143 "z_solve.c"
  for (i_6 = (hmpprt::gr_btidx()), end_4 = nx2_3 - 1 ; i_6 <= end_4 ; i_6 = i_6 + (hmpprt::gr_btnumx()))
  {
   }
  # 17 "<preprocessor>"
  # 17 "<preprocessor>"
  hmpprt::s32 end_6;
  # 17 "<preprocessor>"
  hmpprt::s32 k_4;
  # 159 "z_solve.c"
  # 159 "z_solve.c"
  # 159 "z_solve.c"
  for (k_4 = (hmpprt::gr_btidx()), end_6 = nz2_2 - 1 ; k_4 <= end_6 ; k_4 = k_4 + (hmpprt::gr_btnumx()))
  {
   # 19 "<preprocessor>"
   hmpprt::s32 end_5;
   # 19 "<preprocessor>"
   hmpprt::s32 i_7;
   # 160 "z_solve.c"
   # 160 "z_solve.c"
   # 160 "z_solve.c"
   for (i_7 = 0, end_5 = nx2_3 - 1 ; i_7 <= end_5 ; i_7 = i_7 + 1)
   {
    }
   # 5 "<preprocessor>"
  }
  # 5 "<preprocessor>"
 }
 # 5 "<preprocessor>"
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifdef __HIPCC__

extern "C" __global__ void __hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_2(hmpprt::s32 ni_1, hmpprt::s32 ny2_1)
{
 # 18 "<preprocessor>"
 hmpprt::s32 iter_per_gang_2;
 # 18 "<preprocessor>"
 hmpprt::s32 first_gang_iter_2;
 # 18 "<preprocessor>"
 hmpprt::s32 last_gang_iter_2;
 # 18 "<preprocessor>"
 iter_per_gang_2 = 1 + (ny2_1 - 1) / 192;
 # 18 "<preprocessor>"
 first_gang_iter_2 = (hmpprt::gr_gbidx()) * iter_per_gang_2;
 # 18 "<preprocessor>"
 last_gang_iter_2 = ((first_gang_iter_2 + iter_per_gang_2 - 1) < (ny2_1 - 1) ? (first_gang_iter_2 + iter_per_gang_2 - 1) : (ny2_1 - 1));
 # 18 "<preprocessor>"
 hmpprt::s32 j_1;
 # 67 "z_solve.c"
 # 67 "z_solve.c"
 for (j_1 = first_gang_iter_2 ; j_1 <= last_gang_iter_2 ; j_1 = j_1 + 1)
 {
  # 19 "<preprocessor>"
  hmpprt::s32 end_9;
  # 19 "<preprocessor>"
  hmpprt::s32 i_2;
  # 69 "z_solve.c"
  # 69 "z_solve.c"
  # 69 "z_solve.c"
  for (i_2 = (hmpprt::gr_btidy()), end_9 = ni_1 - 1 ; i_2 <= end_9 ; i_2 = i_2 + (hmpprt::gr_btnumy()))
  {
   # 14 "<preprocessor>"
   hmpprt::s32 m_1;
   # 70 "z_solve.c"
   # 70 "z_solve.c"
   for (m_1 = (hmpprt::gr_btidx()) ; m_1 <= 4 ; m_1 = m_1 + (hmpprt::gr_btnumx()))
   {
    }
   # 5 "<preprocessor>"
  }
  # 5 "<preprocessor>"
 }
 # 5 "<preprocessor>"
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifdef __HIPCC__

extern "C" __global__ void __hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_3(hmpprt::s32 gp2_1, hmpprt::s32 nx2_2, hmpprt::s32 ny2_2)
{
 # 18 "<preprocessor>"
 hmpprt::s32 iter_per_gang_3;
 # 18 "<preprocessor>"
 hmpprt::s32 first_gang_iter_3;
 # 18 "<preprocessor>"
 hmpprt::s32 last_gang_iter_3;
 # 18 "<preprocessor>"
 iter_per_gang_3 = ((1 + (ny2_2 - 1) / 192) > 8LL ? (1 + (ny2_2 - 1) / 192) : 8LL);
 # 18 "<preprocessor>"
 first_gang_iter_3 = (hmpprt::gr_gbidx()) * iter_per_gang_3;
 # 18 "<preprocessor>"
 last_gang_iter_3 = ((first_gang_iter_3 + iter_per_gang_3 - 1) < (ny2_2 - 1) ? (first_gang_iter_3 + iter_per_gang_3 - 1) : (ny2_2 - 1));
 # 18 "<preprocessor>"
 hmpprt::s32 j_4;
 # 177 "z_solve.c"
 # 177 "z_solve.c"
 for (j_4 = first_gang_iter_3 + (hmpprt::gr_btidy()) ; j_4 <= last_gang_iter_3 ; j_4 = j_4 + (hmpprt::gr_btnumy()))
 {
  # 17 "<preprocessor>"
  hmpprt::s32 end_15;
  # 17 "<preprocessor>"
  hmpprt::s32 k_5;
  # 181 "z_solve.c"
  # 181 "z_solve.c"
  # 181 "z_solve.c"
  for (k_5 = (hmpprt::gr_btidx()), end_15 = gp2_1 - 3 ; k_5 <= end_15 ; k_5 = k_5 + (hmpprt::gr_btnumx()))
  {
   # 19 "<preprocessor>"
   hmpprt::s32 end_14;
   # 19 "<preprocessor>"
   hmpprt::s32 i_8;
   # 184 "z_solve.c"
   # 184 "z_solve.c"
   # 184 "z_solve.c"
   for (i_8 = 0, end_14 = nx2_2 - 1 ; i_8 <= end_14 ; i_8 = i_8 + 1)
   {
    # 14 "<preprocessor>"
    hmpprt::s32 m_2;
    # 188 "z_solve.c"
    # 188 "z_solve.c"
    for (m_2 = 0 ; m_2 <= 2 ; m_2 = m_2 + 1)
    {
     }
    # 14 "<preprocessor>"
    # 14 "<preprocessor>"
    hmpprt::s32 m_3;
    # 193 "z_solve.c"
    # 193 "z_solve.c"
    for (m_3 = 0 ; m_3 <= 2 ; m_3 = m_3 + 1)
    {
     }
    # 14 "<preprocessor>"
    # 14 "<preprocessor>"
    hmpprt::s32 m_4;
    # 198 "z_solve.c"
    # 198 "z_solve.c"
    for (m_4 = 0 ; m_4 <= 2 ; m_4 = m_4 + 1)
    {
     }
    # 19 "<preprocessor>"
   }
   # 19 "<preprocessor>"
  }
  # 19 "<preprocessor>"
  # 19 "<preprocessor>"
  hmpprt::s32 end_19;
  # 19 "<preprocessor>"
  hmpprt::s32 i_9;
  # 211 "z_solve.c"
  # 211 "z_solve.c"
  # 211 "z_solve.c"
  for (i_9 = (hmpprt::gr_btidx()), end_19 = nx2_2 - 1 ; i_9 <= end_19 ; i_9 = i_9 + (hmpprt::gr_btnumx()))
  {
   # 14 "<preprocessor>"
   hmpprt::s32 m_5;
   # 215 "z_solve.c"
   # 215 "z_solve.c"
   for (m_5 = 0 ; m_5 <= 2 ; m_5 = m_5 + 1)
   {
    }
   # 14 "<preprocessor>"
   # 14 "<preprocessor>"
   hmpprt::s32 m_6;
   # 220 "z_solve.c"
   # 220 "z_solve.c"
   for (m_6 = 0 ; m_6 <= 2 ; m_6 = m_6 + 1)
   {
    }
   # 14 "<preprocessor>"
   # 14 "<preprocessor>"
   hmpprt::s32 m_7;
   # 228 "z_solve.c"
   # 228 "z_solve.c"
   for (m_7 = 0 ; m_7 <= 2 ; m_7 = m_7 + 1)
   {
    }
   # 17 "<preprocessor>"
  }
  # 17 "<preprocessor>"
  # 17 "<preprocessor>"
  hmpprt::s32 end_21;
  # 17 "<preprocessor>"
  hmpprt::s32 k_6;
  # 236 "z_solve.c"
  # 236 "z_solve.c"
  # 236 "z_solve.c"
  for (k_6 = (hmpprt::gr_btidx()), end_21 = gp2_1 - 3 ; k_6 <= end_21 ; k_6 = k_6 + (hmpprt::gr_btnumx()))
  {
   # 19 "<preprocessor>"
   hmpprt::s32 end_20;
   # 19 "<preprocessor>"
   hmpprt::s32 i_10;
   # 239 "z_solve.c"
   # 239 "z_solve.c"
   # 239 "z_solve.c"
   for (i_10 = 0, end_20 = nx2_2 - 1 ; i_10 <= end_20 ; i_10 = i_10 + 1)
   {
    }
   # 19 "<preprocessor>"
  }
  # 19 "<preprocessor>"
  # 19 "<preprocessor>"
  hmpprt::s32 end_22;
  # 19 "<preprocessor>"
  hmpprt::s32 i_11;
  # 271 "z_solve.c"
  # 271 "z_solve.c"
  # 271 "z_solve.c"
  for (i_11 = (hmpprt::gr_btidx()), end_22 = nx2_2 - 1 ; i_11 <= end_22 ; i_11 = i_11 + (hmpprt::gr_btnumx()))
  {
   }
  # 19 "<preprocessor>"
  # 19 "<preprocessor>"
  hmpprt::s32 end_24;
  # 19 "<preprocessor>"
  hmpprt::s32 i_12;
  # 304 "z_solve.c"
  # 304 "z_solve.c"
  # 304 "z_solve.c"
  for (i_12 = (hmpprt::gr_btidx()), end_24 = nx2_2 - 1 ; i_12 <= end_24 ; i_12 = i_12 + (hmpprt::gr_btnumx()))
  {
   # 14 "<preprocessor>"
   hmpprt::s32 m_8;
   # 305 "z_solve.c"
   # 305 "z_solve.c"
   for (m_8 = 0 ; m_8 <= 2 ; m_8 = m_8 + 1)
   {
    }
   # 17 "<preprocessor>"
  }
  # 17 "<preprocessor>"
  # 17 "<preprocessor>"
  hmpprt::s32 end_27;
  # 17 "<preprocessor>"
  hmpprt::s32 k_7;
  # 321 "z_solve.c"
  # 321 "z_solve.c"
  # 321 "z_solve.c"
  for (k_7 = (hmpprt::gr_btidx()), end_27 = gp2_1 - 3 ; k_7 <= end_27 ; k_7 = k_7 + (hmpprt::gr_btnumx()))
  {
   # 19 "<preprocessor>"
   hmpprt::s32 end_26;
   # 19 "<preprocessor>"
   hmpprt::s32 i_13;
   # 324 "z_solve.c"
   # 324 "z_solve.c"
   # 324 "z_solve.c"
   for (i_13 = 0, end_26 = nx2_2 - 1 ; i_13 <= end_26 ; i_13 = i_13 + 1)
   {
    # 14 "<preprocessor>"
    hmpprt::s32 m_9;
    # 325 "z_solve.c"
    # 325 "z_solve.c"
    for (m_9 = 0 ; m_9 <= 2 ; m_9 = m_9 + 1)
    {
     }
    # 5 "<preprocessor>"
   }
   # 5 "<preprocessor>"
  }
  # 5 "<preprocessor>"
 }
 # 5 "<preprocessor>"
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifdef __HIPCC__

extern "C" __global__ void __hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_4(hmpprt::s32 nx2_1, hmpprt::s32 ny2_3, hmpprt::s32 nz2_1)
{
 # 18 "<preprocessor>"
 hmpprt::s32 iter_per_gang_4;
 # 18 "<preprocessor>"
 hmpprt::s32 first_gang_iter_4;
 # 18 "<preprocessor>"
 hmpprt::s32 last_gang_iter_4;
 # 18 "<preprocessor>"
 iter_per_gang_4 = ((1 + (ny2_3 - 1) / 192) > 8LL ? (1 + (ny2_3 - 1) / 192) : 8LL);
 # 18 "<preprocessor>"
 first_gang_iter_4 = (hmpprt::gr_gbidx()) * iter_per_gang_4;
 # 18 "<preprocessor>"
 last_gang_iter_4 = ((first_gang_iter_4 + iter_per_gang_4 - 1) < (ny2_3 - 1) ? (first_gang_iter_4 + iter_per_gang_4 - 1) : (ny2_3 - 1));
 # 18 "<preprocessor>"
 hmpprt::s32 j_2;
 # 87 "z_solve.c"
 # 87 "z_solve.c"
 for (j_2 = first_gang_iter_4 + (hmpprt::gr_btidy()) ; j_2 <= last_gang_iter_4 ; j_2 = j_2 + (hmpprt::gr_btnumy()))
 {
  # 19 "<preprocessor>"
  hmpprt::s32 end_31;
  # 19 "<preprocessor>"
  hmpprt::s32 i_3;
  # 95 "z_solve.c"
  # 95 "z_solve.c"
  # 95 "z_solve.c"
  for (i_3 = (hmpprt::gr_btidx()), end_31 = nx2_1 - 1 ; i_3 <= end_31 ; i_3 = i_3 + (hmpprt::gr_btnumx()))
  {
   # 17 "<preprocessor>"
   hmpprt::s32 end_29;
   # 17 "<preprocessor>"
   hmpprt::s32 k_1;
   # 96 "z_solve.c"
   # 96 "z_solve.c"
   # 96 "z_solve.c"
   for (k_1 = 0, end_29 = nz2_1 + 1 ; k_1 <= end_29 ; k_1 = k_1 + 1)
   {
    }
   # 17 "<preprocessor>"
   # 17 "<preprocessor>"
   hmpprt::s32 end_30;
   # 17 "<preprocessor>"
   hmpprt::s32 k_2;
   # 102 "z_solve.c"
   # 102 "z_solve.c"
   # 102 "z_solve.c"
   for (k_2 = 0, end_30 = nz2_1 - 1 ; k_2 <= end_30 ; k_2 = k_2 + 1)
   {
    }
   # 5 "<preprocessor>"
  }
  # 5 "<preprocessor>"
 }
 # 5 "<preprocessor>"
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__z_solve_65__uxet4axu_internal_1(hmpprt::s32 nx2, hmpprt::s32 nz2, hmpprt::s32 nj, hmpprt::s32 ni, hmpprt::s32 gp2)
{
 # 38 "<preprocessor>"
 hmpprt::s32 ny2;
 # 5 "<preprocessor>"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(8LL);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (ni), "ni_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (ny2), "ny2_1");
  __hmppcg_call.launch(__hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_2, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 5 "<preprocessor>"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(8LL);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nx2), "nx2_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (ny2), "ny2_3");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nz2), "nz2_1");
  __hmppcg_call.launch(__hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_4, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 5 "<preprocessor>"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(8LL);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nx2), "nx2_3");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (ny2), "ny2_4");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nz2), "nz2_2");
  __hmppcg_call.launch(__hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 5 "<preprocessor>"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(8LL);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (gp2), "gp2_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nx2), "nx2_2");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (ny2), "ny2_2");
  __hmppcg_call.launch(__hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_3, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__z_solve_65__uxet4axu(hmpprt::s32 nx2_4, hmpprt::s32 nz2_3, hmpprt::s32 nj_1, hmpprt::s32 ni_2, hmpprt::s32 gp2_2)
{
 # 1 "<preprocessor>"
 (__hmpp_acc_region__z_solve_65__uxet4axu_internal_1(nx2_4, nz2_3, nj_1, ni_2, gp2_2));
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      __hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "__hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_1");
      __hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_2 = new hmpprt::CUDAGrid(hmpprt_module, "__hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_2");
      __hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_3 = new hmpprt::CUDAGrid(hmpprt_module, "__hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_3");
      __hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_4 = new hmpprt::CUDAGrid(hmpprt_module, "__hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_4");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("__hmpp_acc_region__z_solve_65__uxet4axu", "prototype __hmpp_acc_region__z_solve_65__uxet4axu(nx2: s32, nz2: s32, nj: s32, ni: s32, gp2: s32)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete __hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_1;
      delete __hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_2;
      delete __hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_3;
      delete __hmpp_acc_region__z_solve_65__uxet4axu_parallel_region_4;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
