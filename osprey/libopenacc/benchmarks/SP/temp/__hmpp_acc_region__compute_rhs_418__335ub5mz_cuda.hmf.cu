#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__compute_rhs_418__335ub5mz(hmpprt::s32 nx2, hmpprt::s32 ny2_1, double dssp_2, double* u_1, double* rhs_1, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j, hmpprt::s32 k_1, hmpprt::s32* __hmpp_addr__m)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__compute_rhs_418__335ub5mz_internal_1(hmpprt::s32 nx2_2, hmpprt::s32 ny2_2, double dssp_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  u_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs_2, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32 k_2, hmpprt::s32* __hmpp_addr__m_1)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__compute_rhs_418__335ub5mz_parallel_region_1 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__compute_rhs_418__335ub5mz_parallel_region_1(double dssp, hmpprt::s32 k, hmpprt::s32* m_2, hmpprt::s32 nx2_1, hmpprt::s32 ny2, double* rhs, double* u);
#endif // __HIPCC__




# 5 "<preprocessor>"

#ifdef __HIPCC__

extern "C" __global__ void __hmpp_acc_region__compute_rhs_418__335ub5mz_parallel_region_1(double dssp, hmpprt::s32 k, hmpprt::s32* m_2, hmpprt::s32 nx2_1, hmpprt::s32 ny2, double* rhs, double* u)
{
 # 8 "<preprocessor>"
 hmpprt::s32 iter_per_gang_1;
 # 8 "<preprocessor>"
 hmpprt::s32 first_gang_iter_1;
 # 8 "<preprocessor>"
 hmpprt::s32 last_gang_iter_1;
 # 8 "<preprocessor>"
 iter_per_gang_1 = ((1 + (ny2 - 1) / 192) > 32 ? (1 + (ny2 - 1) / 192) : 32);
 # 8 "<preprocessor>"
 first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
 # 8 "<preprocessor>"
 last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < (ny2 - 1) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : (ny2 - 1));
 # 8 "<preprocessor>"
 hmpprt::s32 j_1;
 # 419 "rhs.c"
 # 419 "rhs.c"
 for (j_1 = first_gang_iter_1 + (hmpprt::gr_btidx()) ; j_1 <= last_gang_iter_1 ; j_1 = j_1 + (hmpprt::gr_btnumx()))
 {
  # 9 "<preprocessor>"
  hmpprt::s32 end_2;
  # 9 "<preprocessor>"
  hmpprt::s32 i_2;
  # 421 "rhs.c"
  # 421 "rhs.c"
  # 421 "rhs.c"
  for (i_2 = 0, end_2 = nx2_1 - 1 ; i_2 <= end_2 ; i_2 = i_2 + 1)
  {
   # 7 "<preprocessor>"
   hmpprt::s32 m_1;
   # 422 "rhs.c"
   # 422 "rhs.c"
   for (m_1 = 0 ; m_1 <= 4 ; m_1 = m_1 + 1)
   {
    # 425 "rhs.c"
    *(rhs + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k + 36uLL * m_1))) = *(rhs + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k + 36uLL * m_1))) - dssp * (*(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k - 2 + 36uLL * m_1))) - (double) 4.0 * *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k - 1 + 36uLL * m_1))) + (double) 6.0 * *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k + 36uLL * m_1))) - (double) 4.0 * *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k + 1 + 36uLL * m_1))));
   }
   # 5 "<preprocessor>"
   # 5 "<preprocessor>"
   *m_2 = 5;
  }
  # 5 "<preprocessor>"
 }
 # 5 "<preprocessor>"
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__compute_rhs_418__335ub5mz_internal_1(hmpprt::s32 nx2_2, hmpprt::s32 ny2_2, double dssp_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  u_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs_2, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32 k_2, hmpprt::s32* __hmpp_addr__m_1)
{
 # 7 "<preprocessor>"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  m;
 # 7 "<preprocessor>"
 hmpprt::Context::getInstance()->allocate((void **) (&m), hmpprt::MS_CUDA_GLOB, 4);
 # 7 "<preprocessor>"
 *m = *__hmpp_addr__m_1;
 # 8 "<preprocessor>"
 hmpprt::s32 j;
 # 9 "<preprocessor>"
 hmpprt::s32 i_1;
 # 9 "<preprocessor>"
 i_1 = *__hmpp_addr__i_1;
 # 28 "<preprocessor>"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(1);
  __hmppcg_call.addLocalParameter(&dssp_1, 8, "dssp");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (k_2), "k");
  __hmppcg_call.addLocalParameter(&m, 8, "m_2");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nx2_2), "nx2_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (ny2_2), "ny2");
  __hmppcg_call.addLocalParameter(&rhs_2, 8, "rhs");
  __hmppcg_call.addLocalParameter(&u_2, 8, "u");
  __hmppcg_call.launch(__hmpp_acc_region__compute_rhs_418__335ub5mz_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 28 "<preprocessor>"
 *__hmpp_addr__i_1 = i_1;
 # 29 "<preprocessor>"
 *__hmpp_addr__j_1 = j;
 # 30 "<preprocessor>"
 *__hmpp_addr__m_1 = *m;
 # 5 "<preprocessor>"
 hmpprt::Context::getInstance()->free((void **) (&m));
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__compute_rhs_418__335ub5mz(hmpprt::s32 nx2, hmpprt::s32 ny2_1, double dssp_2, double* u_1, double* rhs_1, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j, hmpprt::s32 k_1, hmpprt::s32* __hmpp_addr__m)
{
 # 1 "<preprocessor>"
 (__hmpp_acc_region__compute_rhs_418__335ub5mz_internal_1(nx2, ny2_1, dssp_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (u_1), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (rhs_1), __hmpp_addr__i, __hmpp_addr__j, k_1, __hmpp_addr__m));
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      __hmpp_acc_region__compute_rhs_418__335ub5mz_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "__hmpp_acc_region__compute_rhs_418__335ub5mz_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("__hmpp_acc_region__compute_rhs_418__335ub5mz", "prototype __hmpp_acc_region__compute_rhs_418__335ub5mz(nx2: s32, ny2: s32, dssp: double, u: ^cudaglob double, rhs: ^cudaglob double, __hmpp_addr__i: ^host s32, __hmpp_addr__j: ^host s32, k: s32, __hmpp_addr__m: ^host s32)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete __hmpp_acc_region__compute_rhs_418__335ub5mz_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
