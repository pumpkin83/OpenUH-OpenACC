#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__compute_rhs_214__lwub1sww(hmpprt::s32 nx2, hmpprt::s32 ny2, hmpprt::s32 nz2, double c1_2, double c2_2, double con43_2, double* u_2, double* us_2, double* vs_2, double* ws_2, double* qs, double* rho_i_2, double* square, double* rhs_2, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32* __hmpp_addr__k_1)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__compute_rhs_214__lwub1sww_internal_1(hmpprt::s32 nx2_2, hmpprt::s32 ny2_2, hmpprt::s32 nz2_2, double c1, double c2, double con43, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  u, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  us_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  vs, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  ws, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  qs_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rho_i, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  square_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j, hmpprt::s32* __hmpp_addr__k)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__compute_rhs_214__lwub1sww_parallel_region_1 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__compute_rhs_214__lwub1sww_parallel_region_1(double c1_1, double c2_1, double con43_1, double dy1ty1_1, double dy2ty1_1, double dy3ty1_1, double dy4ty1_1, double dy5ty1_1, hmpprt::s32 nx2_1, hmpprt::s32 ny2_1, hmpprt::s32 nz2_1, double* qs_1, double* rho_i_1, double* rhs_1, double* square_1, double ty2_1, double* u_1, double* us, double* vs_1, double* ws_1, double yycon2_1, double yycon3_1, double yycon4_1, double yycon5_1);
#endif // __HIPCC__




# 5 "<preprocessor>"

#ifdef __HIPCC__

extern "C" __global__ void __hmpp_acc_region__compute_rhs_214__lwub1sww_parallel_region_1(double c1_1, double c2_1, double con43_1, double dy1ty1_1, double dy2ty1_1, double dy3ty1_1, double dy4ty1_1, double dy5ty1_1, hmpprt::s32 nx2_1, hmpprt::s32 ny2_1, hmpprt::s32 nz2_1, double* qs_1, double* rho_i_1, double* rhs_1, double* square_1, double ty2_1, double* u_1, double* us, double* vs_1, double* ws_1, double yycon2_1, double yycon3_1, double yycon4_1, double yycon5_1)
{
 # 10 "<preprocessor>"
 hmpprt::s32 iter_per_gang_1;
 # 10 "<preprocessor>"
 hmpprt::s32 first_gang_iter_1;
 # 10 "<preprocessor>"
 hmpprt::s32 last_gang_iter_1;
 # 10 "<preprocessor>"
 iter_per_gang_1 = 1 + (nz2_1 - 1) / 192;
 # 10 "<preprocessor>"
 first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
 # 10 "<preprocessor>"
 last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < (nz2_1 - 1) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : (nz2_1 - 1));
 # 10 "<preprocessor>"
 hmpprt::s32 k_1;
 # 215 "rhs.c"
 # 215 "rhs.c"
 for (k_1 = first_gang_iter_1 ; k_1 <= last_gang_iter_1 ; k_1 = k_1 + 1)
 {
  # 11 "<preprocessor>"
  hmpprt::s32 end_2;
  # 11 "<preprocessor>"
  hmpprt::s32 j_1;
  # 217 "rhs.c"
  # 217 "rhs.c"
  # 217 "rhs.c"
  for (j_1 = (hmpprt::gr_btidx()), end_2 = ny2_1 - 1 ; j_1 <= end_2 ; j_1 = j_1 + (hmpprt::gr_btnumx()))
  {
   # 12 "<preprocessor>"
   hmpprt::s32 end_1;
   # 12 "<preprocessor>"
   hmpprt::s32 i_2;
   # 219 "rhs.c"
   # 219 "rhs.c"
   # 219 "rhs.c"
   for (i_2 = 0, end_1 = nx2_1 - 1 ; i_2 <= end_1 ; i_2 = i_2 + 1)
   {
    # 220 "rhs.c"
    double vp1_1;
    # 220 "rhs.c"
    double vm1_1;
    # 220 "rhs.c"
    double vijk_1;
    # 220 "rhs.c"
    vijk_1 = *(vs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1)));
    # 221 "rhs.c"
    vp1_1 = *(vs_1 + (i_2 + 1) + 37uLL * (j_1 + 2 + 37uLL * (k_1 + 1)));
    # 222 "rhs.c"
    vm1_1 = *(vs_1 + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 1)));
    # 226 "rhs.c"
    *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) = *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) + dy1ty1_1 * (*(u_1 + (i_2 + 1) + 37uLL * (j_1 + 2 + 37uLL * (k_1 + 1))) - (double) 2.0 * *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) + *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 1)))) - ty2_1 * (*(u_1 + (i_2 + 1) + 37uLL * (j_1 + 2 + 37uLL * (k_1 + 73))) - *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 73))));
    # 231 "rhs.c"
    *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 37))) = *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 37))) + dy2ty1_1 * (*(u_1 + (i_2 + 1) + 37uLL * (j_1 + 2 + 37uLL * (k_1 + 37))) - (double) 2.0 * *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 37))) + *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 37)))) + yycon2_1 * (*(us + (i_2 + 1) + 37uLL * (j_1 + 2 + 37uLL * (k_1 + 1))) - (double) 2.0 * *(us + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) + *(us + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 1)))) - ty2_1 * (*(u_1 + (i_2 + 1) + 37uLL * (j_1 + 2 + 37uLL * (k_1 + 37))) * vp1_1 - *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 37))) * vm1_1);
    # 238 "rhs.c"
    *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 73))) = *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 73))) + dy3ty1_1 * (*(u_1 + (i_2 + 1) + 37uLL * (j_1 + 2 + 37uLL * (k_1 + 73))) - (double) 2.0 * *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 73))) + *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 73)))) + yycon2_1 * con43_1 * (vp1_1 - (double) 2.0 * vijk_1 + vm1_1) - ty2_1 * (*(u_1 + (i_2 + 1) + 37uLL * (j_1 + 2 + 37uLL * (k_1 + 73))) * vp1_1 - *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 73))) * vm1_1 + (*(u_1 + (i_2 + 1) + 37uLL * (j_1 + 2 + 37uLL * (k_1 + 145))) - *(square_1 + (i_2 + 1) + 37uLL * (j_1 + 2 + 37uLL * (k_1 + 1))) - *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 145))) + *(square_1 + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 1)))) * c2_1);
    # 243 "rhs.c"
    *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 109))) = *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 109))) + dy4ty1_1 * (*(u_1 + (i_2 + 1) + 37uLL * (j_1 + 2 + 37uLL * (k_1 + 109))) - (double) 2.0 * *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 109))) + *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 109)))) + yycon2_1 * (*(ws_1 + (i_2 + 1) + 37uLL * (j_1 + 2 + 37uLL * (k_1 + 1))) - (double) 2.0 * *(ws_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) + *(ws_1 + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 1)))) - ty2_1 * (*(u_1 + (i_2 + 1) + 37uLL * (j_1 + 2 + 37uLL * (k_1 + 109))) * vp1_1 - *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 109))) * vm1_1);
    # 253 "rhs.c"
    *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 145))) = *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 145))) + dy5ty1_1 * (*(u_1 + (i_2 + 1) + 37uLL * (j_1 + 2 + 37uLL * (k_1 + 145))) - (double) 2.0 * *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 145))) + *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 145)))) + yycon3_1 * (*(qs_1 + (i_2 + 1) + 37uLL * (j_1 + 2 + 37uLL * (k_1 + 1))) - (double) 2.0 * *(qs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) + *(qs_1 + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 1)))) + yycon4_1 * (vp1_1 * vp1_1 - (double) 2.0 * vijk_1 * vijk_1 + vm1_1 * vm1_1) + yycon5_1 * (*(u_1 + (i_2 + 1) + 37uLL * (j_1 + 2 + 37uLL * (k_1 + 145))) * *(rho_i_1 + (i_2 + 1) + 37uLL * (j_1 + 2 + 37uLL * (k_1 + 1))) - (double) 2.0 * *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 145))) * *(rho_i_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) + *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 145))) * *(rho_i_1 + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 1)))) - ty2_1 * ((c1_1 * *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 2 + 37uLL * (k_1 + 145))) - c2_1 * *(square_1 + (i_2 + 1) + 37uLL * (j_1 + 2 + 37uLL * (k_1 + 1)))) * vp1_1 - (c1_1 * *(u_1 + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 145))) - c2_1 * *(square_1 + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 1)))) * vm1_1);
   }
   # 5 "<preprocessor>"
  }
  # 5 "<preprocessor>"
 }
 # 5 "<preprocessor>"
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__compute_rhs_214__lwub1sww_internal_1(hmpprt::s32 nx2_2, hmpprt::s32 ny2_2, hmpprt::s32 nz2_2, double c1, double c2, double con43, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  u, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  us_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  vs, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  ws, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  qs_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rho_i, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  square_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j, hmpprt::s32* __hmpp_addr__k)
{
 # 10 "<preprocessor>"
 hmpprt::s32 k;
 # 11 "<preprocessor>"
 hmpprt::s32 j;
 # 11 "<preprocessor>"
 j = *__hmpp_addr__j;
 # 12 "<preprocessor>"
 hmpprt::s32 i_1;
 # 12 "<preprocessor>"
 i_1 = *__hmpp_addr__i;
 # 13 "<preprocessor>"
 double dy5ty1;
 # 14 "<preprocessor>"
 double dy4ty1;
 # 15 "<preprocessor>"
 double dy3ty1;
 # 16 "<preprocessor>"
 double dy2ty1;
 # 17 "<preprocessor>"
 double dy1ty1;
 # 18 "<preprocessor>"
 double yycon5;
 # 19 "<preprocessor>"
 double yycon4;
 # 20 "<preprocessor>"
 double yycon3;
 # 21 "<preprocessor>"
 double yycon2;
 # 22 "<preprocessor>"
 double ty2;
 # 76 "<preprocessor>"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(1);
  __hmppcg_call.addLocalParameter(&c1, 8, "c1_1");
  __hmppcg_call.addLocalParameter(&c2, 8, "c2_1");
  __hmppcg_call.addLocalParameter(&con43, 8, "con43_1");
  __hmppcg_call.addLocalParameter(&dy1ty1, 8, "dy1ty1_1");
  __hmppcg_call.addLocalParameter(&dy2ty1, 8, "dy2ty1_1");
  __hmppcg_call.addLocalParameter(&dy3ty1, 8, "dy3ty1_1");
  __hmppcg_call.addLocalParameter(&dy4ty1, 8, "dy4ty1_1");
  __hmppcg_call.addLocalParameter(&dy5ty1, 8, "dy5ty1_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nx2_2), "nx2_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (ny2_2), "ny2_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nz2_2), "nz2_1");
  __hmppcg_call.addLocalParameter(&qs_2, 8, "qs_1");
  __hmppcg_call.addLocalParameter(&rho_i, 8, "rho_i_1");
  __hmppcg_call.addLocalParameter(&rhs, 8, "rhs_1");
  __hmppcg_call.addLocalParameter(&square_2, 8, "square_1");
  __hmppcg_call.addLocalParameter(&ty2, 8, "ty2_1");
  __hmppcg_call.addLocalParameter(&u, 8, "u_1");
  __hmppcg_call.addLocalParameter(&us_1, 8, "us");
  __hmppcg_call.addLocalParameter(&vs, 8, "vs_1");
  __hmppcg_call.addLocalParameter(&ws, 8, "ws_1");
  __hmppcg_call.addLocalParameter(&yycon2, 8, "yycon2_1");
  __hmppcg_call.addLocalParameter(&yycon3, 8, "yycon3_1");
  __hmppcg_call.addLocalParameter(&yycon4, 8, "yycon4_1");
  __hmppcg_call.addLocalParameter(&yycon5, 8, "yycon5_1");
  __hmppcg_call.launch(__hmpp_acc_region__compute_rhs_214__lwub1sww_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 76 "<preprocessor>"
 *__hmpp_addr__i = i_1;
 # 77 "<preprocessor>"
 *__hmpp_addr__j = j;
 # 78 "<preprocessor>"
 *__hmpp_addr__k = k;
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__compute_rhs_214__lwub1sww(hmpprt::s32 nx2, hmpprt::s32 ny2, hmpprt::s32 nz2, double c1_2, double c2_2, double con43_2, double* u_2, double* us_2, double* vs_2, double* ws_2, double* qs, double* rho_i_2, double* square, double* rhs_2, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32* __hmpp_addr__k_1)
{
 # 1 "<preprocessor>"
 (__hmpp_acc_region__compute_rhs_214__lwub1sww_internal_1(nx2, ny2, nz2, c1_2, c2_2, con43_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (u_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (us_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (vs_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (ws_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (qs), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (rho_i_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (square), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (rhs_2), __hmpp_addr__i_1, __hmpp_addr__j_1, __hmpp_addr__k_1));
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      __hmpp_acc_region__compute_rhs_214__lwub1sww_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "__hmpp_acc_region__compute_rhs_214__lwub1sww_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("__hmpp_acc_region__compute_rhs_214__lwub1sww", "prototype __hmpp_acc_region__compute_rhs_214__lwub1sww(nx2: s32, ny2: s32, nz2: s32, c1: double, c2: double, con43: double, u: ^cudaglob double, us: ^cudaglob double, vs: ^cudaglob double, ws: ^cudaglob double, qs: ^cudaglob double, rho_i: ^cudaglob double, square: ^cudaglob double, rhs: ^cudaglob double, __hmpp_addr__i: ^host s32, __hmpp_addr__j: ^host s32, __hmpp_addr__k: ^host s32)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete __hmpp_acc_region__compute_rhs_214__lwub1sww_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
