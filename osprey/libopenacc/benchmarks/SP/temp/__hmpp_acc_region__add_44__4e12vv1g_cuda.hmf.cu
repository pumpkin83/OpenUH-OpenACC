#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__add_44__4e12vv1g()
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__add_44__4e12vv1g_internal_1()
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__add_44__4e12vv1g_parallel_region_1 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__add_44__4e12vv1g_parallel_region_1(hmpprt::s32 nx2_1, hmpprt::s32 ny2_1, hmpprt::s32 nz2_1);
#endif // __HIPCC__




# 5 "<preprocessor>"

#ifdef __HIPCC__

extern "C" __global__ void __hmpp_acc_region__add_44__4e12vv1g_parallel_region_1(hmpprt::s32 nx2_1, hmpprt::s32 ny2_1, hmpprt::s32 nz2_1)
{
 # 8 "<preprocessor>"
 hmpprt::s32 iter_per_gang_1;
 # 8 "<preprocessor>"
 hmpprt::s32 first_gang_iter_1;
 # 8 "<preprocessor>"
 hmpprt::s32 last_gang_iter_1;
 # 8 "<preprocessor>"
 iter_per_gang_1 = 1 + (nz2_1 - 1) / 192;
 # 8 "<preprocessor>"
 first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
 # 8 "<preprocessor>"
 last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < (nz2_1 - 1) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : (nz2_1 - 1));
 # 8 "<preprocessor>"
 hmpprt::s32 k_1;
 # 45 "add.c"
 # 45 "add.c"
 for (k_1 = first_gang_iter_1 ; k_1 <= last_gang_iter_1 ; k_1 = k_1 + 1)
 {
  # 9 "<preprocessor>"
  hmpprt::s32 end_3;
  # 9 "<preprocessor>"
  hmpprt::s32 j_1;
  # 47 "add.c"
  # 47 "add.c"
  # 47 "add.c"
  for (j_1 = (hmpprt::gr_btidx()), end_3 = ny2_1 - 1 ; j_1 <= end_3 ; j_1 = j_1 + (hmpprt::gr_btnumx()))
  {
   # 10 "<preprocessor>"
   hmpprt::s32 end_2;
   # 10 "<preprocessor>"
   hmpprt::s32 i_2;
   # 49 "add.c"
   # 49 "add.c"
   # 49 "add.c"
   for (i_2 = 0, end_2 = nx2_1 - 1 ; i_2 <= end_2 ; i_2 = i_2 + 1)
   {
    # 7 "<preprocessor>"
    hmpprt::s32 m_1;
    # 50 "add.c"
    # 50 "add.c"
    for (m_1 = 0 ; m_1 <= 4 ; m_1 = m_1 + 1)
    {
     }
    # 5 "<preprocessor>"
   }
   # 5 "<preprocessor>"
  }
  # 5 "<preprocessor>"
 }
 # 5 "<preprocessor>"
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__add_44__4e12vv1g_internal_1()
{
 # 13 "<preprocessor>"
 hmpprt::s32 nz2;
 # 14 "<preprocessor>"
 hmpprt::s32 ny2;
 # 15 "<preprocessor>"
 hmpprt::s32 nx2;
 # 5 "<preprocessor>"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(1);
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nx2), "nx2_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (ny2), "ny2_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nz2), "nz2_1");
  __hmppcg_call.launch(__hmpp_acc_region__add_44__4e12vv1g_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__add_44__4e12vv1g()
{
 # 1 "<preprocessor>"
 (__hmpp_acc_region__add_44__4e12vv1g_internal_1());
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      __hmpp_acc_region__add_44__4e12vv1g_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "__hmpp_acc_region__add_44__4e12vv1g_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("__hmpp_acc_region__add_44__4e12vv1g", "prototype __hmpp_acc_region__add_44__4e12vv1g()");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete __hmpp_acc_region__add_44__4e12vv1g_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
