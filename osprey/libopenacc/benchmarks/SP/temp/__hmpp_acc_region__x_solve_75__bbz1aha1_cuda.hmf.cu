#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__x_solve_75__bbz1aha1(hmpprt::u64* __hmpp_vla_sizes__rhsX_2, hmpprt::s32 nx2_2, hmpprt::s32 nz2_2, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32* __hmpp_addr__k_1, hmpprt::s32 ni_2, hmpprt::s32 gp0_2, double* rhsX_2)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__x_solve_75__bbz1aha1_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u64>  __hmpp_vla_sizes__rhsX_1, hmpprt::s32 nx2, hmpprt::s32 nz2, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j, hmpprt::s32* __hmpp_addr__k, hmpprt::s32 ni, hmpprt::s32 gp0_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhsX)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__x_solve_75__bbz1aha1_parallel_region_1 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__x_solve_75__bbz1aha1_parallel_region_1(double c1c5_1, double c2dttx1_1, double c3c4_1, double comz1_1, double comz4_1, double comz5_1, double comz6_1, double con43_1, double dttx1_1, double dttx2_1, double dx1_1, double dx2_1, double dx5_1, double dxmax_1, hmpprt::s32 gp0, hmpprt::s32* i_9, hmpprt::s32* j_13, double* lhsX_1, double* lhsmX_1, double* lhspX, hmpprt::s32 ni_1, hmpprt::s32 nx2_1, hmpprt::s32 ny2_1, hmpprt::s32 nz2_1, double* rho_i_1, double* rhonX_1, hmpprt::u64* __hmpp_vla_sizes__rhsX, double* rhsX_1, double* speed_1, double* us_1);
#endif // __HIPCC__




# 5 "<preprocessor>"

#ifdef __HIPCC__

extern "C" __global__ void __hmpp_acc_region__x_solve_75__bbz1aha1_parallel_region_1(double c1c5_1, double c2dttx1_1, double c3c4_1, double comz1_1, double comz4_1, double comz5_1, double comz6_1, double con43_1, double dttx1_1, double dttx2_1, double dx1_1, double dx2_1, double dx5_1, double dxmax_1, hmpprt::s32 gp0, hmpprt::s32* i_9, hmpprt::s32* j_13, double* lhsX_1, double* lhsmX_1, double* lhspX, hmpprt::s32 ni_1, hmpprt::s32 nx2_1, hmpprt::s32 ny2_1, hmpprt::s32 nz2_1, double* rho_i_1, double* rhonX_1, hmpprt::u64* __hmpp_vla_sizes__rhsX, double* rhsX_1, double* speed_1, double* us_1)
{
 # 17 "<preprocessor>"
 hmpprt::s32 end_29;
 # 17 "<preprocessor>"
 hmpprt::s32 k_1;
 # 77 "x_solve.c"
 # 77 "x_solve.c"
 # 77 "x_solve.c"
 for (k_1 = 0, end_29 = nz2_1 - 1 ; k_1 <= end_29 ; k_1 = k_1 + 1)
 {
  # 18 "<preprocessor>"
  hmpprt::s32 iter_per_gang_1;
  # 18 "<preprocessor>"
  hmpprt::s32 first_gang_iter_1;
  # 18 "<preprocessor>"
  hmpprt::s32 last_gang_iter_1;
  # 18 "<preprocessor>"
  iter_per_gang_1 = ((1 + (ny2_1 - 1) / 192) > 8LL ? (1 + (ny2_1 - 1) / 192) : 8LL);
  # 18 "<preprocessor>"
  first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
  # 18 "<preprocessor>"
  last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < (ny2_1 - 1) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : (ny2_1 - 1));
  # 18 "<preprocessor>"
  hmpprt::s32 j_1;
  # 79 "x_solve.c"
  # 79 "x_solve.c"
  for (j_1 = first_gang_iter_1 + (hmpprt::gr_btidy()) ; j_1 <= last_gang_iter_1 ; j_1 = j_1 + (hmpprt::gr_btnumy()))
  {
   # 14 "<preprocessor>"
   hmpprt::s32 m_1;
   # 80 "x_solve.c"
   # 80 "x_solve.c"
   for (m_1 = (hmpprt::gr_btidx()) ; m_1 <= 4 ; m_1 = m_1 + (hmpprt::gr_btnumx()))
   {
    # 81 "x_solve.c"
    *(lhsX_1 + (j_1 + 1) + 37uLL * (37uLL * (k_1 + 1 + 36uLL * m_1))) = (double) 0.0;
    # 82 "x_solve.c"
    *(lhspX + (j_1 + 1) + 37uLL * (37uLL * (k_1 + 1 + 36uLL * m_1))) = (double) 0.0;
    # 83 "x_solve.c"
    *(lhsmX_1 + (j_1 + 1) + 37uLL * (37uLL * (k_1 + 1 + 36uLL * m_1))) = (double) 0.0;
    # 84 "x_solve.c"
    *(lhsX_1 + (j_1 + 1) + 37uLL * (ni_1 + 37uLL * (k_1 + 1 + 36uLL * m_1))) = (double) 0.0;
    # 85 "x_solve.c"
    *(lhspX + (j_1 + 1) + 37uLL * (ni_1 + 37uLL * (k_1 + 1 + 36uLL * m_1))) = (double) 0.0;
    # 86 "x_solve.c"
    *(lhsmX_1 + (j_1 + 1) + 37uLL * (ni_1 + 37uLL * (k_1 + 1 + 36uLL * m_1))) = (double) 0.0;
   }
   # 88 "x_solve.c"
   # 88 "x_solve.c"
   *(lhsX_1 + (j_1 + 1) + 37uLL * (37uLL * (k_1 + 73))) = (double) 1.0;
   # 89 "x_solve.c"
   *(lhspX + (j_1 + 1) + 37uLL * (37uLL * (k_1 + 73))) = (double) 1.0;
   # 90 "x_solve.c"
   *(lhsmX_1 + (j_1 + 1) + 37uLL * (37uLL * (k_1 + 73))) = (double) 1.0;
   # 91 "x_solve.c"
   *(lhsX_1 + (j_1 + 1) + 37uLL * (ni_1 + 37uLL * (k_1 + 73))) = (double) 1.0;
   # 92 "x_solve.c"
   *(lhspX + (j_1 + 1) + 37uLL * (ni_1 + 37uLL * (k_1 + 73))) = (double) 1.0;
   # 93 "x_solve.c"
   *(lhsmX_1 + (j_1 + 1) + 37uLL * (ni_1 + 37uLL * (k_1 + 73))) = (double) 1.0;
  }
  # 18 "<preprocessor>"
  # 18 "<preprocessor>"
  hmpprt::s32 end_5;
  # 18 "<preprocessor>"
  hmpprt::s32 j_2;
  # 105 "x_solve.c"
  # 105 "x_solve.c"
  # 105 "x_solve.c"
  for (j_2 = 0, end_5 = ny2_1 - 1 ; j_2 <= end_5 ; j_2 = j_2 + 1)
  {
   # 19 "<preprocessor>"
   hmpprt::s32 iter_per_gang_2;
   # 19 "<preprocessor>"
   hmpprt::s32 first_gang_iter_2;
   # 19 "<preprocessor>"
   hmpprt::s32 last_gang_iter_2;
   # 19 "<preprocessor>"
   iter_per_gang_2 = ((1 + (gp0 - 1) / 192) > 8LL ? (1 + (gp0 - 1) / 192) : 8LL);
   # 19 "<preprocessor>"
   first_gang_iter_2 = (hmpprt::gr_gbidx()) * iter_per_gang_2;
   # 19 "<preprocessor>"
   last_gang_iter_2 = ((first_gang_iter_2 + iter_per_gang_2 - 1) < (gp0 - 1) ? (first_gang_iter_2 + iter_per_gang_2 - 1) : (gp0 - 1));
   # 19 "<preprocessor>"
   hmpprt::s32 i_2;
   # 106 "x_solve.c"
   # 106 "x_solve.c"
   for (i_2 = first_gang_iter_2 + (hmpprt::gr_btidy()) ; i_2 <= last_gang_iter_2 ; i_2 = i_2 + (hmpprt::gr_btnumy()))
   {
    # 107 "x_solve.c"
    double ru1_1;
    # 107 "x_solve.c"
    ru1_1 = c3c4_1 * *(rho_i_1 + i_2 + 37uLL * (j_2 + 1 + 37uLL * (k_1 + 1)));
    # 109 "x_solve.c"
    *(rhonX_1 + (j_2 + 1) + 37uLL * (i_2 + 36uLL * (k_1 + 1))) = ( (( (dx2_1 + con43_1 * ru1_1 > dx5_1 + c1c5_1 * ru1_1) ? (dx2_1 + con43_1 * ru1_1) : (dx5_1 + c1c5_1 * ru1_1)) > ( (dxmax_1 + ru1_1 > dx1_1) ? (dxmax_1 + ru1_1) : dx1_1)) ? ( (dx2_1 + con43_1 * ru1_1 > dx5_1 + c1c5_1 * ru1_1) ? (dx2_1 + con43_1 * ru1_1) : (dx5_1 + c1c5_1 * ru1_1)) : ( (dxmax_1 + ru1_1 > dx1_1) ? (dxmax_1 + ru1_1) : dx1_1));
   }
   # 18 "<preprocessor>"
   # 18 "<preprocessor>"
   if (1 <= nx2_1)
   {
    # 19 "<preprocessor>"
    hmpprt::s32 iter_per_gang_3;
    # 19 "<preprocessor>"
    hmpprt::s32 first_gang_iter_3;
    # 19 "<preprocessor>"
    hmpprt::s32 last_gang_iter_3;
    # 19 "<preprocessor>"
    hmpprt::s32 tmp_30;
    # 19 "<preprocessor>"
    iter_per_gang_3 = ((1 + (nx2_1 - 1) / 192) > 8LL ? (1 + (nx2_1 - 1) / 192) : 8LL);
    # 19 "<preprocessor>"
    first_gang_iter_3 = (hmpprt::gr_gbidx()) * iter_per_gang_3;
    # 19 "<preprocessor>"
    last_gang_iter_3 = ((first_gang_iter_3 + iter_per_gang_3 - 1) < (nx2_1 - 1) ? (first_gang_iter_3 + iter_per_gang_3 - 1) : (nx2_1 - 1));
    # 19 "<preprocessor>"
    hmpprt::s32 i_3;
    # 112 "x_solve.c"
    # 112 "x_solve.c"
    for (i_3 = first_gang_iter_3 + (hmpprt::gr_btidy()) ; i_3 <= last_gang_iter_3 ; i_3 = i_3 + (hmpprt::gr_btnumy()))
    {
     # 113 "x_solve.c"
     *(lhsX_1 + (j_2 + 1) + 37uLL * (i_3 + 1 + 37uLL * (k_1 + 1))) = (double) 0.0;
     # 117 "x_solve.c"
     *(lhsX_1 + (j_2 + 1) + 37uLL * (i_3 + 1 + 37uLL * (k_1 + 37))) =  - dttx2_1 * *(us_1 + i_3 + 37uLL * (j_2 + 1 + 37uLL * (k_1 + 1))) - dttx1_1 * *(rhonX_1 + (j_2 + 1) + 37uLL * (i_3 + 36uLL * (k_1 + 1)));
     # 118 "x_solve.c"
     *(lhsX_1 + (j_2 + 1) + 37uLL * (i_3 + 1 + 37uLL * (k_1 + 73))) = (double) 1.0 + c2dttx1_1 * *(rhonX_1 + (j_2 + 1) + 37uLL * (i_3 + 1 + 36uLL * (k_1 + 1)));
     # 119 "x_solve.c"
     *(lhsX_1 + (j_2 + 1) + 37uLL * (i_3 + 1 + 37uLL * (k_1 + 109))) = dttx2_1 * *(us_1 + (i_3 + 2) + 37uLL * (j_2 + 1 + 37uLL * (k_1 + 1))) - dttx1_1 * *(rhonX_1 + (j_2 + 1) + 37uLL * (i_3 + 2 + 36uLL * (k_1 + 1)));
     # 120 "x_solve.c"
     *(lhsX_1 + (j_2 + 1) + 37uLL * (i_3 + 1 + 37uLL * (k_1 + 145))) = (double) 0.0;
    }
    # 18 "<preprocessor>"
    # 18 "<preprocessor>"
    tmp_30 = nx2_1 + 1;
    # 18 "<preprocessor>"
    if ((hmpprt::gr_btidy()) == 0)
    {
     # 18 "<preprocessor>"
     *i_9 = tmp_30;
    }
    # 18 "<preprocessor>"
    (hmpprt::gr_barrier());
   }
   else
   {
    # 18 "<preprocessor>"
    if ((hmpprt::gr_btidy()) == 0)
    {
     # 18 "<preprocessor>"
     *i_9 = 1;
    }
    # 18 "<preprocessor>"
    (hmpprt::gr_barrier());
   }
  }
  # 18 "<preprocessor>"
  # 18 "<preprocessor>"
  hmpprt::s32 end_6;
  # 18 "<preprocessor>"
  hmpprt::s32 j_3;
  # 127 "x_solve.c"
  # 127 "x_solve.c"
  # 127 "x_solve.c"
  for (j_3 = 0, end_6 = ny2_1 - 1 ; j_3 <= end_6 ; j_3 = j_3 + 1)
  {
   # 18 "<preprocessor>"
   double tmp_33;
   # 18 "<preprocessor>"
   double tmp_34;
   # 18 "<preprocessor>"
   double tmp_35;
   # 18 "<preprocessor>"
   double tmp_36;
   # 18 "<preprocessor>"
   double tmp_37;
   # 18 "<preprocessor>"
   double tmp_38;
   # 18 "<preprocessor>"
   double tmp_39;
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *i_9 = 1;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_33 = *(lhsX_1 + (j_3 + 1) + 37uLL * (1 + 37uLL * (k_1 + 73))) + comz5_1;
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(lhsX_1 + (j_3 + 1) + 37uLL * (1 + 37uLL * (k_1 + 73))) = tmp_33;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_34 = *(lhsX_1 + (j_3 + 1) + 37uLL * (1 + 37uLL * (k_1 + 109))) - comz4_1;
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(lhsX_1 + (j_3 + 1) + 37uLL * (1 + 37uLL * (k_1 + 109))) = tmp_34;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_35 = *(lhsX_1 + (j_3 + 1) + 37uLL * (1 + 37uLL * (k_1 + 145))) + comz1_1;
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(lhsX_1 + (j_3 + 1) + 37uLL * (1 + 37uLL * (k_1 + 145))) = tmp_35;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_36 = *(lhsX_1 + (j_3 + 1) + 37uLL * (2 + 37uLL * (k_1 + 37))) - comz4_1;
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(lhsX_1 + (j_3 + 1) + 37uLL * (2 + 37uLL * (k_1 + 37))) = tmp_36;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_37 = *(lhsX_1 + (j_3 + 1) + 37uLL * (2 + 37uLL * (k_1 + 73))) + comz6_1;
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(lhsX_1 + (j_3 + 1) + 37uLL * (2 + 37uLL * (k_1 + 73))) = tmp_37;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_38 = *(lhsX_1 + (j_3 + 1) + 37uLL * (2 + 37uLL * (k_1 + 109))) - comz4_1;
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(lhsX_1 + (j_3 + 1) + 37uLL * (2 + 37uLL * (k_1 + 109))) = tmp_38;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_39 = *(lhsX_1 + (j_3 + 1) + 37uLL * (2 + 37uLL * (k_1 + 145))) + comz1_1;
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(lhsX_1 + (j_3 + 1) + 37uLL * (2 + 37uLL * (k_1 + 145))) = tmp_39;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
  }
  # 18 "<preprocessor>"
  # 18 "<preprocessor>"
  hmpprt::s32 end_8;
  # 18 "<preprocessor>"
  hmpprt::s32 j_4;
  # 139 "x_solve.c"
  # 139 "x_solve.c"
  # 139 "x_solve.c"
  for (j_4 = 0, end_8 = ny2_1 - 1 ; j_4 <= end_8 ; j_4 = j_4 + 1)
  {
   # 18 "<preprocessor>"
   if (3 <= gp0 - 4)
   {
    # 19 "<preprocessor>"
    hmpprt::s32 iter_per_gang_4;
    # 19 "<preprocessor>"
    hmpprt::s32 first_gang_iter_4;
    # 19 "<preprocessor>"
    hmpprt::s32 last_gang_iter_4;
    # 19 "<preprocessor>"
    hmpprt::s32 tmp_40;
    # 19 "<preprocessor>"
    iter_per_gang_4 = ((1 + (gp0 - 7) / 192) > 8LL ? (1 + (gp0 - 7) / 192) : 8LL);
    # 19 "<preprocessor>"
    first_gang_iter_4 = (hmpprt::gr_gbidx()) * iter_per_gang_4;
    # 19 "<preprocessor>"
    last_gang_iter_4 = ((first_gang_iter_4 + iter_per_gang_4 - 1) < (gp0 - 7) ? (first_gang_iter_4 + iter_per_gang_4 - 1) : (gp0 - 7));
    # 19 "<preprocessor>"
    hmpprt::s32 i_4;
    # 140 "x_solve.c"
    # 140 "x_solve.c"
    for (i_4 = first_gang_iter_4 + (hmpprt::gr_btidy()) ; i_4 <= last_gang_iter_4 ; i_4 = i_4 + (hmpprt::gr_btnumy()))
    {
     # 141 "x_solve.c"
     *(lhsX_1 + (j_4 + 1) + 37uLL * (i_4 + 3 + 37uLL * (k_1 + 1))) = *(lhsX_1 + (j_4 + 1) + 37uLL * (i_4 + 3 + 37uLL * (k_1 + 1))) + comz1_1;
     # 142 "x_solve.c"
     *(lhsX_1 + (j_4 + 1) + 37uLL * (i_4 + 3 + 37uLL * (k_1 + 37))) = *(lhsX_1 + (j_4 + 1) + 37uLL * (i_4 + 3 + 37uLL * (k_1 + 37))) - comz4_1;
     # 143 "x_solve.c"
     *(lhsX_1 + (j_4 + 1) + 37uLL * (i_4 + 3 + 37uLL * (k_1 + 73))) = *(lhsX_1 + (j_4 + 1) + 37uLL * (i_4 + 3 + 37uLL * (k_1 + 73))) + comz6_1;
     # 144 "x_solve.c"
     *(lhsX_1 + (j_4 + 1) + 37uLL * (i_4 + 3 + 37uLL * (k_1 + 109))) = *(lhsX_1 + (j_4 + 1) + 37uLL * (i_4 + 3 + 37uLL * (k_1 + 109))) - comz4_1;
     # 145 "x_solve.c"
     *(lhsX_1 + (j_4 + 1) + 37uLL * (i_4 + 3 + 37uLL * (k_1 + 145))) = *(lhsX_1 + (j_4 + 1) + 37uLL * (i_4 + 3 + 37uLL * (k_1 + 145))) + comz1_1;
    }
    # 18 "<preprocessor>"
    # 18 "<preprocessor>"
    tmp_40 = gp0 +  -3;
    # 18 "<preprocessor>"
    if ((hmpprt::gr_btidy()) == 0)
    {
     # 18 "<preprocessor>"
     *i_9 = tmp_40;
    }
    # 18 "<preprocessor>"
    (hmpprt::gr_barrier());
   }
   else
   {
    # 18 "<preprocessor>"
    if ((hmpprt::gr_btidy()) == 0)
    {
     # 18 "<preprocessor>"
     *i_9 = 3;
    }
    # 18 "<preprocessor>"
    (hmpprt::gr_barrier());
   }
  }
  # 18 "<preprocessor>"
  # 18 "<preprocessor>"
  hmpprt::s32 end_9;
  # 18 "<preprocessor>"
  hmpprt::s32 j_5;
  # 149 "x_solve.c"
  # 149 "x_solve.c"
  # 149 "x_solve.c"
  for (j_5 = 0, end_9 = ny2_1 - 1 ; j_5 <= end_9 ; j_5 = j_5 + 1)
  {
   # 18 "<preprocessor>"
   hmpprt::s32 tmp_42;
   # 18 "<preprocessor>"
   double tmp_43;
   # 18 "<preprocessor>"
   double tmp_44;
   # 18 "<preprocessor>"
   double tmp_45;
   # 18 "<preprocessor>"
   double tmp_46;
   # 18 "<preprocessor>"
   double tmp_47;
   # 18 "<preprocessor>"
   double tmp_48;
   # 18 "<preprocessor>"
   double tmp_49;
   # 18 "<preprocessor>"
   tmp_42 = gp0 - 3;
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *i_9 = tmp_42;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_43 = *(lhsX_1 + (j_5 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 1))) + comz1_1;
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(lhsX_1 + (j_5 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 1))) = tmp_43;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_44 = *(lhsX_1 + (j_5 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 37))) - comz4_1;
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(lhsX_1 + (j_5 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 37))) = tmp_44;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_45 = *(lhsX_1 + (j_5 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 73))) + comz6_1;
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(lhsX_1 + (j_5 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 73))) = tmp_45;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_46 = *(lhsX_1 + (j_5 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 109))) - comz4_1;
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(lhsX_1 + (j_5 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 109))) = tmp_46;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_47 = *(lhsX_1 + (j_5 + 1) + 37uLL * (*i_9 + 1 + 37uLL * (k_1 + 1))) + comz1_1;
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(lhsX_1 + (j_5 + 1) + 37uLL * (*i_9 + 1 + 37uLL * (k_1 + 1))) = tmp_47;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_48 = *(lhsX_1 + (j_5 + 1) + 37uLL * (*i_9 + 1 + 37uLL * (k_1 + 37))) - comz4_1;
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(lhsX_1 + (j_5 + 1) + 37uLL * (*i_9 + 1 + 37uLL * (k_1 + 37))) = tmp_48;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_49 = *(lhsX_1 + (j_5 + 1) + 37uLL * (*i_9 + 1 + 37uLL * (k_1 + 73))) + comz5_1;
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(lhsX_1 + (j_5 + 1) + 37uLL * (*i_9 + 1 + 37uLL * (k_1 + 73))) = tmp_49;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
  }
  # 18 "<preprocessor>"
  # 18 "<preprocessor>"
  hmpprt::s32 end_11;
  # 18 "<preprocessor>"
  hmpprt::s32 j_6;
  # 165 "x_solve.c"
  # 165 "x_solve.c"
  # 165 "x_solve.c"
  for (j_6 = 0, end_11 = ny2_1 - 1 ; j_6 <= end_11 ; j_6 = j_6 + 1)
  {
   # 18 "<preprocessor>"
   if (1 <= nx2_1)
   {
    # 19 "<preprocessor>"
    hmpprt::s32 iter_per_gang_5;
    # 19 "<preprocessor>"
    hmpprt::s32 first_gang_iter_5;
    # 19 "<preprocessor>"
    hmpprt::s32 last_gang_iter_5;
    # 19 "<preprocessor>"
    hmpprt::s32 tmp_50;
    # 19 "<preprocessor>"
    iter_per_gang_5 = ((1 + (nx2_1 - 1) / 192) > 8LL ? (1 + (nx2_1 - 1) / 192) : 8LL);
    # 19 "<preprocessor>"
    first_gang_iter_5 = (hmpprt::gr_gbidx()) * iter_per_gang_5;
    # 19 "<preprocessor>"
    last_gang_iter_5 = ((first_gang_iter_5 + iter_per_gang_5 - 1) < (nx2_1 - 1) ? (first_gang_iter_5 + iter_per_gang_5 - 1) : (nx2_1 - 1));
    # 19 "<preprocessor>"
    hmpprt::s32 i_5;
    # 166 "x_solve.c"
    # 166 "x_solve.c"
    for (i_5 = first_gang_iter_5 + (hmpprt::gr_btidy()) ; i_5 <= last_gang_iter_5 ; i_5 = i_5 + (hmpprt::gr_btnumy()))
    {
     # 167 "x_solve.c"
     *(lhspX + (j_6 + 1) + 37uLL * (i_5 + 1 + 37uLL * (k_1 + 1))) = *(lhsX_1 + (j_6 + 1) + 37uLL * (i_5 + 1 + 37uLL * (k_1 + 1)));
     # 168 "x_solve.c"
     *(lhspX + (j_6 + 1) + 37uLL * (i_5 + 1 + 37uLL * (k_1 + 37))) = *(lhsX_1 + (j_6 + 1) + 37uLL * (i_5 + 1 + 37uLL * (k_1 + 37))) - dttx2_1 * *(speed_1 + i_5 + 37uLL * (j_6 + 1 + 37uLL * (k_1 + 1)));
     # 169 "x_solve.c"
     *(lhspX + (j_6 + 1) + 37uLL * (i_5 + 1 + 37uLL * (k_1 + 73))) = *(lhsX_1 + (j_6 + 1) + 37uLL * (i_5 + 1 + 37uLL * (k_1 + 73)));
     # 170 "x_solve.c"
     *(lhspX + (j_6 + 1) + 37uLL * (i_5 + 1 + 37uLL * (k_1 + 109))) = *(lhsX_1 + (j_6 + 1) + 37uLL * (i_5 + 1 + 37uLL * (k_1 + 109))) + dttx2_1 * *(speed_1 + (i_5 + 2) + 37uLL * (j_6 + 1 + 37uLL * (k_1 + 1)));
     # 171 "x_solve.c"
     *(lhspX + (j_6 + 1) + 37uLL * (i_5 + 1 + 37uLL * (k_1 + 145))) = *(lhsX_1 + (j_6 + 1) + 37uLL * (i_5 + 1 + 37uLL * (k_1 + 145)));
     # 172 "x_solve.c"
     *(lhsmX_1 + (j_6 + 1) + 37uLL * (i_5 + 1 + 37uLL * (k_1 + 1))) = *(lhsX_1 + (j_6 + 1) + 37uLL * (i_5 + 1 + 37uLL * (k_1 + 1)));
     # 173 "x_solve.c"
     *(lhsmX_1 + (j_6 + 1) + 37uLL * (i_5 + 1 + 37uLL * (k_1 + 37))) = *(lhsX_1 + (j_6 + 1) + 37uLL * (i_5 + 1 + 37uLL * (k_1 + 37))) + dttx2_1 * *(speed_1 + i_5 + 37uLL * (j_6 + 1 + 37uLL * (k_1 + 1)));
     # 174 "x_solve.c"
     *(lhsmX_1 + (j_6 + 1) + 37uLL * (i_5 + 1 + 37uLL * (k_1 + 73))) = *(lhsX_1 + (j_6 + 1) + 37uLL * (i_5 + 1 + 37uLL * (k_1 + 73)));
     # 175 "x_solve.c"
     *(lhsmX_1 + (j_6 + 1) + 37uLL * (i_5 + 1 + 37uLL * (k_1 + 109))) = *(lhsX_1 + (j_6 + 1) + 37uLL * (i_5 + 1 + 37uLL * (k_1 + 109))) - dttx2_1 * *(speed_1 + (i_5 + 2) + 37uLL * (j_6 + 1 + 37uLL * (k_1 + 1)));
     # 176 "x_solve.c"
     *(lhsmX_1 + (j_6 + 1) + 37uLL * (i_5 + 1 + 37uLL * (k_1 + 145))) = *(lhsX_1 + (j_6 + 1) + 37uLL * (i_5 + 1 + 37uLL * (k_1 + 145)));
    }
    # 18 "<preprocessor>"
    # 18 "<preprocessor>"
    tmp_50 = nx2_1 + 1;
    # 18 "<preprocessor>"
    if ((hmpprt::gr_btidy()) == 0)
    {
     # 18 "<preprocessor>"
     *i_9 = tmp_50;
    }
    # 18 "<preprocessor>"
    (hmpprt::gr_barrier());
   }
   else
   {
    # 18 "<preprocessor>"
    if ((hmpprt::gr_btidy()) == 0)
    {
     # 18 "<preprocessor>"
     *i_9 = 1;
    }
    # 18 "<preprocessor>"
    (hmpprt::gr_barrier());
   }
  }
  # 18 "<preprocessor>"
  # 18 "<preprocessor>"
  hmpprt::s32 end_16;
  # 18 "<preprocessor>"
  hmpprt::s32 j_7;
  # 187 "x_solve.c"
  # 187 "x_solve.c"
  # 187 "x_solve.c"
  for (j_7 = 0, end_16 = ny2_1 - 1 ; j_7 <= end_16 ; j_7 = j_7 + 1)
  {
   # 18 "<preprocessor>"
   if (0 <= gp0 - 3)
   {
    # 19 "<preprocessor>"
    hmpprt::s32 tmp_58;
    # 19 "<preprocessor>"
    hmpprt::s32 end_15;
    # 19 "<preprocessor>"
    hmpprt::s32 i_6;
    # 188 "x_solve.c"
    # 188 "x_solve.c"
    # 188 "x_solve.c"
    for (i_6 = 0, end_15 = gp0 - 3 ; i_6 <= end_15 ; i_6 = i_6 + 1)
    {
     # 189 "x_solve.c"
     double fac1_1;
     # 189 "x_solve.c"
     hmpprt::s32 i1_1;
     # 189 "x_solve.c"
     hmpprt::s32 i2_2;
     # 189 "x_solve.c"
     hmpprt::s32 first_gang_iter_6;
     # 189 "x_solve.c"
     hmpprt::s32 last_gang_iter_6;
     # 189 "x_solve.c"
     hmpprt::s32 first_gang_iter_7;
     # 189 "x_solve.c"
     hmpprt::s32 last_gang_iter_7;
     # 189 "x_solve.c"
     hmpprt::s32 first_gang_iter_8;
     # 189 "x_solve.c"
     hmpprt::s32 last_gang_iter_8;
     # 189 "x_solve.c"
     double tmp_52;
     # 189 "x_solve.c"
     double tmp_53;
     # 189 "x_solve.c"
     double tmp_54;
     # 189 "x_solve.c"
     double tmp_55;
     # 189 "x_solve.c"
     double tmp_56;
     # 189 "x_solve.c"
     double tmp_57;
     # 189 "x_solve.c"
     i1_1 = i_6 + 1;
     # 190 "x_solve.c"
     i2_2 = i_6 + 2;
     # 191 "x_solve.c"
     fac1_1 = (double) 1.0 / *(lhsX_1 + (j_7 + 1) + 37uLL * (i_6 + 37uLL * (k_1 + 73)));
     # 14 "<preprocessor>"
     tmp_52 = fac1_1 * *(lhsX_1 + (j_7 + 1) + 37uLL * (i_6 + 37uLL * (k_1 + 109)));
     # 14 "<preprocessor>"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 14 "<preprocessor>"
      *(lhsX_1 + (j_7 + 1) + 37uLL * (i_6 + 37uLL * (k_1 + 109))) = tmp_52;
     }
     # 14 "<preprocessor>"
     (hmpprt::gr_barrier());
     # 14 "<preprocessor>"
     tmp_53 = fac1_1 * *(lhsX_1 + (j_7 + 1) + 37uLL * (i_6 + 37uLL * (k_1 + 145)));
     # 14 "<preprocessor>"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 14 "<preprocessor>"
      *(lhsX_1 + (j_7 + 1) + 37uLL * (i_6 + 37uLL * (k_1 + 145))) = tmp_53;
     }
     # 14 "<preprocessor>"
     (hmpprt::gr_barrier());
     # 14 "<preprocessor>"
     first_gang_iter_6 = (hmpprt::gr_gbidx()) * 8;
     # 14 "<preprocessor>"
     last_gang_iter_6 = ((first_gang_iter_6 + 7) < 2 ? (first_gang_iter_6 + 7) : 2);
     # 14 "<preprocessor>"
     hmpprt::s32 m_2;
     # 194 "x_solve.c"
     # 194 "x_solve.c"
     for (m_2 = first_gang_iter_6 + (hmpprt::gr_btidy()) ; m_2 <= last_gang_iter_6 ; m_2 = m_2 + (hmpprt::gr_btnumy()))
     {
      # 195 "x_solve.c"
      *(rhsX_1 + (j_7 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i_6 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_2))) = fac1_1 * *(rhsX_1 + (j_7 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i_6 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_2)));
     }
     # 14 "<preprocessor>"
     # 14 "<preprocessor>"
     tmp_54 = *(lhsX_1 + (j_7 + 1) + 37uLL * (i1_1 + 37uLL * (k_1 + 73))) - *(lhsX_1 + (j_7 + 1) + 37uLL * (i1_1 + 37uLL * (k_1 + 37))) * *(lhsX_1 + (j_7 + 1) + 37uLL * (i_6 + 37uLL * (k_1 + 109)));
     # 14 "<preprocessor>"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 14 "<preprocessor>"
      *(lhsX_1 + (j_7 + 1) + 37uLL * (i1_1 + 37uLL * (k_1 + 73))) = tmp_54;
     }
     # 14 "<preprocessor>"
     (hmpprt::gr_barrier());
     # 14 "<preprocessor>"
     tmp_55 = *(lhsX_1 + (j_7 + 1) + 37uLL * (i1_1 + 37uLL * (k_1 + 109))) - *(lhsX_1 + (j_7 + 1) + 37uLL * (i1_1 + 37uLL * (k_1 + 37))) * *(lhsX_1 + (j_7 + 1) + 37uLL * (i_6 + 37uLL * (k_1 + 145)));
     # 14 "<preprocessor>"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 14 "<preprocessor>"
      *(lhsX_1 + (j_7 + 1) + 37uLL * (i1_1 + 37uLL * (k_1 + 109))) = tmp_55;
     }
     # 14 "<preprocessor>"
     (hmpprt::gr_barrier());
     # 14 "<preprocessor>"
     first_gang_iter_7 = (hmpprt::gr_gbidx()) * 8;
     # 14 "<preprocessor>"
     last_gang_iter_7 = ((first_gang_iter_7 + 7) < 2 ? (first_gang_iter_7 + 7) : 2);
     # 14 "<preprocessor>"
     hmpprt::s32 m_3;
     # 199 "x_solve.c"
     # 199 "x_solve.c"
     for (m_3 = first_gang_iter_7 + (hmpprt::gr_btidy()) ; m_3 <= last_gang_iter_7 ; m_3 = m_3 + (hmpprt::gr_btnumy()))
     {
      # 200 "x_solve.c"
      *(rhsX_1 + (j_7 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_1 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_3))) = *(rhsX_1 + (j_7 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_1 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_3))) - *(lhsX_1 + (j_7 + 1) + 37uLL * (i1_1 + 37uLL * (k_1 + 37))) * *(rhsX_1 + (j_7 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i_6 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_3)));
     }
     # 14 "<preprocessor>"
     # 14 "<preprocessor>"
     tmp_56 = *(lhsX_1 + (j_7 + 1) + 37uLL * (i2_2 + 37uLL * (k_1 + 37))) - *(lhsX_1 + (j_7 + 1) + 37uLL * (i2_2 + 37uLL * (k_1 + 1))) * *(lhsX_1 + (j_7 + 1) + 37uLL * (i_6 + 37uLL * (k_1 + 109)));
     # 14 "<preprocessor>"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 14 "<preprocessor>"
      *(lhsX_1 + (j_7 + 1) + 37uLL * (i2_2 + 37uLL * (k_1 + 37))) = tmp_56;
     }
     # 14 "<preprocessor>"
     (hmpprt::gr_barrier());
     # 14 "<preprocessor>"
     tmp_57 = *(lhsX_1 + (j_7 + 1) + 37uLL * (i2_2 + 37uLL * (k_1 + 73))) - *(lhsX_1 + (j_7 + 1) + 37uLL * (i2_2 + 37uLL * (k_1 + 1))) * *(lhsX_1 + (j_7 + 1) + 37uLL * (i_6 + 37uLL * (k_1 + 145)));
     # 14 "<preprocessor>"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 14 "<preprocessor>"
      *(lhsX_1 + (j_7 + 1) + 37uLL * (i2_2 + 37uLL * (k_1 + 73))) = tmp_57;
     }
     # 14 "<preprocessor>"
     (hmpprt::gr_barrier());
     # 14 "<preprocessor>"
     first_gang_iter_8 = (hmpprt::gr_gbidx()) * 8;
     # 14 "<preprocessor>"
     last_gang_iter_8 = ((first_gang_iter_8 + 7) < 2 ? (first_gang_iter_8 + 7) : 2);
     # 14 "<preprocessor>"
     hmpprt::s32 m_4;
     # 204 "x_solve.c"
     # 204 "x_solve.c"
     for (m_4 = first_gang_iter_8 + (hmpprt::gr_btidy()) ; m_4 <= last_gang_iter_8 ; m_4 = m_4 + (hmpprt::gr_btnumy()))
     {
      # 205 "x_solve.c"
      *(rhsX_1 + (j_7 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i2_2 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_4))) = *(rhsX_1 + (j_7 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i2_2 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_4))) - *(lhsX_1 + (j_7 + 1) + 37uLL * (i2_2 + 37uLL * (k_1 + 1))) * *(rhsX_1 + (j_7 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i_6 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_4)));
     }
     # 18 "<preprocessor>"
    }
    # 18 "<preprocessor>"
    # 18 "<preprocessor>"
    tmp_58 = gp0 +  -2;
    # 18 "<preprocessor>"
    if ((hmpprt::gr_btidy()) == 0)
    {
     # 18 "<preprocessor>"
     *i_9 = tmp_58;
    }
    # 18 "<preprocessor>"
    (hmpprt::gr_barrier());
   }
   else
   {
    # 18 "<preprocessor>"
    if ((hmpprt::gr_btidy()) == 0)
    {
     # 18 "<preprocessor>"
     *i_9 = 0;
    }
    # 18 "<preprocessor>"
    (hmpprt::gr_barrier());
   }
  }
  # 18 "<preprocessor>"
  # 18 "<preprocessor>"
  hmpprt::s32 end_20;
  # 18 "<preprocessor>"
  hmpprt::s32 j_8;
  # 215 "x_solve.c"
  # 215 "x_solve.c"
  # 215 "x_solve.c"
  for (j_8 = 0, end_20 = ny2_1 - 1 ; j_8 <= end_20 ; j_8 = j_8 + 1)
  {
   # 217 "x_solve.c"
   double fac1_2;
   # 217 "x_solve.c"
   hmpprt::s32 i1_3;
   # 217 "x_solve.c"
   double fac2_1;
   # 217 "x_solve.c"
   hmpprt::s32 first_gang_iter_9;
   # 217 "x_solve.c"
   hmpprt::s32 last_gang_iter_9;
   # 217 "x_solve.c"
   hmpprt::s32 first_gang_iter_10;
   # 217 "x_solve.c"
   hmpprt::s32 last_gang_iter_10;
   # 217 "x_solve.c"
   hmpprt::s32 first_gang_iter_11;
   # 217 "x_solve.c"
   hmpprt::s32 last_gang_iter_11;
   # 217 "x_solve.c"
   hmpprt::s32 tmp_60;
   # 217 "x_solve.c"
   double tmp_61;
   # 217 "x_solve.c"
   double tmp_62;
   # 217 "x_solve.c"
   double tmp_63;
   # 217 "x_solve.c"
   double tmp_64;
   # 217 "x_solve.c"
   tmp_60 = gp0 - 2;
   # 217 "x_solve.c"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 217 "x_solve.c"
    *i_9 = tmp_60;
   }
   # 217 "x_solve.c"
   (hmpprt::gr_barrier());
   # 217 "x_solve.c"
   i1_3 = gp0 - 1;
   # 218 "x_solve.c"
   fac1_2 = (double) 1.0 / *(lhsX_1 + (j_8 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 73)));
   # 14 "<preprocessor>"
   tmp_61 = fac1_2 * *(lhsX_1 + (j_8 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 109)));
   # 14 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 14 "<preprocessor>"
    *(lhsX_1 + (j_8 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 109))) = tmp_61;
   }
   # 14 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 14 "<preprocessor>"
   tmp_62 = fac1_2 * *(lhsX_1 + (j_8 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 145)));
   # 14 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 14 "<preprocessor>"
    *(lhsX_1 + (j_8 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 145))) = tmp_62;
   }
   # 14 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 14 "<preprocessor>"
   first_gang_iter_9 = (hmpprt::gr_gbidx()) * 8;
   # 14 "<preprocessor>"
   last_gang_iter_9 = ((first_gang_iter_9 + 7) < 2 ? (first_gang_iter_9 + 7) : 2);
   # 14 "<preprocessor>"
   hmpprt::s32 m_5;
   # 221 "x_solve.c"
   # 221 "x_solve.c"
   for (m_5 = first_gang_iter_9 + (hmpprt::gr_btidy()) ; m_5 <= last_gang_iter_9 ; m_5 = m_5 + (hmpprt::gr_btnumy()))
   {
    # 222 "x_solve.c"
    *(rhsX_1 + (j_8 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (*i_9 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_5))) = fac1_2 * *(rhsX_1 + (j_8 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (*i_9 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_5)));
   }
   # 14 "<preprocessor>"
   # 14 "<preprocessor>"
   tmp_63 = *(lhsX_1 + (j_8 + 1) + 37uLL * (i1_3 + 37uLL * (k_1 + 73))) - *(lhsX_1 + (j_8 + 1) + 37uLL * (i1_3 + 37uLL * (k_1 + 37))) * *(lhsX_1 + (j_8 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 109)));
   # 14 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 14 "<preprocessor>"
    *(lhsX_1 + (j_8 + 1) + 37uLL * (i1_3 + 37uLL * (k_1 + 73))) = tmp_63;
   }
   # 14 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 14 "<preprocessor>"
   tmp_64 = *(lhsX_1 + (j_8 + 1) + 37uLL * (i1_3 + 37uLL * (k_1 + 109))) - *(lhsX_1 + (j_8 + 1) + 37uLL * (i1_3 + 37uLL * (k_1 + 37))) * *(lhsX_1 + (j_8 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 145)));
   # 14 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 14 "<preprocessor>"
    *(lhsX_1 + (j_8 + 1) + 37uLL * (i1_3 + 37uLL * (k_1 + 109))) = tmp_64;
   }
   # 14 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 14 "<preprocessor>"
   first_gang_iter_10 = (hmpprt::gr_gbidx()) * 8;
   # 14 "<preprocessor>"
   last_gang_iter_10 = ((first_gang_iter_10 + 7) < 2 ? (first_gang_iter_10 + 7) : 2);
   # 14 "<preprocessor>"
   hmpprt::s32 m_6;
   # 226 "x_solve.c"
   # 226 "x_solve.c"
   for (m_6 = first_gang_iter_10 + (hmpprt::gr_btidy()) ; m_6 <= last_gang_iter_10 ; m_6 = m_6 + (hmpprt::gr_btnumy()))
   {
    # 227 "x_solve.c"
    *(rhsX_1 + (j_8 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_3 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_6))) = *(rhsX_1 + (j_8 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_3 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_6))) - *(lhsX_1 + (j_8 + 1) + 37uLL * (i1_3 + 37uLL * (k_1 + 37))) * *(rhsX_1 + (j_8 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (*i_9 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_6)));
   }
   # 233 "x_solve.c"
   # 233 "x_solve.c"
   fac2_1 = (double) 1.0 / *(lhsX_1 + (j_8 + 1) + 37uLL * (i1_3 + 37uLL * (k_1 + 73)));
   # 14 "<preprocessor>"
   first_gang_iter_11 = (hmpprt::gr_gbidx()) * 8;
   # 14 "<preprocessor>"
   last_gang_iter_11 = ((first_gang_iter_11 + 7) < 2 ? (first_gang_iter_11 + 7) : 2);
   # 14 "<preprocessor>"
   hmpprt::s32 m_7;
   # 234 "x_solve.c"
   # 234 "x_solve.c"
   for (m_7 = first_gang_iter_11 + (hmpprt::gr_btidy()) ; m_7 <= last_gang_iter_11 ; m_7 = m_7 + (hmpprt::gr_btnumy()))
   {
    # 235 "x_solve.c"
    *(rhsX_1 + (j_8 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_3 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_7))) = fac2_1 * *(rhsX_1 + (j_8 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_3 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_7)));
   }
   # 18 "<preprocessor>"
  }
  # 18 "<preprocessor>"
  # 18 "<preprocessor>"
  hmpprt::s32 end_22;
  # 18 "<preprocessor>"
  hmpprt::s32 j_9;
  # 242 "x_solve.c"
  # 242 "x_solve.c"
  # 242 "x_solve.c"
  for (j_9 = 0, end_22 = ny2_1 - 1 ; j_9 <= end_22 ; j_9 = j_9 + 1)
  {
   # 18 "<preprocessor>"
   if (0 <= gp0 - 3)
   {
    # 19 "<preprocessor>"
    hmpprt::s32 tmp_83;
    # 19 "<preprocessor>"
    hmpprt::s32 end_21;
    # 19 "<preprocessor>"
    hmpprt::s32 i_7;
    # 243 "x_solve.c"
    # 243 "x_solve.c"
    # 243 "x_solve.c"
    for (i_7 = 0, end_21 = gp0 - 3 ; i_7 <= end_21 ; i_7 = i_7 + 1)
    {
     # 244 "x_solve.c"
     double fac1_3;
     # 244 "x_solve.c"
     hmpprt::s32 i1_4;
     # 244 "x_solve.c"
     hmpprt::s32 i2_3;
     # 244 "x_solve.c"
     double fac1_4;
     # 244 "x_solve.c"
     double tmp_65;
     # 244 "x_solve.c"
     double tmp_66;
     # 244 "x_solve.c"
     double tmp_67;
     # 244 "x_solve.c"
     double tmp_68;
     # 244 "x_solve.c"
     double tmp_69;
     # 244 "x_solve.c"
     double tmp_70;
     # 244 "x_solve.c"
     double tmp_71;
     # 244 "x_solve.c"
     double tmp_72;
     # 244 "x_solve.c"
     double tmp_73;
     # 244 "x_solve.c"
     double tmp_74;
     # 244 "x_solve.c"
     double tmp_75;
     # 244 "x_solve.c"
     double tmp_76;
     # 244 "x_solve.c"
     double tmp_77;
     # 244 "x_solve.c"
     double tmp_78;
     # 244 "x_solve.c"
     double tmp_79;
     # 244 "x_solve.c"
     double tmp_80;
     # 244 "x_solve.c"
     double tmp_81;
     # 244 "x_solve.c"
     double tmp_82;
     # 244 "x_solve.c"
     i1_4 = i_7 + 1;
     # 245 "x_solve.c"
     i2_3 = i_7 + 2;
     # 248 "x_solve.c"
     fac1_3 = (double) 1.0 / *(lhspX + (j_9 + 1) + 37uLL * (i_7 + 37uLL * (k_1 + 73)));
     # 260 "x_solve.c"
     tmp_65 = fac1_3 * *(lhspX + (j_9 + 1) + 37uLL * (i_7 + 37uLL * (k_1 + 109)));
     # 260 "x_solve.c"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 260 "x_solve.c"
      *(lhspX + (j_9 + 1) + 37uLL * (i_7 + 37uLL * (k_1 + 109))) = tmp_65;
     }
     # 260 "x_solve.c"
     (hmpprt::gr_barrier());
     # 260 "x_solve.c"
     tmp_66 = fac1_3 * *(lhspX + (j_9 + 1) + 37uLL * (i_7 + 37uLL * (k_1 + 145)));
     # 260 "x_solve.c"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 260 "x_solve.c"
      *(lhspX + (j_9 + 1) + 37uLL * (i_7 + 37uLL * (k_1 + 145))) = tmp_66;
     }
     # 260 "x_solve.c"
     (hmpprt::gr_barrier());
     # 260 "x_solve.c"
     tmp_67 = fac1_3 * *(rhsX_1 + (j_9 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i_7 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3)));
     # 260 "x_solve.c"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 260 "x_solve.c"
      *(rhsX_1 + (j_9 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i_7 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3))) = tmp_67;
     }
     # 260 "x_solve.c"
     (hmpprt::gr_barrier());
     # 260 "x_solve.c"
     tmp_68 = *(lhspX + (j_9 + 1) + 37uLL * (i1_4 + 37uLL * (k_1 + 73))) - *(lhspX + (j_9 + 1) + 37uLL * (i1_4 + 37uLL * (k_1 + 37))) * *(lhspX + (j_9 + 1) + 37uLL * (i_7 + 37uLL * (k_1 + 109)));
     # 260 "x_solve.c"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 260 "x_solve.c"
      *(lhspX + (j_9 + 1) + 37uLL * (i1_4 + 37uLL * (k_1 + 73))) = tmp_68;
     }
     # 260 "x_solve.c"
     (hmpprt::gr_barrier());
     # 260 "x_solve.c"
     tmp_69 = *(lhspX + (j_9 + 1) + 37uLL * (i1_4 + 37uLL * (k_1 + 109))) - *(lhspX + (j_9 + 1) + 37uLL * (i1_4 + 37uLL * (k_1 + 37))) * *(lhspX + (j_9 + 1) + 37uLL * (i_7 + 37uLL * (k_1 + 145)));
     # 260 "x_solve.c"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 260 "x_solve.c"
      *(lhspX + (j_9 + 1) + 37uLL * (i1_4 + 37uLL * (k_1 + 109))) = tmp_69;
     }
     # 260 "x_solve.c"
     (hmpprt::gr_barrier());
     # 260 "x_solve.c"
     tmp_70 = *(rhsX_1 + (j_9 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_4 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3))) - *(lhspX + (j_9 + 1) + 37uLL * (i1_4 + 37uLL * (k_1 + 37))) * *(rhsX_1 + (j_9 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i_7 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3)));
     # 260 "x_solve.c"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 260 "x_solve.c"
      *(rhsX_1 + (j_9 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_4 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3))) = tmp_70;
     }
     # 260 "x_solve.c"
     (hmpprt::gr_barrier());
     # 260 "x_solve.c"
     tmp_71 = *(lhspX + (j_9 + 1) + 37uLL * (i2_3 + 37uLL * (k_1 + 37))) - *(lhspX + (j_9 + 1) + 37uLL * (i2_3 + 37uLL * (k_1 + 1))) * *(lhspX + (j_9 + 1) + 37uLL * (i_7 + 37uLL * (k_1 + 109)));
     # 260 "x_solve.c"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 260 "x_solve.c"
      *(lhspX + (j_9 + 1) + 37uLL * (i2_3 + 37uLL * (k_1 + 37))) = tmp_71;
     }
     # 260 "x_solve.c"
     (hmpprt::gr_barrier());
     # 260 "x_solve.c"
     tmp_72 = *(lhspX + (j_9 + 1) + 37uLL * (i2_3 + 37uLL * (k_1 + 73))) - *(lhspX + (j_9 + 1) + 37uLL * (i2_3 + 37uLL * (k_1 + 1))) * *(lhspX + (j_9 + 1) + 37uLL * (i_7 + 37uLL * (k_1 + 145)));
     # 260 "x_solve.c"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 260 "x_solve.c"
      *(lhspX + (j_9 + 1) + 37uLL * (i2_3 + 37uLL * (k_1 + 73))) = tmp_72;
     }
     # 260 "x_solve.c"
     (hmpprt::gr_barrier());
     # 260 "x_solve.c"
     tmp_73 = *(rhsX_1 + (j_9 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i2_3 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3))) - *(lhspX + (j_9 + 1) + 37uLL * (i2_3 + 37uLL * (k_1 + 1))) * *(rhsX_1 + (j_9 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i_7 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3)));
     # 260 "x_solve.c"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 260 "x_solve.c"
      *(rhsX_1 + (j_9 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i2_3 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3))) = tmp_73;
     }
     # 260 "x_solve.c"
     (hmpprt::gr_barrier());
     # 260 "x_solve.c"
     fac1_4 = (double) 1.0 / *(lhsmX_1 + (j_9 + 1) + 37uLL * (i_7 + 37uLL * (k_1 + 73)));
     # 18 "<preprocessor>"
     tmp_74 = fac1_4 * *(lhsmX_1 + (j_9 + 1) + 37uLL * (i_7 + 37uLL * (k_1 + 109)));
     # 18 "<preprocessor>"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 18 "<preprocessor>"
      *(lhsmX_1 + (j_9 + 1) + 37uLL * (i_7 + 37uLL * (k_1 + 109))) = tmp_74;
     }
     # 18 "<preprocessor>"
     (hmpprt::gr_barrier());
     # 18 "<preprocessor>"
     tmp_75 = fac1_4 * *(lhsmX_1 + (j_9 + 1) + 37uLL * (i_7 + 37uLL * (k_1 + 145)));
     # 18 "<preprocessor>"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 18 "<preprocessor>"
      *(lhsmX_1 + (j_9 + 1) + 37uLL * (i_7 + 37uLL * (k_1 + 145))) = tmp_75;
     }
     # 18 "<preprocessor>"
     (hmpprt::gr_barrier());
     # 18 "<preprocessor>"
     tmp_76 = fac1_4 * *(rhsX_1 + (j_9 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i_7 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4)));
     # 18 "<preprocessor>"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 18 "<preprocessor>"
      *(rhsX_1 + (j_9 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i_7 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4))) = tmp_76;
     }
     # 18 "<preprocessor>"
     (hmpprt::gr_barrier());
     # 18 "<preprocessor>"
     tmp_77 = *(lhsmX_1 + (j_9 + 1) + 37uLL * (i1_4 + 37uLL * (k_1 + 73))) - *(lhsmX_1 + (j_9 + 1) + 37uLL * (i1_4 + 37uLL * (k_1 + 37))) * *(lhsmX_1 + (j_9 + 1) + 37uLL * (i_7 + 37uLL * (k_1 + 109)));
     # 18 "<preprocessor>"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 18 "<preprocessor>"
      *(lhsmX_1 + (j_9 + 1) + 37uLL * (i1_4 + 37uLL * (k_1 + 73))) = tmp_77;
     }
     # 18 "<preprocessor>"
     (hmpprt::gr_barrier());
     # 18 "<preprocessor>"
     tmp_78 = *(lhsmX_1 + (j_9 + 1) + 37uLL * (i1_4 + 37uLL * (k_1 + 109))) - *(lhsmX_1 + (j_9 + 1) + 37uLL * (i1_4 + 37uLL * (k_1 + 37))) * *(lhsmX_1 + (j_9 + 1) + 37uLL * (i_7 + 37uLL * (k_1 + 145)));
     # 18 "<preprocessor>"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 18 "<preprocessor>"
      *(lhsmX_1 + (j_9 + 1) + 37uLL * (i1_4 + 37uLL * (k_1 + 109))) = tmp_78;
     }
     # 18 "<preprocessor>"
     (hmpprt::gr_barrier());
     # 18 "<preprocessor>"
     tmp_79 = *(rhsX_1 + (j_9 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_4 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4))) - *(lhsmX_1 + (j_9 + 1) + 37uLL * (i1_4 + 37uLL * (k_1 + 37))) * *(rhsX_1 + (j_9 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i_7 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4)));
     # 18 "<preprocessor>"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 18 "<preprocessor>"
      *(rhsX_1 + (j_9 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_4 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4))) = tmp_79;
     }
     # 18 "<preprocessor>"
     (hmpprt::gr_barrier());
     # 18 "<preprocessor>"
     tmp_80 = *(lhsmX_1 + (j_9 + 1) + 37uLL * (i2_3 + 37uLL * (k_1 + 37))) - *(lhsmX_1 + (j_9 + 1) + 37uLL * (i2_3 + 37uLL * (k_1 + 1))) * *(lhsmX_1 + (j_9 + 1) + 37uLL * (i_7 + 37uLL * (k_1 + 109)));
     # 18 "<preprocessor>"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 18 "<preprocessor>"
      *(lhsmX_1 + (j_9 + 1) + 37uLL * (i2_3 + 37uLL * (k_1 + 37))) = tmp_80;
     }
     # 18 "<preprocessor>"
     (hmpprt::gr_barrier());
     # 18 "<preprocessor>"
     tmp_81 = *(lhsmX_1 + (j_9 + 1) + 37uLL * (i2_3 + 37uLL * (k_1 + 73))) - *(lhsmX_1 + (j_9 + 1) + 37uLL * (i2_3 + 37uLL * (k_1 + 1))) * *(lhsmX_1 + (j_9 + 1) + 37uLL * (i_7 + 37uLL * (k_1 + 145)));
     # 18 "<preprocessor>"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 18 "<preprocessor>"
      *(lhsmX_1 + (j_9 + 1) + 37uLL * (i2_3 + 37uLL * (k_1 + 73))) = tmp_81;
     }
     # 18 "<preprocessor>"
     (hmpprt::gr_barrier());
     # 18 "<preprocessor>"
     tmp_82 = *(rhsX_1 + (j_9 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i2_3 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4))) - *(lhsmX_1 + (j_9 + 1) + 37uLL * (i2_3 + 37uLL * (k_1 + 1))) * *(rhsX_1 + (j_9 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i_7 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4)));
     # 18 "<preprocessor>"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 18 "<preprocessor>"
      *(rhsX_1 + (j_9 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i2_3 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4))) = tmp_82;
     }
     # 18 "<preprocessor>"
     (hmpprt::gr_barrier());
    }
    # 18 "<preprocessor>"
    # 18 "<preprocessor>"
    tmp_83 = gp0 +  -2;
    # 18 "<preprocessor>"
    if ((hmpprt::gr_btidy()) == 0)
    {
     # 18 "<preprocessor>"
     *i_9 = tmp_83;
    }
    # 18 "<preprocessor>"
    (hmpprt::gr_barrier());
   }
   else
   {
    # 18 "<preprocessor>"
    if ((hmpprt::gr_btidy()) == 0)
    {
     # 18 "<preprocessor>"
     *i_9 = 0;
    }
    # 18 "<preprocessor>"
    (hmpprt::gr_barrier());
   }
  }
  # 18 "<preprocessor>"
  # 18 "<preprocessor>"
  hmpprt::s32 end_23;
  # 18 "<preprocessor>"
  hmpprt::s32 j_10;
  # 276 "x_solve.c"
  # 276 "x_solve.c"
  # 276 "x_solve.c"
  for (j_10 = 0, end_23 = ny2_1 - 1 ; j_10 <= end_23 ; j_10 = j_10 + 1)
  {
   # 278 "x_solve.c"
   double fac1_5;
   # 278 "x_solve.c"
   hmpprt::s32 i1_5;
   # 278 "x_solve.c"
   double fac1_6;
   # 278 "x_solve.c"
   hmpprt::s32 tmp_85;
   # 278 "x_solve.c"
   double tmp_86;
   # 278 "x_solve.c"
   double tmp_87;
   # 278 "x_solve.c"
   double tmp_88;
   # 278 "x_solve.c"
   double tmp_89;
   # 278 "x_solve.c"
   double tmp_90;
   # 278 "x_solve.c"
   double tmp_91;
   # 278 "x_solve.c"
   double tmp_92;
   # 278 "x_solve.c"
   double tmp_93;
   # 278 "x_solve.c"
   double tmp_94;
   # 278 "x_solve.c"
   double tmp_95;
   # 278 "x_solve.c"
   double tmp_96;
   # 278 "x_solve.c"
   double tmp_97;
   # 278 "x_solve.c"
   double tmp_98;
   # 278 "x_solve.c"
   double tmp_99;
   # 278 "x_solve.c"
   tmp_85 = gp0 - 2;
   # 278 "x_solve.c"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 278 "x_solve.c"
    *i_9 = tmp_85;
   }
   # 278 "x_solve.c"
   (hmpprt::gr_barrier());
   # 278 "x_solve.c"
   i1_5 = gp0 - 1;
   # 281 "x_solve.c"
   fac1_5 = (double) 1.0 / *(lhspX + (j_10 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 73)));
   # 290 "x_solve.c"
   tmp_86 = fac1_5 * *(lhspX + (j_10 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 109)));
   # 290 "x_solve.c"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 290 "x_solve.c"
    *(lhspX + (j_10 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 109))) = tmp_86;
   }
   # 290 "x_solve.c"
   (hmpprt::gr_barrier());
   # 290 "x_solve.c"
   tmp_87 = fac1_5 * *(lhspX + (j_10 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 145)));
   # 290 "x_solve.c"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 290 "x_solve.c"
    *(lhspX + (j_10 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 145))) = tmp_87;
   }
   # 290 "x_solve.c"
   (hmpprt::gr_barrier());
   # 290 "x_solve.c"
   tmp_88 = fac1_5 * *(rhsX_1 + (j_10 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (*i_9 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3)));
   # 290 "x_solve.c"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 290 "x_solve.c"
    *(rhsX_1 + (j_10 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (*i_9 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3))) = tmp_88;
   }
   # 290 "x_solve.c"
   (hmpprt::gr_barrier());
   # 290 "x_solve.c"
   tmp_89 = *(lhspX + (j_10 + 1) + 37uLL * (i1_5 + 37uLL * (k_1 + 73))) - *(lhspX + (j_10 + 1) + 37uLL * (i1_5 + 37uLL * (k_1 + 37))) * *(lhspX + (j_10 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 109)));
   # 290 "x_solve.c"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 290 "x_solve.c"
    *(lhspX + (j_10 + 1) + 37uLL * (i1_5 + 37uLL * (k_1 + 73))) = tmp_89;
   }
   # 290 "x_solve.c"
   (hmpprt::gr_barrier());
   # 290 "x_solve.c"
   tmp_90 = *(lhspX + (j_10 + 1) + 37uLL * (i1_5 + 37uLL * (k_1 + 109))) - *(lhspX + (j_10 + 1) + 37uLL * (i1_5 + 37uLL * (k_1 + 37))) * *(lhspX + (j_10 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 145)));
   # 290 "x_solve.c"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 290 "x_solve.c"
    *(lhspX + (j_10 + 1) + 37uLL * (i1_5 + 37uLL * (k_1 + 109))) = tmp_90;
   }
   # 290 "x_solve.c"
   (hmpprt::gr_barrier());
   # 290 "x_solve.c"
   tmp_91 = *(rhsX_1 + (j_10 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_5 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3))) - *(lhspX + (j_10 + 1) + 37uLL * (i1_5 + 37uLL * (k_1 + 37))) * *(rhsX_1 + (j_10 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (*i_9 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3)));
   # 290 "x_solve.c"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 290 "x_solve.c"
    *(rhsX_1 + (j_10 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_5 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3))) = tmp_91;
   }
   # 290 "x_solve.c"
   (hmpprt::gr_barrier());
   # 290 "x_solve.c"
   fac1_6 = (double) 1.0 / *(lhsmX_1 + (j_10 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 73)));
   # 18 "<preprocessor>"
   tmp_92 = fac1_6 * *(lhsmX_1 + (j_10 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 109)));
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(lhsmX_1 + (j_10 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 109))) = tmp_92;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_93 = fac1_6 * *(lhsmX_1 + (j_10 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 145)));
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(lhsmX_1 + (j_10 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 145))) = tmp_93;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_94 = fac1_6 * *(rhsX_1 + (j_10 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (*i_9 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4)));
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(rhsX_1 + (j_10 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (*i_9 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4))) = tmp_94;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_95 = *(lhsmX_1 + (j_10 + 1) + 37uLL * (i1_5 + 37uLL * (k_1 + 73))) - *(lhsmX_1 + (j_10 + 1) + 37uLL * (i1_5 + 37uLL * (k_1 + 37))) * *(lhsmX_1 + (j_10 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 109)));
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(lhsmX_1 + (j_10 + 1) + 37uLL * (i1_5 + 37uLL * (k_1 + 73))) = tmp_95;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_96 = *(lhsmX_1 + (j_10 + 1) + 37uLL * (i1_5 + 37uLL * (k_1 + 109))) - *(lhsmX_1 + (j_10 + 1) + 37uLL * (i1_5 + 37uLL * (k_1 + 37))) * *(lhsmX_1 + (j_10 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 145)));
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(lhsmX_1 + (j_10 + 1) + 37uLL * (i1_5 + 37uLL * (k_1 + 109))) = tmp_96;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_97 = *(rhsX_1 + (j_10 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_5 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4))) - *(lhsmX_1 + (j_10 + 1) + 37uLL * (i1_5 + 37uLL * (k_1 + 37))) * *(rhsX_1 + (j_10 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (*i_9 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4)));
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(rhsX_1 + (j_10 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_5 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4))) = tmp_97;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_98 = *(rhsX_1 + (j_10 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_5 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3))) / *(lhspX + (j_10 + 1) + 37uLL * (i1_5 + 37uLL * (k_1 + 73)));
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(rhsX_1 + (j_10 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_5 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3))) = tmp_98;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 18 "<preprocessor>"
   tmp_99 = *(rhsX_1 + (j_10 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_5 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4))) / *(lhsmX_1 + (j_10 + 1) + 37uLL * (i1_5 + 37uLL * (k_1 + 73)));
   # 18 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 18 "<preprocessor>"
    *(rhsX_1 + (j_10 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_5 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4))) = tmp_99;
   }
   # 18 "<preprocessor>"
   (hmpprt::gr_barrier());
  }
  # 18 "<preprocessor>"
  # 18 "<preprocessor>"
  hmpprt::s32 end_25;
  # 18 "<preprocessor>"
  hmpprt::s32 j_11;
  # 308 "x_solve.c"
  # 308 "x_solve.c"
  # 308 "x_solve.c"
  for (j_11 = 0, end_25 = ny2_1 - 1 ; j_11 <= end_25 ; j_11 = j_11 + 1)
  {
   # 310 "x_solve.c"
   hmpprt::s32 i1_6;
   # 310 "x_solve.c"
   hmpprt::s32 first_gang_iter_12;
   # 310 "x_solve.c"
   hmpprt::s32 last_gang_iter_12;
   # 310 "x_solve.c"
   hmpprt::s32 tmp_100;
   # 310 "x_solve.c"
   double tmp_101;
   # 310 "x_solve.c"
   double tmp_102;
   # 310 "x_solve.c"
   tmp_100 = gp0 - 2;
   # 310 "x_solve.c"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 310 "x_solve.c"
    *i_9 = tmp_100;
   }
   # 310 "x_solve.c"
   (hmpprt::gr_barrier());
   # 310 "x_solve.c"
   i1_6 = gp0 - 1;
   # 14 "<preprocessor>"
   first_gang_iter_12 = (hmpprt::gr_gbidx()) * 8;
   # 14 "<preprocessor>"
   last_gang_iter_12 = ((first_gang_iter_12 + 7) < 2 ? (first_gang_iter_12 + 7) : 2);
   # 14 "<preprocessor>"
   hmpprt::s32 m_8;
   # 311 "x_solve.c"
   # 311 "x_solve.c"
   for (m_8 = first_gang_iter_12 + (hmpprt::gr_btidy()) ; m_8 <= last_gang_iter_12 ; m_8 = m_8 + (hmpprt::gr_btnumy()))
   {
    # 312 "x_solve.c"
    *(rhsX_1 + (j_11 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (*i_9 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_8))) = *(rhsX_1 + (j_11 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (*i_9 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_8))) - *(lhsX_1 + (j_11 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 109))) * *(rhsX_1 + (j_11 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_6 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_8)));
   }
   # 5 "<preprocessor>"
   # 5 "<preprocessor>"
   tmp_101 = *(rhsX_1 + (j_11 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (*i_9 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3))) - *(lhspX + (j_11 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 109))) * *(rhsX_1 + (j_11 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_6 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3)));
   # 5 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 5 "<preprocessor>"
    *(rhsX_1 + (j_11 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (*i_9 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3))) = tmp_101;
   }
   # 5 "<preprocessor>"
   (hmpprt::gr_barrier());
   # 5 "<preprocessor>"
   tmp_102 = *(rhsX_1 + (j_11 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (*i_9 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4))) - *(lhsmX_1 + (j_11 + 1) + 37uLL * (*i_9 + 37uLL * (k_1 + 109))) * *(rhsX_1 + (j_11 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_6 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4)));
   # 5 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 5 "<preprocessor>"
    *(rhsX_1 + (j_11 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (*i_9 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4))) = tmp_102;
   }
   # 5 "<preprocessor>"
   (hmpprt::gr_barrier());
  }
  # 5 "<preprocessor>"
  # 5 "<preprocessor>"
  if (1 <= ny2_1)
  {
   # 18 "<preprocessor>"
   hmpprt::s32 tmp_107;
   # 18 "<preprocessor>"
   hmpprt::s32 end_28;
   # 18 "<preprocessor>"
   hmpprt::s32 j_12;
   # 322 "x_solve.c"
   # 322 "x_solve.c"
   # 322 "x_solve.c"
   for (j_12 = 0, end_28 = ny2_1 - 1 ; j_12 <= end_28 ; j_12 = j_12 + 1)
   {
    # 5 "<preprocessor>"
    if (gp0 - 3 >= 0)
    {
     # 19 "<preprocessor>"
     hmpprt::s32 tmp_105;
     # 19 "<preprocessor>"
     hmpprt::s32 end_27;
     # 19 "<preprocessor>"
     hmpprt::s32 i_8;
     # 323 "x_solve.c"
     # 323 "x_solve.c"
     # 323 "x_solve.c"
     for (i_8 = 0, end_27 = gp0 - 3 ; i_8 <= end_27 ; i_8 = i_8 + 1)
     {
      # 324 "x_solve.c"
      hmpprt::s32 i2_1;
      # 324 "x_solve.c"
      hmpprt::s32 i1_2;
      # 324 "x_solve.c"
      hmpprt::s32 first_gang_iter_13;
      # 324 "x_solve.c"
      hmpprt::s32 last_gang_iter_13;
      # 324 "x_solve.c"
      double tmp_103;
      # 324 "x_solve.c"
      double tmp_104;
      # 324 "x_solve.c"
      i1_2 = gp0 - 3 - i_8 + 1;
      # 325 "x_solve.c"
      i2_1 = gp0 - 3 - i_8 + 2;
      # 14 "<preprocessor>"
      first_gang_iter_13 = (hmpprt::gr_gbidx()) * 8;
      # 14 "<preprocessor>"
      last_gang_iter_13 = ((first_gang_iter_13 + 7) < 2 ? (first_gang_iter_13 + 7) : 2);
      # 14 "<preprocessor>"
      hmpprt::s32 m_9;
      # 326 "x_solve.c"
      # 326 "x_solve.c"
      for (m_9 = first_gang_iter_13 + (hmpprt::gr_btidy()) ; m_9 <= last_gang_iter_13 ; m_9 = m_9 + (hmpprt::gr_btnumy()))
      {
       # 329 "x_solve.c"
       *(rhsX_1 + (j_12 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (gp0 - 3 - i_8 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_9))) = *(rhsX_1 + (j_12 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (gp0 - 3 - i_8 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_9))) - *(lhsX_1 + (j_12 + 1) + 37uLL * (gp0 - 3 - i_8 + 37uLL * (k_1 + 109))) * *(rhsX_1 + (j_12 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_2 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_9))) - *(lhsX_1 + (j_12 + 1) + 37uLL * (gp0 - 3 - i_8 + 37uLL * (k_1 + 145))) * *(rhsX_1 + (j_12 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i2_1 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * m_9)));
      }
      # 5 "<preprocessor>"
      # 5 "<preprocessor>"
      tmp_103 = *(rhsX_1 + (j_12 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (gp0 - 3 - i_8 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3))) - *(lhspX + (j_12 + 1) + 37uLL * (gp0 - 3 - i_8 + 37uLL * (k_1 + 109))) * *(rhsX_1 + (j_12 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_2 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3))) - *(lhspX + (j_12 + 1) + 37uLL * (gp0 - 3 - i_8 + 37uLL * (k_1 + 145))) * *(rhsX_1 + (j_12 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i2_1 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3)));
      # 5 "<preprocessor>"
      if ((hmpprt::gr_btidy()) == 0)
      {
       # 5 "<preprocessor>"
       *(rhsX_1 + (j_12 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (gp0 - 3 - i_8 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 3))) = tmp_103;
      }
      # 5 "<preprocessor>"
      (hmpprt::gr_barrier());
      # 5 "<preprocessor>"
      tmp_104 = *(rhsX_1 + (j_12 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (gp0 - 3 - i_8 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4))) - *(lhsmX_1 + (j_12 + 1) + 37uLL * (gp0 - 3 - i_8 + 37uLL * (k_1 + 109))) * *(rhsX_1 + (j_12 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i1_2 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4))) - *(lhsmX_1 + (j_12 + 1) + 37uLL * (gp0 - 3 - i_8 + 37uLL * (k_1 + 145))) * *(rhsX_1 + (j_12 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (i2_1 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4)));
      # 5 "<preprocessor>"
      if ((hmpprt::gr_btidy()) == 0)
      {
       # 5 "<preprocessor>"
       *(rhsX_1 + (j_12 + 1) + *(__hmpp_vla_sizes__rhsX + 3) * (gp0 - 3 - i_8 + *(__hmpp_vla_sizes__rhsX + 2) * (k_1 + 1 + *(__hmpp_vla_sizes__rhsX + 1) * 4))) = tmp_104;
      }
      # 5 "<preprocessor>"
      (hmpprt::gr_barrier());
     }
     # 5 "<preprocessor>"
     # 5 "<preprocessor>"
     tmp_105 = gp0 - 3 - (gp0 +  -2);
     # 5 "<preprocessor>"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 5 "<preprocessor>"
      *i_9 = tmp_105;
     }
     # 5 "<preprocessor>"
     (hmpprt::gr_barrier());
    }
    else
    {
     # 5 "<preprocessor>"
     hmpprt::s32 tmp_106;
     # 5 "<preprocessor>"
     tmp_106 = gp0 - 3;
     # 5 "<preprocessor>"
     if ((hmpprt::gr_btidy()) == 0)
     {
      # 5 "<preprocessor>"
      *i_9 = tmp_106;
     }
     # 5 "<preprocessor>"
     (hmpprt::gr_barrier());
    }
   }
   # 5 "<preprocessor>"
   # 5 "<preprocessor>"
   tmp_107 = ny2_1 + 1;
   # 5 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 5 "<preprocessor>"
    *j_13 = tmp_107;
   }
   # 5 "<preprocessor>"
   (hmpprt::gr_barrier());
  }
  else
  {
   # 5 "<preprocessor>"
   if ((hmpprt::gr_btidy()) == 0)
   {
    # 5 "<preprocessor>"
    *j_13 = 1;
   }
   # 5 "<preprocessor>"
   (hmpprt::gr_barrier());
  }
 }
 # 5 "<preprocessor>"
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__x_solve_75__bbz1aha1_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u64>  __hmpp_vla_sizes__rhsX_1, hmpprt::s32 nx2, hmpprt::s32 nz2, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j, hmpprt::s32* __hmpp_addr__k, hmpprt::s32 ni, hmpprt::s32 gp0_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhsX)
{
 # 7 "<preprocessor>"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhonX;
 # 8 "<preprocessor>"
 hmpprt::Context::getInstance()->allocate((void **) (&rhonX), hmpprt::MS_CUDA_GLOB, 383616uLL);
 # 8 "<preprocessor>"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  lhsmX;
 # 9 "<preprocessor>"
 hmpprt::Context::getInstance()->allocate((void **) (&lhsmX), hmpprt::MS_CUDA_GLOB, 1971360uLL);
 # 9 "<preprocessor>"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  lhspX_1;
 # 10 "<preprocessor>"
 hmpprt::Context::getInstance()->allocate((void **) (&lhspX_1), hmpprt::MS_CUDA_GLOB, 1971360uLL);
 # 10 "<preprocessor>"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  lhsX;
 # 17 "<preprocessor>"
 hmpprt::Context::getInstance()->allocate((void **) (&lhsX), hmpprt::MS_CUDA_GLOB, 1971360uLL);
 # 17 "<preprocessor>"
 hmpprt::s32 k;
 # 18 "<preprocessor>"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  j;
 # 18 "<preprocessor>"
 hmpprt::Context::getInstance()->allocate((void **) (&j), hmpprt::MS_CUDA_GLOB, 4);
 # 18 "<preprocessor>"
 *j = *__hmpp_addr__j;
 # 19 "<preprocessor>"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  i_1;
 # 19 "<preprocessor>"
 hmpprt::Context::getInstance()->allocate((void **) (&i_1), hmpprt::MS_CUDA_GLOB, 4);
 # 19 "<preprocessor>"
 *i_1 = *__hmpp_addr__i;
 # 20 "<preprocessor>"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  speed;
 # 21 "<preprocessor>"
 hmpprt::Context::getInstance()->allocate((void **) (&speed), hmpprt::MS_CUDA_GLOB, 394272uLL);
 # 21 "<preprocessor>"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rho_i;
 # 22 "<preprocessor>"
 hmpprt::Context::getInstance()->allocate((void **) (&rho_i), hmpprt::MS_CUDA_GLOB, 394272uLL);
 # 22 "<preprocessor>"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  us;
 # 23 "<preprocessor>"
 hmpprt::Context::getInstance()->allocate((void **) (&us), hmpprt::MS_CUDA_GLOB, 394272uLL);
 # 23 "<preprocessor>"
 double con43;
 # 24 "<preprocessor>"
 double comz6;
 # 25 "<preprocessor>"
 double comz5;
 # 26 "<preprocessor>"
 double comz4;
 # 27 "<preprocessor>"
 double comz1;
 # 28 "<preprocessor>"
 double c2dttx1;
 # 29 "<preprocessor>"
 double dttx2;
 # 30 "<preprocessor>"
 double dttx1;
 # 31 "<preprocessor>"
 double c3c4;
 # 32 "<preprocessor>"
 double c1c5;
 # 33 "<preprocessor>"
 double dxmax;
 # 34 "<preprocessor>"
 double dx5;
 # 35 "<preprocessor>"
 double dx2;
 # 36 "<preprocessor>"
 double dx1;
 # 37 "<preprocessor>"
 hmpprt::s32 ny2;
 # 312 "<preprocessor>"
 if (1 <= nz2)
 {
  # 312 "<preprocessor>"
  if (1)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX(192);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(8LL);
   __hmppcg_call.addLocalParameter(&c1c5, 8, "c1c5_1");
   __hmppcg_call.addLocalParameter(&c2dttx1, 8, "c2dttx1_1");
   __hmppcg_call.addLocalParameter(&c3c4, 8, "c3c4_1");
   __hmppcg_call.addLocalParameter(&comz1, 8, "comz1_1");
   __hmppcg_call.addLocalParameter(&comz4, 8, "comz4_1");
   __hmppcg_call.addLocalParameter(&comz5, 8, "comz5_1");
   __hmppcg_call.addLocalParameter(&comz6, 8, "comz6_1");
   __hmppcg_call.addLocalParameter(&con43, 8, "con43_1");
   __hmppcg_call.addLocalParameter(&dttx1, 8, "dttx1_1");
   __hmppcg_call.addLocalParameter(&dttx2, 8, "dttx2_1");
   __hmppcg_call.addLocalParameter(&dx1, 8, "dx1_1");
   __hmppcg_call.addLocalParameter(&dx2, 8, "dx2_1");
   __hmppcg_call.addLocalParameter(&dx5, 8, "dx5_1");
   __hmppcg_call.addLocalParameter(&dxmax, 8, "dxmax_1");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (gp0_1), "gp0");
   __hmppcg_call.addLocalParameter(&i_1, 8, "i_9");
   __hmppcg_call.addLocalParameter(&j, 8, "j_13");
   __hmppcg_call.addLocalParameter(&lhsX, 8, "lhsX_1");
   __hmppcg_call.addLocalParameter(&lhsmX, 8, "lhsmX_1");
   __hmppcg_call.addLocalParameter(&lhspX_1, 8, "lhspX");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (ni), "ni_1");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (nx2), "nx2_1");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (ny2), "ny2_1");
   __hmppcg_call.addLocalParameter((hmpprt::s32) (nz2), "nz2_1");
   __hmppcg_call.addLocalParameter(&rho_i, 8, "rho_i_1");
   __hmppcg_call.addLocalParameter(&rhonX, 8, "rhonX_1");
   __hmppcg_call.addLocalParameter(&__hmpp_vla_sizes__rhsX_1, 8, "__hmpp_vla_sizes__rhsX");
   __hmppcg_call.addLocalParameter(&rhsX, 8, "rhsX_1");
   __hmppcg_call.addLocalParameter(&speed, 8, "speed_1");
   __hmppcg_call.addLocalParameter(&us, 8, "us_1");
   __hmppcg_call.launch(__hmpp_acc_region__x_solve_75__bbz1aha1_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
  # 312 "<preprocessor>"
  k = nz2 + 1;
 }
 else
 {
  # 312 "<preprocessor>"
  k = 1;
 }
 # 312 "<preprocessor>"
 *__hmpp_addr__i = *i_1;
 # 313 "<preprocessor>"
 *__hmpp_addr__j = *j;
 # 314 "<preprocessor>"
 *__hmpp_addr__k = k;
 # 5 "<preprocessor>"
 hmpprt::Context::getInstance()->free((void **) (&lhsX));
 # 5 "<preprocessor>"
 hmpprt::Context::getInstance()->free((void **) (&j));
 # 5 "<preprocessor>"
 hmpprt::Context::getInstance()->free((void **) (&i_1));
 # 5 "<preprocessor>"
 hmpprt::Context::getInstance()->free((void **) (&speed));
 # 5 "<preprocessor>"
 hmpprt::Context::getInstance()->free((void **) (&rho_i));
 # 5 "<preprocessor>"
 hmpprt::Context::getInstance()->free((void **) (&us));
 # 5 "<preprocessor>"
 hmpprt::Context::getInstance()->free((void **) (&rhonX));
 # 5 "<preprocessor>"
 hmpprt::Context::getInstance()->free((void **) (&lhsmX));
 # 5 "<preprocessor>"
 hmpprt::Context::getInstance()->free((void **) (&lhspX_1));
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__x_solve_75__bbz1aha1(hmpprt::u64* __hmpp_vla_sizes__rhsX_2, hmpprt::s32 nx2_2, hmpprt::s32 nz2_2, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32* __hmpp_addr__k_1, hmpprt::s32 ni_2, hmpprt::s32 gp0_2, double* rhsX_2)
{
 # 1 "<preprocessor>"
 (__hmpp_acc_region__x_solve_75__bbz1aha1_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u64> (__hmpp_vla_sizes__rhsX_2), nx2_2, nz2_2, __hmpp_addr__i_1, __hmpp_addr__j_1, __hmpp_addr__k_1, ni_2, gp0_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (rhsX_2)));
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      __hmpp_acc_region__x_solve_75__bbz1aha1_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "__hmpp_acc_region__x_solve_75__bbz1aha1_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("__hmpp_acc_region__x_solve_75__bbz1aha1", "prototype __hmpp_acc_region__x_solve_75__bbz1aha1(__hmpp_vla_sizes__rhsX: ^cudaglob u64, nx2: s32, nz2: s32, __hmpp_addr__i: ^host s32, __hmpp_addr__j: ^host s32, __hmpp_addr__k: ^host s32, ni: s32, gp0: s32, rhsX: ^cudaglob double)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete __hmpp_acc_region__x_solve_75__bbz1aha1_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
