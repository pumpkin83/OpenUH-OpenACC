#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__compute_rhs_262__uvxq12z9(hmpprt::s32 nx2, hmpprt::s32 nz2_2, double dssp_1, double* u_1, double* rhs_1, hmpprt::s32* __hmpp_addr__i, hmpprt::s32 j, hmpprt::s32* __hmpp_addr__k, hmpprt::s32* __hmpp_addr__m)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__compute_rhs_262__uvxq12z9_internal_1(hmpprt::s32 nx2_2, hmpprt::s32 nz2_1, double dssp_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  u_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs_2, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32 j_2, hmpprt::s32* __hmpp_addr__k_1, hmpprt::s32* __hmpp_addr__m_1)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__compute_rhs_262__uvxq12z9_parallel_region_1 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__compute_rhs_262__uvxq12z9_parallel_region_1(double dssp, hmpprt::s32 j_1, hmpprt::s32* m, hmpprt::s32 nx2_1, hmpprt::s32 nz2, double* rhs, double* u);
#endif // __HIPCC__




# 5 "<preprocessor>"

#ifdef __HIPCC__

extern "C" __global__ void __hmpp_acc_region__compute_rhs_262__uvxq12z9_parallel_region_1(double dssp, hmpprt::s32 j_1, hmpprt::s32* m, hmpprt::s32 nx2_1, hmpprt::s32 nz2, double* rhs, double* u)
{
 # 8 "<preprocessor>"
 hmpprt::s32 iter_per_gang_1;
 # 8 "<preprocessor>"
 hmpprt::s32 first_gang_iter_1;
 # 8 "<preprocessor>"
 hmpprt::s32 last_gang_iter_1;
 # 8 "<preprocessor>"
 iter_per_gang_1 = 1 + (nz2 - 1) / 192;
 # 8 "<preprocessor>"
 first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
 # 8 "<preprocessor>"
 last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < (nz2 - 1) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : (nz2 - 1));
 # 8 "<preprocessor>"
 hmpprt::s32 k_1;
 # 263 "rhs.c"
 # 263 "rhs.c"
 for (k_1 = first_gang_iter_1 ; k_1 <= last_gang_iter_1 ; k_1 = k_1 + 1)
 {
  # 9 "<preprocessor>"
  hmpprt::s32 end_2;
  # 9 "<preprocessor>"
  hmpprt::s32 i_2;
  # 265 "rhs.c"
  # 265 "rhs.c"
  # 265 "rhs.c"
  for (i_2 = (hmpprt::gr_btidx()), end_2 = nx2_1 - 1 ; i_2 <= end_2 ; i_2 = i_2 + (hmpprt::gr_btnumx()))
  {
   # 7 "<preprocessor>"
   hmpprt::s32 m_1;
   # 266 "rhs.c"
   # 266 "rhs.c"
   for (m_1 = 0 ; m_1 <= 4 ; m_1 = m_1 + 1)
   {
    # 268 "rhs.c"
    *(rhs + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 1 + 36uLL * m_1))) = *(rhs + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 1 + 36uLL * m_1))) - dssp * ((double) 5.0 * *(u + (i_2 + 1) + 37uLL * (j_1 + 37uLL * (k_1 + 1 + 36uLL * m_1))) - (double) 4.0 * *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1 + 36uLL * m_1))) + *(u + (i_2 + 1) + 37uLL * (j_1 + 2 + 37uLL * (k_1 + 1 + 36uLL * m_1))));
   }
   # 5 "<preprocessor>"
   # 5 "<preprocessor>"
   *m = 5;
  }
  # 5 "<preprocessor>"
 }
 # 5 "<preprocessor>"
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__compute_rhs_262__uvxq12z9_internal_1(hmpprt::s32 nx2_2, hmpprt::s32 nz2_1, double dssp_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  u_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs_2, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32 j_2, hmpprt::s32* __hmpp_addr__k_1, hmpprt::s32* __hmpp_addr__m_1)
{
 # 7 "<preprocessor>"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  m_2;
 # 7 "<preprocessor>"
 hmpprt::Context::getInstance()->allocate((void **) (&m_2), hmpprt::MS_CUDA_GLOB, 4);
 # 7 "<preprocessor>"
 *m_2 = *__hmpp_addr__m_1;
 # 8 "<preprocessor>"
 hmpprt::s32 k;
 # 9 "<preprocessor>"
 hmpprt::s32 i_1;
 # 9 "<preprocessor>"
 i_1 = *__hmpp_addr__i_1;
 # 27 "<preprocessor>"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(1);
  __hmppcg_call.addLocalParameter(&dssp_2, 8, "dssp");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (j_2), "j_1");
  __hmppcg_call.addLocalParameter(&m_2, 8, "m");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nx2_2), "nx2_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nz2_1), "nz2");
  __hmppcg_call.addLocalParameter(&rhs_2, 8, "rhs");
  __hmppcg_call.addLocalParameter(&u_2, 8, "u");
  __hmppcg_call.launch(__hmpp_acc_region__compute_rhs_262__uvxq12z9_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 27 "<preprocessor>"
 *__hmpp_addr__i_1 = i_1;
 # 28 "<preprocessor>"
 *__hmpp_addr__k_1 = k;
 # 29 "<preprocessor>"
 *__hmpp_addr__m_1 = *m_2;
 # 5 "<preprocessor>"
 hmpprt::Context::getInstance()->free((void **) (&m_2));
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__compute_rhs_262__uvxq12z9(hmpprt::s32 nx2, hmpprt::s32 nz2_2, double dssp_1, double* u_1, double* rhs_1, hmpprt::s32* __hmpp_addr__i, hmpprt::s32 j, hmpprt::s32* __hmpp_addr__k, hmpprt::s32* __hmpp_addr__m)
{
 # 1 "<preprocessor>"
 (__hmpp_acc_region__compute_rhs_262__uvxq12z9_internal_1(nx2, nz2_2, dssp_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (u_1), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (rhs_1), __hmpp_addr__i, j, __hmpp_addr__k, __hmpp_addr__m));
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      __hmpp_acc_region__compute_rhs_262__uvxq12z9_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "__hmpp_acc_region__compute_rhs_262__uvxq12z9_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("__hmpp_acc_region__compute_rhs_262__uvxq12z9", "prototype __hmpp_acc_region__compute_rhs_262__uvxq12z9(nx2: s32, nz2: s32, dssp: double, u: ^cudaglob double, rhs: ^cudaglob double, __hmpp_addr__i: ^host s32, j: s32, __hmpp_addr__k: ^host s32, __hmpp_addr__m: ^host s32)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete __hmpp_acc_region__compute_rhs_262__uvxq12z9_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
