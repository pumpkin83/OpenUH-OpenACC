#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 7 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__compute_rhs_51__kqpyzlwg(hmpprt::s32* grid_points, double* u_1, double* us, double* vs_2, double* ws_1, double* qs, double* rho_i_1, double* square, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j, hmpprt::s32* __hmpp_addr__k_1)
;
#endif // __HIPCC__



# 7 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__compute_rhs_51__kqpyzlwg_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  grid_points_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  u_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  us_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  vs, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  ws_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  qs_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rho_i_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  square_2, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32* __hmpp_addr__k)
;
#endif // __HIPCC__



# 7 "<preprocessor>"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__compute_rhs_51__kqpyzlwg_parallel_region_1 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__compute_rhs_51__kqpyzlwg_parallel_region_1(hmpprt::s32* grid_points_1, double* qs_1, double* rho_i, double* square_1, double* u, double* us_1, double* vs_1, double* ws);
#endif // __HIPCC__




# 7 "<preprocessor>"

#ifdef __HIPCC__

extern "C" __global__ void __hmpp_acc_region__compute_rhs_51__kqpyzlwg_parallel_region_1(hmpprt::s32* grid_points_1, double* qs_1, double* rho_i, double* square_1, double* u, double* us_1, double* vs_1, double* ws)
{
 # 11 "<preprocessor>"
 hmpprt::s32 iter_per_gang_1;
 # 11 "<preprocessor>"
 hmpprt::s32 first_gang_iter_1;
 # 11 "<preprocessor>"
 hmpprt::s32 last_gang_iter_1;
 # 11 "<preprocessor>"
 iter_per_gang_1 = 1 + (*(grid_points_1 + 2) - 1) / 192;
 # 11 "<preprocessor>"
 first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
 # 11 "<preprocessor>"
 last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < (*(grid_points_1 + 2) - 1) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : (*(grid_points_1 + 2) - 1));
 # 11 "<preprocessor>"
 hmpprt::s32 k_1;
 # 52 "rhs.c"
 # 52 "rhs.c"
 for (k_1 = first_gang_iter_1 ; k_1 <= last_gang_iter_1 ; k_1 = k_1 + 1)
 {
  # 12 "<preprocessor>"
  hmpprt::s32 end_2;
  # 12 "<preprocessor>"
  hmpprt::s32 j_1;
  # 54 "rhs.c"
  # 54 "rhs.c"
  # 54 "rhs.c"
  for (j_1 = (hmpprt::gr_btidx()), end_2 = *(grid_points_1 + 1) - 1 ; j_1 <= end_2 ; j_1 = j_1 + (hmpprt::gr_btnumx()))
  {
   # 13 "<preprocessor>"
   hmpprt::s32 end_1;
   # 13 "<preprocessor>"
   hmpprt::s32 i_2;
   # 56 "rhs.c"
   # 56 "rhs.c"
   # 56 "rhs.c"
   for (i_2 = 0, end_1 = *grid_points_1 - 1 ; i_2 <= end_1 ; i_2 = i_2 + 1)
   {
    # 57 "rhs.c"
    double rho_inv_1;
    # 57 "rhs.c"
    rho_inv_1 = (double) 1.0 / *(u + i_2 + 37uLL * (j_1 + 37uLL * k_1));
    # 58 "rhs.c"
    *(rho_i + i_2 + 37uLL * (j_1 + 37uLL * k_1)) = rho_inv_1;
    # 59 "rhs.c"
    *(us_1 + i_2 + 37uLL * (j_1 + 37uLL * k_1)) = *(u + i_2 + 37uLL * (j_1 + 37uLL * (k_1 + 36uLL))) * rho_inv_1;
    # 60 "rhs.c"
    *(vs_1 + i_2 + 37uLL * (j_1 + 37uLL * k_1)) = *(u + i_2 + 37uLL * (j_1 + 37uLL * (k_1 + 72uLL))) * rho_inv_1;
    # 61 "rhs.c"
    *(ws + i_2 + 37uLL * (j_1 + 37uLL * k_1)) = *(u + i_2 + 37uLL * (j_1 + 37uLL * (k_1 + 108uLL))) * rho_inv_1;
    # 65 "rhs.c"
    *(square_1 + i_2 + 37uLL * (j_1 + 37uLL * k_1)) = (double) 0.5 * (*(u + i_2 + 37uLL * (j_1 + 37uLL * (k_1 + 36uLL))) * *(u + i_2 + 37uLL * (j_1 + 37uLL * (k_1 + 36uLL))) + *(u + i_2 + 37uLL * (j_1 + 37uLL * (k_1 + 72uLL))) * *(u + i_2 + 37uLL * (j_1 + 37uLL * (k_1 + 72uLL))) + *(u + i_2 + 37uLL * (j_1 + 37uLL * (k_1 + 108uLL))) * *(u + i_2 + 37uLL * (j_1 + 37uLL * (k_1 + 108uLL)))) * rho_inv_1;
    # 66 "rhs.c"
    *(qs_1 + i_2 + 37uLL * (j_1 + 37uLL * k_1)) = *(square_1 + i_2 + 37uLL * (j_1 + 37uLL * k_1)) * rho_inv_1;
   }
   # 7 "<preprocessor>"
  }
  # 7 "<preprocessor>"
 }
 # 7 "<preprocessor>"
}
#endif // __HIPCC__



# 7 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__compute_rhs_51__kqpyzlwg_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  grid_points_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  u_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  us_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  vs, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  ws_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  qs_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rho_i_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  square_2, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32* __hmpp_addr__k)
{
 # 11 "<preprocessor>"
 hmpprt::s32 k;
 # 12 "<preprocessor>"
 hmpprt::s32 j;
 # 12 "<preprocessor>"
 j = *__hmpp_addr__j_1;
 # 13 "<preprocessor>"
 hmpprt::s32 i_1;
 # 13 "<preprocessor>"
 i_1 = *__hmpp_addr__i_1;
 # 50 "<preprocessor>"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(1);
  __hmppcg_call.addLocalParameter(&grid_points_2, 8, "grid_points_1");
  __hmppcg_call.addLocalParameter(&qs_2, 8, "qs_1");
  __hmppcg_call.addLocalParameter(&rho_i_2, 8, "rho_i");
  __hmppcg_call.addLocalParameter(&square_2, 8, "square_1");
  __hmppcg_call.addLocalParameter(&u_2, 8, "u");
  __hmppcg_call.addLocalParameter(&us_2, 8, "us_1");
  __hmppcg_call.addLocalParameter(&vs, 8, "vs_1");
  __hmppcg_call.addLocalParameter(&ws_2, 8, "ws");
  __hmppcg_call.launch(__hmpp_acc_region__compute_rhs_51__kqpyzlwg_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 50 "<preprocessor>"
 *__hmpp_addr__i_1 = i_1;
 # 51 "<preprocessor>"
 *__hmpp_addr__j_1 = j;
 # 52 "<preprocessor>"
 *__hmpp_addr__k = k;
}
#endif // __HIPCC__



# 7 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__compute_rhs_51__kqpyzlwg(hmpprt::s32* grid_points, double* u_1, double* us, double* vs_2, double* ws_1, double* qs, double* rho_i_1, double* square, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j, hmpprt::s32* __hmpp_addr__k_1)
{
 # 1 "<preprocessor>"
 (__hmpp_acc_region__compute_rhs_51__kqpyzlwg_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32> (grid_points), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (u_1), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (us), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (vs_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (ws_1), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (qs), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (rho_i_1), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (square), __hmpp_addr__i, __hmpp_addr__j, __hmpp_addr__k_1));
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      __hmpp_acc_region__compute_rhs_51__kqpyzlwg_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "__hmpp_acc_region__compute_rhs_51__kqpyzlwg_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("__hmpp_acc_region__compute_rhs_51__kqpyzlwg", "prototype __hmpp_acc_region__compute_rhs_51__kqpyzlwg(grid_points: ^cudaglob s32, u: ^cudaglob double, us: ^cudaglob double, vs: ^cudaglob double, ws: ^cudaglob double, qs: ^cudaglob double, rho_i: ^cudaglob double, square: ^cudaglob double, __hmpp_addr__i: ^host s32, __hmpp_addr__j: ^host s32, __hmpp_addr__k: ^host s32)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete __hmpp_acc_region__compute_rhs_51__kqpyzlwg_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
