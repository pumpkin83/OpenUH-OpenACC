#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__compute_rhs_330__wsmtc0wg(hmpprt::s32 nx2, hmpprt::s32 ny2, hmpprt::s32 nz2_1, double c1_1, double c2, double con43_2, double* u_2, double* us_2, double* vs, double* ws_2, double* qs, double* rho_i_2, double* square_2, double* rhs_2, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j, hmpprt::s32* __hmpp_addr__k_1)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__compute_rhs_330__wsmtc0wg_internal_1(hmpprt::s32 nx2_2, hmpprt::s32 ny2_2, hmpprt::s32 nz2_2, double c1_2, double c2_2, double con43, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  u_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  us, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  vs_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  ws_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  qs_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rho_i, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  square_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32* __hmpp_addr__k)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__compute_rhs_330__wsmtc0wg_parallel_region_1 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__compute_rhs_330__wsmtc0wg_parallel_region_1(double c1, double c2_1, double con43_1, double dz1tz1_1, double dz2tz1_1, double dz3tz1, double dz4tz1, double dz5tz1_1, hmpprt::s32 nx2_1, hmpprt::s32 ny2_1, hmpprt::s32 nz2, double* qs_1, double* rho_i_1, double* rhs_1, double* square, double tz2, double* u, double* us_1, double* vs_1, double* ws, double zzcon2_1, double zzcon3_1, double zzcon4_1, double zzcon5_1);
#endif // __HIPCC__




# 5 "<preprocessor>"

#ifdef __HIPCC__

extern "C" __global__ void __hmpp_acc_region__compute_rhs_330__wsmtc0wg_parallel_region_1(double c1, double c2_1, double con43_1, double dz1tz1_1, double dz2tz1_1, double dz3tz1, double dz4tz1, double dz5tz1_1, hmpprt::s32 nx2_1, hmpprt::s32 ny2_1, hmpprt::s32 nz2, double* qs_1, double* rho_i_1, double* rhs_1, double* square, double tz2, double* u, double* us_1, double* vs_1, double* ws, double zzcon2_1, double zzcon3_1, double zzcon4_1, double zzcon5_1)
{
 # 10 "<preprocessor>"
 hmpprt::s32 iter_per_gang_1;
 # 10 "<preprocessor>"
 hmpprt::s32 first_gang_iter_1;
 # 10 "<preprocessor>"
 hmpprt::s32 last_gang_iter_1;
 # 10 "<preprocessor>"
 iter_per_gang_1 = 1 + (nz2 - 1) / 192;
 # 10 "<preprocessor>"
 first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
 # 10 "<preprocessor>"
 last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < (nz2 - 1) ? (first_gang_iter_1 + iter_per_gang_1 - 1) : (nz2 - 1));
 # 10 "<preprocessor>"
 hmpprt::s32 k_1;
 # 331 "rhs.c"
 # 331 "rhs.c"
 for (k_1 = first_gang_iter_1 ; k_1 <= last_gang_iter_1 ; k_1 = k_1 + 1)
 {
  # 11 "<preprocessor>"
  hmpprt::s32 end_2;
  # 11 "<preprocessor>"
  hmpprt::s32 j_1;
  # 333 "rhs.c"
  # 333 "rhs.c"
  # 333 "rhs.c"
  for (j_1 = (hmpprt::gr_btidx()), end_2 = ny2_1 - 1 ; j_1 <= end_2 ; j_1 = j_1 + (hmpprt::gr_btnumx()))
  {
   # 12 "<preprocessor>"
   hmpprt::s32 end_1;
   # 12 "<preprocessor>"
   hmpprt::s32 i_2;
   # 335 "rhs.c"
   # 335 "rhs.c"
   # 335 "rhs.c"
   for (i_2 = 0, end_1 = nx2_1 - 1 ; i_2 <= end_1 ; i_2 = i_2 + 1)
   {
    # 336 "rhs.c"
    double wp1_1;
    # 336 "rhs.c"
    double wm1_1;
    # 336 "rhs.c"
    double wijk_1;
    # 336 "rhs.c"
    wijk_1 = *(ws + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1)));
    # 337 "rhs.c"
    wp1_1 = *(ws + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 2)));
    # 338 "rhs.c"
    wm1_1 = *(ws + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * k_1));
    # 342 "rhs.c"
    *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) = *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) + dz1tz1_1 * (*(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 2))) - (double) 2.0 * *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) + *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * k_1))) - tz2 * (*(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 110))) - *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 108))));
    # 347 "rhs.c"
    *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 37))) = *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 37))) + dz2tz1_1 * (*(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 38))) - (double) 2.0 * *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 37))) + *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 36)))) + zzcon2_1 * (*(us_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 2))) - (double) 2.0 * *(us_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) + *(us_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * k_1))) - tz2 * (*(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 38))) * wp1_1 - *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 36))) * wm1_1);
    # 352 "rhs.c"
    *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 73))) = *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 73))) + dz3tz1 * (*(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 74))) - (double) 2.0 * *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 73))) + *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 72)))) + zzcon2_1 * (*(vs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 2))) - (double) 2.0 * *(vs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) + *(vs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * k_1))) - tz2 * (*(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 74))) * wp1_1 - *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 72))) * wm1_1);
    # 359 "rhs.c"
    *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 109))) = *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 109))) + dz4tz1 * (*(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 110))) - (double) 2.0 * *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 109))) + *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 108)))) + zzcon2_1 * con43_1 * (wp1_1 - (double) 2.0 * wijk_1 + wm1_1) - tz2 * (*(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 110))) * wp1_1 - *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 108))) * wm1_1 + (*(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 146))) - *(square + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 2))) - *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 144))) + *(square + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * k_1))) * c2_1);
    # 369 "rhs.c"
    *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 145))) = *(rhs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 145))) + dz5tz1_1 * (*(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 146))) - (double) 2.0 * *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 145))) + *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 144)))) + zzcon3_1 * (*(qs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 2))) - (double) 2.0 * *(qs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) + *(qs_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * k_1))) + zzcon4_1 * (wp1_1 * wp1_1 - (double) 2.0 * wijk_1 * wijk_1 + wm1_1 * wm1_1) + zzcon5_1 * (*(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 146))) * *(rho_i_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 2))) - (double) 2.0 * *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 145))) * *(rho_i_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 1))) + *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 144))) * *(rho_i_1 + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * k_1))) - tz2 * ((c1 * *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 146))) - c2_1 * *(square + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 2)))) * wp1_1 - (c1 * *(u + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * (k_1 + 144))) - c2_1 * *(square + (i_2 + 1) + 37uLL * (j_1 + 1 + 37uLL * k_1))) * wm1_1);
   }
   # 5 "<preprocessor>"
  }
  # 5 "<preprocessor>"
 }
 # 5 "<preprocessor>"
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__compute_rhs_330__wsmtc0wg_internal_1(hmpprt::s32 nx2_2, hmpprt::s32 ny2_2, hmpprt::s32 nz2_2, double c1_2, double c2_2, double con43, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  u_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  us, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  vs_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  ws_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  qs_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rho_i, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  square_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32* __hmpp_addr__k)
{
 # 10 "<preprocessor>"
 hmpprt::s32 k;
 # 11 "<preprocessor>"
 hmpprt::s32 j;
 # 11 "<preprocessor>"
 j = *__hmpp_addr__j_1;
 # 12 "<preprocessor>"
 hmpprt::s32 i_1;
 # 12 "<preprocessor>"
 i_1 = *__hmpp_addr__i_1;
 # 13 "<preprocessor>"
 double dz5tz1;
 # 14 "<preprocessor>"
 double dz4tz1_1;
 # 15 "<preprocessor>"
 double dz3tz1_1;
 # 16 "<preprocessor>"
 double dz2tz1;
 # 17 "<preprocessor>"
 double dz1tz1;
 # 18 "<preprocessor>"
 double zzcon5;
 # 19 "<preprocessor>"
 double zzcon4;
 # 20 "<preprocessor>"
 double zzcon3;
 # 21 "<preprocessor>"
 double zzcon2;
 # 22 "<preprocessor>"
 double tz2_1;
 # 76 "<preprocessor>"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(1);
  __hmppcg_call.addLocalParameter(&c1_2, 8, "c1");
  __hmppcg_call.addLocalParameter(&c2_2, 8, "c2_1");
  __hmppcg_call.addLocalParameter(&con43, 8, "con43_1");
  __hmppcg_call.addLocalParameter(&dz1tz1, 8, "dz1tz1_1");
  __hmppcg_call.addLocalParameter(&dz2tz1, 8, "dz2tz1_1");
  __hmppcg_call.addLocalParameter(&dz3tz1_1, 8, "dz3tz1");
  __hmppcg_call.addLocalParameter(&dz4tz1_1, 8, "dz4tz1");
  __hmppcg_call.addLocalParameter(&dz5tz1, 8, "dz5tz1_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nx2_2), "nx2_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (ny2_2), "ny2_1");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nz2_2), "nz2");
  __hmppcg_call.addLocalParameter(&qs_2, 8, "qs_1");
  __hmppcg_call.addLocalParameter(&rho_i, 8, "rho_i_1");
  __hmppcg_call.addLocalParameter(&rhs, 8, "rhs_1");
  __hmppcg_call.addLocalParameter(&square_1, 8, "square");
  __hmppcg_call.addLocalParameter(&tz2_1, 8, "tz2");
  __hmppcg_call.addLocalParameter(&u_1, 8, "u");
  __hmppcg_call.addLocalParameter(&us, 8, "us_1");
  __hmppcg_call.addLocalParameter(&vs_2, 8, "vs_1");
  __hmppcg_call.addLocalParameter(&ws_1, 8, "ws");
  __hmppcg_call.addLocalParameter(&zzcon2, 8, "zzcon2_1");
  __hmppcg_call.addLocalParameter(&zzcon3, 8, "zzcon3_1");
  __hmppcg_call.addLocalParameter(&zzcon4, 8, "zzcon4_1");
  __hmppcg_call.addLocalParameter(&zzcon5, 8, "zzcon5_1");
  __hmppcg_call.launch(__hmpp_acc_region__compute_rhs_330__wsmtc0wg_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 76 "<preprocessor>"
 *__hmpp_addr__i_1 = i_1;
 # 77 "<preprocessor>"
 *__hmpp_addr__j_1 = j;
 # 78 "<preprocessor>"
 *__hmpp_addr__k = k;
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__compute_rhs_330__wsmtc0wg(hmpprt::s32 nx2, hmpprt::s32 ny2, hmpprt::s32 nz2_1, double c1_1, double c2, double con43_2, double* u_2, double* us_2, double* vs, double* ws_2, double* qs, double* rho_i_2, double* square_2, double* rhs_2, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j, hmpprt::s32* __hmpp_addr__k_1)
{
 # 1 "<preprocessor>"
 (__hmpp_acc_region__compute_rhs_330__wsmtc0wg_internal_1(nx2, ny2, nz2_1, c1_1, c2, con43_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (u_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (us_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (vs), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (ws_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (qs), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (rho_i_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (square_2), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (rhs_2), __hmpp_addr__i, __hmpp_addr__j, __hmpp_addr__k_1));
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      __hmpp_acc_region__compute_rhs_330__wsmtc0wg_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "__hmpp_acc_region__compute_rhs_330__wsmtc0wg_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("__hmpp_acc_region__compute_rhs_330__wsmtc0wg", "prototype __hmpp_acc_region__compute_rhs_330__wsmtc0wg(nx2: s32, ny2: s32, nz2: s32, c1: double, c2: double, con43: double, u: ^cudaglob double, us: ^cudaglob double, vs: ^cudaglob double, ws: ^cudaglob double, qs: ^cudaglob double, rho_i: ^cudaglob double, square: ^cudaglob double, rhs: ^cudaglob double, __hmpp_addr__i: ^host s32, __hmpp_addr__j: ^host s32, __hmpp_addr__k: ^host s32)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete __hmpp_acc_region__compute_rhs_330__wsmtc0wg_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
