#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */



#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__



#ifndef __HIPCC__

#else


#endif

#define HMPPCG_SIMD_LENGTH 32

# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__x_solve_62__duvl2zjb(hmpprt::u64* __hmpp_vla_sizes__rhsX_2, hmpprt::s32 nz2_2, double* rhs, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j, hmpprt::s32* __hmpp_addr__k, double* rhsX)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__x_solve_62__duvl2zjb_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u64>  __hmpp_vla_sizes__rhsX, hmpprt::s32 nz2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs_2, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32* __hmpp_addr__k_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhsX_2)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__x_solve_62__duvl2zjb_parallel_region_1 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__x_solve_62__duvl2zjb_parallel_region_1(hmpprt::s32* i_3, hmpprt::s32* j, hmpprt::s32 nz2_1, double* rhs_1, hmpprt::u64* __hmpp_vla_sizes__rhsX_1, double* rhsX_1);
#endif // __HIPCC__




# 5 "<preprocessor>"

#ifdef __HIPCC__

extern "C" __global__ void __hmpp_acc_region__x_solve_62__duvl2zjb_parallel_region_1(hmpprt::s32* i_3, hmpprt::s32* j, hmpprt::s32 nz2_1, double* rhs_1, hmpprt::u64* __hmpp_vla_sizes__rhsX_1, double* rhsX_1)
{
 # 7 "<preprocessor>"
 hmpprt::s32 iter_per_gang_1;
 # 7 "<preprocessor>"
 hmpprt::s32 first_gang_iter_1;
 # 7 "<preprocessor>"
 hmpprt::s32 last_gang_iter_1;
 # 7 "<preprocessor>"
 iter_per_gang_1 = ((1 + nz2_1 / 192) > 8LL ? (1 + nz2_1 / 192) : 8LL);
 # 7 "<preprocessor>"
 first_gang_iter_1 = (hmpprt::gr_gbidx()) * iter_per_gang_1;
 # 7 "<preprocessor>"
 last_gang_iter_1 = ((first_gang_iter_1 + iter_per_gang_1 - 1) < nz2_1 ? (first_gang_iter_1 + iter_per_gang_1 - 1) : nz2_1);
 # 7 "<preprocessor>"
 hmpprt::s32 k_1;
 # 63 "x_solve.c"
 # 63 "x_solve.c"
 for (k_1 = first_gang_iter_1 + (hmpprt::gr_btidy()) ; k_1 <= last_gang_iter_1 ; k_1 = k_1 + (hmpprt::gr_btnumy()))
 {
  # 8 "<preprocessor>"
  hmpprt::s32 j_1;
  # 64 "x_solve.c"
  # 64 "x_solve.c"
  for (j_1 = 0 ; j_1 <= 36 ; j_1 = j_1 + 1)
  {
   # 9 "<preprocessor>"
   hmpprt::s32 i_2;
   # 65 "x_solve.c"
   # 65 "x_solve.c"
   for (i_2 = (hmpprt::gr_btidx()) ; i_2 <= 36 ; i_2 = i_2 + (hmpprt::gr_btnumx()))
   {
    # 66 "x_solve.c"
    *(rhsX_1 + j_1 + *(__hmpp_vla_sizes__rhsX_1 + 3) * (i_2 + *(__hmpp_vla_sizes__rhsX_1 + 2) * k_1)) = *(rhs_1 + i_2 + 37uLL * (j_1 + 37uLL * k_1));
    # 67 "x_solve.c"
    *(rhsX_1 + j_1 + *(__hmpp_vla_sizes__rhsX_1 + 3) * (i_2 + *(__hmpp_vla_sizes__rhsX_1 + 2) * (k_1 + *(__hmpp_vla_sizes__rhsX_1 + 1)))) = *(rhs_1 + i_2 + 37uLL * (j_1 + 37uLL * (k_1 + 36uLL)));
    # 68 "x_solve.c"
    *(rhsX_1 + j_1 + *(__hmpp_vla_sizes__rhsX_1 + 3) * (i_2 + *(__hmpp_vla_sizes__rhsX_1 + 2) * (k_1 + *(__hmpp_vla_sizes__rhsX_1 + 1) * 2))) = *(rhs_1 + i_2 + 37uLL * (j_1 + 37uLL * (k_1 + 72uLL)));
    # 69 "x_solve.c"
    *(rhsX_1 + j_1 + *(__hmpp_vla_sizes__rhsX_1 + 3) * (i_2 + *(__hmpp_vla_sizes__rhsX_1 + 2) * (k_1 + *(__hmpp_vla_sizes__rhsX_1 + 1) * 3))) = *(rhs_1 + i_2 + 37uLL * (j_1 + 37uLL * (k_1 + 108uLL)));
    # 70 "x_solve.c"
    *(rhsX_1 + j_1 + *(__hmpp_vla_sizes__rhsX_1 + 3) * (i_2 + *(__hmpp_vla_sizes__rhsX_1 + 2) * (k_1 + *(__hmpp_vla_sizes__rhsX_1 + 1) * 4))) = *(rhs_1 + i_2 + 37uLL * (j_1 + 37uLL * (k_1 + 144uLL)));
   }
   # 5 "<preprocessor>"
   # 5 "<preprocessor>"
   *i_3 = 37;
  }
  # 5 "<preprocessor>"
  # 5 "<preprocessor>"
  *j = 37;
 }
 # 5 "<preprocessor>"
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
void __hmpp_acc_region__x_solve_62__duvl2zjb_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u64>  __hmpp_vla_sizes__rhsX, hmpprt::s32 nz2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhs_2, hmpprt::s32* __hmpp_addr__i, hmpprt::s32* __hmpp_addr__j_1, hmpprt::s32* __hmpp_addr__k_1, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double>  rhsX_2)
{
 # 7 "<preprocessor>"
 hmpprt::s32 k;
 # 8 "<preprocessor>"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  j_2;
 # 8 "<preprocessor>"
 hmpprt::Context::getInstance()->allocate((void **) (&j_2), hmpprt::MS_CUDA_GLOB, 4);
 # 8 "<preprocessor>"
 *j_2 = *__hmpp_addr__j_1;
 # 9 "<preprocessor>"
 hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  i_1;
 # 9 "<preprocessor>"
 hmpprt::Context::getInstance()->allocate((void **) (&i_1), hmpprt::MS_CUDA_GLOB, 4);
 # 9 "<preprocessor>"
 *i_1 = *__hmpp_addr__i;
 # 26 "<preprocessor>"
 if (1)
 {
  hmpprt::CUDAGridCall __hmppcg_call;
  __hmppcg_call.setSizeX(192);
  __hmppcg_call.setSizeY(1);
  __hmppcg_call.setBlockSizeX(32);
  __hmppcg_call.setBlockSizeY(8LL);
  __hmppcg_call.addLocalParameter(&i_1, 8, "i_3");
  __hmppcg_call.addLocalParameter(&j_2, 8, "j");
  __hmppcg_call.addLocalParameter((hmpprt::s32) (nz2), "nz2_1");
  __hmppcg_call.addLocalParameter(&rhs_2, 8, "rhs_1");
  __hmppcg_call.addLocalParameter(&__hmpp_vla_sizes__rhsX, 8, "__hmpp_vla_sizes__rhsX_1");
  __hmppcg_call.addLocalParameter(&rhsX_2, 8, "rhsX_1");
  __hmppcg_call.launch(__hmpp_acc_region__x_solve_62__duvl2zjb_parallel_region_1, hmpprt::Context::getInstance()->getCUDADevice());
 }
 ;
 # 26 "<preprocessor>"
 *__hmpp_addr__i = *i_1;
 # 27 "<preprocessor>"
 *__hmpp_addr__j_1 = *j_2;
 # 28 "<preprocessor>"
 *__hmpp_addr__k_1 = k;
 # 5 "<preprocessor>"
 hmpprt::Context::getInstance()->free((void **) (&j_2));
 # 5 "<preprocessor>"
 hmpprt::Context::getInstance()->free((void **) (&i_1));
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  void __hmpp_acc_region__x_solve_62__duvl2zjb(hmpprt::u64* __hmpp_vla_sizes__rhsX_2, hmpprt::s32 nz2_2, double* rhs, hmpprt::s32* __hmpp_addr__i_1, hmpprt::s32* __hmpp_addr__j, hmpprt::s32* __hmpp_addr__k, double* rhsX)
{
 # 1 "<preprocessor>"
 (__hmpp_acc_region__x_solve_62__duvl2zjb_internal_1(hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::u64> (__hmpp_vla_sizes__rhsX_2), nz2_2, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (rhs), __hmpp_addr__i_1, __hmpp_addr__j, __hmpp_addr__k, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,double> (rhsX)));
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

static hmpprt::CUDAModule * hmpprt_module = 0;
static int hmpprt_uses = 0;

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    if (hmpprt_uses++ == 0)
    {
      hmpprt_module = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
      __hmpp_acc_region__x_solve_62__duvl2zjb_parallel_region_1 = new hmpprt::CUDAGrid(hmpprt_module, "__hmpp_acc_region__x_solve_62__duvl2zjb_parallel_region_1");

    }
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("__hmpp_acc_region__x_solve_62__duvl2zjb", "prototype __hmpp_acc_region__x_solve_62__duvl2zjb(__hmpp_vla_sizes__rhsX: ^cudaglob u64, nz2: s32, rhs: ^cudaglob double, __hmpp_addr__i: ^host s32, __hmpp_addr__j: ^host s32, __hmpp_addr__k: ^host s32, rhsX: ^cudaglob double)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    if (--hmpprt_uses == 0)
    {
      delete __hmpp_acc_region__x_solve_62__duvl2zjb_parallel_region_1;

      delete hmpprt_module;
      hmpprt_module = 0;
    }
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
