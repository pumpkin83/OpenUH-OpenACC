
#include <hip/hip_runtime.h>
#define __nv50_blockIdx_x blockIdx.x
#define __nv50_blockIdx_y blockIdx.y
#define __nv50_blockIdx_z blockIdx.z
#define __nv50_threadIdx_x threadIdx.x
#define __nv50_threadIdx_y threadIdx.y
#define __nv50_threadIdx_z threadIdx.z
#define __nv50_blockdim_x blockDim.x
#define __nv50_blockdim_y blockDim.y
#define __nv50_blockdim_z blockDim.z
#define __nv50_griddim_x gridDim.x
#define __nv50_griddim_y gridDim.y
#define __nv50_griddim_z gridDim.z

extern "C" __global__ void __accrg_laplacian_1_1(
  int ns,
  int ny,
  int nx,
  double * w0,
  const double alpha,
  const double beta,
  double * w1)
{
  
  int k;
  int j;
  int i;
  unsigned int __acc_tmp_0;
  
  k = __nv50_threadIdx_x;
  __acc_tmp_0 = __nv50_blockdim_y * __nv50_griddim_y;
  while(k < (ns + -1))
  {
    j = (int)(__nv50_blockIdx_y) * (int)(__nv50_blockdim_y);
    j = j + (int)(__nv50_threadIdx_y);
    while(j < (ny + -1))
    {
      i = __nv50_blockIdx_x;
      while(i < (nx + -1))
      {
        if((i >= 1) && ((k >= 1) && (j >= 1)))
        {
          * (w1 + (long long)(i + ((nx * j) + (k * (ny * nx))))) = (*(w0 + (long long)(i + ((nx * j) + (k * (ny * nx))))) * alpha) + (beta * (*(w0 + (long long)(i + ((nx * j) + ((k + -1) * (ny * nx))))) + (*(w0 + (long long)(i + ((nx * j) + ((k + 1) * (ny * nx))))) + (*(w0 + (long long)(i + ((nx * (j + -1)) + (k * (ny * nx))))) + (*(w0 + (long long)(i + ((nx * (j + 1)) + (k * (ny * nx))))) + (*(w0 + (long long)((i + ((nx * j) + (k * (ny * nx)))) + -1)) + *(w0 + (long long)((i + ((nx * j) + (k * (ny * nx)))) + 1))))))));
        }
        i = i + (int)(__nv50_griddim_x);
      }
      j = j + (int)(__acc_tmp_0);
    }
    k = k + (int)(__nv50_blockdim_x);
  }
} /* __accrg_laplacian_1_1 */

