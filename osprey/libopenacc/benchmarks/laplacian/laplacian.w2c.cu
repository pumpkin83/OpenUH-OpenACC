
#include <hip/hip_runtime.h>
#define __nv50_blockIdx_x blockIdx.x
#define __nv50_blockIdx_y blockIdx.y
#define __nv50_blockIdx_z blockIdx.z
#define __nv50_threadIdx_x threadIdx.x
#define __nv50_threadIdx_y threadIdx.y
#define __nv50_threadIdx_z threadIdx.z
#define __nv50_blockdim_x blockDim.x
#define __nv50_blockdim_y blockDim.y
#define __nv50_blockdim_z blockDim.z
#define __nv50_griddim_x gridDim.x
#define __nv50_griddim_y gridDim.y
#define __nv50_griddim_z gridDim.z

extern "C" __global__ void __accrg_laplacian_1_1(
  int ns,
  int ny,
  int nx,
  double * w0,
  const double alpha,
  const double beta,
  double * w1)
{
  
  unsigned int k;
  unsigned int j;
  unsigned int i;
  unsigned int __acc_tmp_0;
  
  k = __nv50_threadIdx_x;
  __acc_tmp_0 = __nv50_blockdim_y * __nv50_griddim_y;
  while((int) k < (ns + -1))
  {
    j = __nv50_blockIdx_y * __nv50_blockdim_y;
    j = __nv50_threadIdx_y + j;
    while((int) j < (ny + -1))
    {
      i = __nv50_blockIdx_x;
      while((int) i < (nx + -1))
      {
        if((i >= (unsigned int)(1)) && ((k >= (unsigned int)(1)) && (j >= (unsigned int)(1))))
        {
          * (w1 + (long long)((int) i + ((nx * (int) j) + ((int) k * (ny * nx))))) = (*(w0 + (long long)((int) i + ((nx * (int) j) + ((int) k * (ny * nx))))) * alpha) + (beta * (*(w0 + (long long)((int) i + ((nx * (int) j) + (((int) k + -1) * (ny * nx))))) + (*(w0 + (long long)((int) i + ((nx * (int) j) + (((int) k + 1) * (ny * nx))))) + (*(w0 + (long long)((int) i + ((nx * ((int) j + -1)) + ((int) k * (ny * nx))))) + (*(w0 + (long long)((int) i + ((nx * ((int) j + 1)) + ((int) k * (ny * nx))))) + (*(w0 + (long long)(((int) i + ((nx * (int) j) + ((int) k * (ny * nx)))) + -1)) + *(w0 + (long long)(((int) i + ((nx * (int) j) + ((int) k * (ny * nx)))) + 1))))))));
        }
        i = __nv50_griddim_x + i;
      }
      j = j + __acc_tmp_0;
    }
    k = __nv50_blockdim_x + k;
  }
} /* __accrg_laplacian_1_1 */

