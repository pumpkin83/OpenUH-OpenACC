#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void vectorAddGPU(float* a, float* b, float* c, int N)
{
  int idx;
  idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < N)
    c[idx] = a[idx] + b[idx];
}

#define MEMORY_ALIGNMENT  4096
#define ALIGN_UP(x,size) ( ((size_t)x+(size-1))&(~(size-1)) )

int main()
{
	int n, nelem;
	size_t bytes;
	// Pinned memory allocation on the CPU
	float *a, *b, *c;
	// Non-4K Aligned Pinned memory on the CPU
	float *a_UA, *b_UA, *c_UA; 

	// Device pointers for mapped memory
	float *d_a, *d_b, *d_c;
  
  	float errorNorm, refNorm, ref, diff;

	hipSetDevice(0);

	/* Allocate mapped CPU memory */
	//nelem = 1048576;
	nelem = 1048575;
	bytes = nelem*sizeof(float);

	/*
    a_UA = (float *) malloc( bytes + MEMORY_ALIGNMENT );
    b_UA = (float *) malloc( bytes + MEMORY_ALIGNMENT );
    c_UA = (float *) malloc( bytes + MEMORY_ALIGNMENT );
	*/
    a_UA = (float *) malloc( bytes );
    b_UA = (float *) malloc( bytes );
    c_UA = (float *) malloc( bytes );
    
	for(n = 0; n < nelem; n++)
    {
    	a_UA[n] = rand() / (float)RAND_MAX;
    	b_UA[n] = rand() / (float)RAND_MAX;
  	}
    
	// We need to ensure memory is aligned to 4K (so we will need to padd memory accordingly)
    a = (float *) ALIGN_UP( a_UA, MEMORY_ALIGNMENT );
    b = (float *) ALIGN_UP( b_UA, MEMORY_ALIGNMENT );
    c = (float *) ALIGN_UP( c_UA, MEMORY_ALIGNMENT );
    
	hipHostRegister(a, bytes, hipHostMallocMapped);
    hipHostRegister(b, bytes, hipHostMallocMapped);
    hipHostRegister(c, bytes, hipHostMallocMapped);
 /* 
    for(n = 0; n < nelem; n++)
    {
    	a[n] = rand() / (float)RAND_MAX;
    	b[n] = rand() / (float)RAND_MAX;
  	}
 */
  	hipHostGetDevicePointer((void **)&d_a, (void *)a, 0);
  	hipHostGetDevicePointer((void **)&d_b, (void *)b, 0);
  	hipHostGetDevicePointer((void **)&d_c, (void *)c, 0);
  
  	/* Call the GPU kernel using the CPU pointers residing in CPU mapped memory. */ 
  	dim3 block(256);
  	dim3 grid((unsigned int)ceil(nelem/(float)block.x));
  
  	vectorAddGPU<<<grid, block>>>(d_a, d_b, d_c, nelem);  
  	hipDeviceSynchronize();

  	errorNorm = 0.f;
  	refNorm = 0.f;
  	for(n = 0; n < nelem; n++)
  	{
    	ref = a[n] + b[n];
    	diff = c[n] - ref;
    	errorNorm += diff*diff;
    	refNorm += ref*ref;
  	}
 	
	errorNorm = (float)sqrt((double)errorNorm);
  	refNorm = (float)sqrt((double)refNorm);

	printf("Error norm: %f\n", errorNorm);
	printf("Ref norm: %f\n", refNorm);

  	/* Memory clean up */
    hipHostUnregister(a);
    hipHostUnregister(b);
    hipHostUnregister(c);
    free(a_UA);
    free(b_UA);
    free(c_UA);
}
