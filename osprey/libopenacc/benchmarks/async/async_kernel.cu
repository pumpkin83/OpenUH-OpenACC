
#include <hip/hip_runtime.h>
extern "C" __global__ void async_kernel(float* a, float* b, float* c, int n)
{
	int ix, iy;
	int k;
	ix = blockIdx.x*blockDim.x + threadIdx.x;
	iy = blockIdx.y*blockDim.y + threadIdx.y;

	if(ix >= 0 && ix < n && iy >=0 && iy < n)
	{
		float tmp = 0.0f;
		for(k=0; k<n; k++)
		{
			tmp += a[iy*n + k]*b[k*n + ix];
		}
		c[iy*n + ix] = tmp;
	}
}
