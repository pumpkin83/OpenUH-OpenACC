
#include <hip/hip_runtime.h>
#define __nv50_blockIdx_x blockIdx.x
#define __nv50_blockIdx_y blockIdx.y
#define __nv50_blockIdx_z blockIdx.z
#define __nv50_threadIdx_x threadIdx.x
#define __nv50_threadIdx_y threadIdx.y
#define __nv50_threadIdx_z threadIdx.z
#define __nv50_blockdim_x blockDim.x
#define __nv50_blockdim_y blockDim.y
#define __nv50_blockdim_z blockDim.z
#define __nv50_griddim_x gridDim.x
#define __nv50_griddim_y gridDim.y
#define __nv50_griddim_z gridDim.z

extern "C" __global__ void __accrg_main_1_1(
  int n1,
  float * a1,
  float * b1,
  float * c1)
{
  
  int i;
  int j;
  float tmp;
  int k;
  unsigned int __acc_tmp_0;
  unsigned int __acc_tmp_1;
  
  i = (int)(__nv50_blockIdx_y) * (int)(__nv50_blockdim_y);
  i = i + (int)(__nv50_threadIdx_y);
  __acc_tmp_0 = __nv50_blockdim_y * __nv50_griddim_y;
  __acc_tmp_1 = __nv50_blockdim_x * __nv50_griddim_x;
  while(n1 > i)
  {
    j = (int)(__nv50_blockIdx_x) * (int)(__nv50_blockdim_x);
    j = j + (int)(__nv50_threadIdx_x);
    while(n1 > j)
    {
      if((i >= 0) && (j >= 0))
      {
        tmp = 0.0F;
        k = 0;
        while(n1 > k)
        {
          _3586 :;
          tmp = tmp + (*(a1 + (long long)(k + (n1 * i))) ** (b1 + (long long)(j + (n1 * k))));
          k = k + 1;
          _3330 :;
        }
        _3842 :;
        * (c1 + (long long)(j + (n1 * i))) = tmp;
      }
      j = j + (int)(__acc_tmp_1);
    }
    i = i + (int)(__acc_tmp_0);
  }
} /* __accrg_main_1_1 */

