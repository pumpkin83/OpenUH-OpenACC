#include "hip/hip_runtime.h"
#define __nv50_blockIdx_x blockIdx.x
#define __nv50_blockIdx_y blockIdx.y
#define __nv50_blockIdx_z blockIdx.z
#define __nv50_threadIdx_x threadIdx.x
#define __nv50_threadIdx_y threadIdx.y
#define __nv50_threadIdx_z threadIdx.z
#define __nv50_blockdim_x blockDim.x
#define __nv50_blockdim_y blockDim.y
#define __nv50_blockdim_z blockDim.z
#define __nv50_griddim_x gridDim.x
#define __nv50_griddim_y gridDim.y
#define __nv50_griddim_z gridDim.z

static extern "C" __global__ void __accrg_compute_step_factor_1_1(
  int nelr,
  float * variables,
  int _w2c_VAR_DENSITY0,
  int _w2c_NVAR0,
  int _w2c_VAR_MOMENTUM0,
  int _w2c_VAR_DENSITY_ENERGY0,
  int _w2c_GAMMA0,
  float * areas,
  float * step_factors)
{
  
  int i;
  float density;
  struct float3 momentum;
  float density_energy;
  struct float3 velocity;
  float speed_sqd;
  float pressure;
  float _temp___sqrt_arg2;
  float _temp___save_sqrt3;
  float speed_of_sound;
  float _temp___sqrt_arg4;
  float _temp___save_sqrt5;
  float _temp___sqrt_arg6;
  float _temp___save_sqrt7;
  unsigned int __acc_tmp_0;
  
  i = __nv50_blockIdx_x * __nv50_blockdim_x;
  i = (unsigned int)(i) + __nv50_threadIdx_x;
  __acc_tmp_0 = __nv50_blockdim_x * __nv50_griddim_x;
  while(nelr > i)
  {
    if(i >= 0)
    {
      density = *(variables + (long long)(_w2c_VAR_DENSITY0 + (_w2c_NVAR0 * i)));
      (momentum).x = *(variables + (long long)(_w2c_VAR_MOMENTUM0 + (_w2c_NVAR0 * i)));
      (momentum).y = *(variables + (long long)((_w2c_VAR_MOMENTUM0 + (_w2c_NVAR0 * i)) + 1));
      (momentum).z = *(variables + (long long)((_w2c_VAR_MOMENTUM0 + (_w2c_NVAR0 * i)) + 2));
      density_energy = *(variables + (long long)(_w2c_VAR_DENSITY_ENERGY0 + (_w2c_NVAR0 * i)));
      (velocity).x = (momentum).x / density;
      (velocity).y = (momentum).y / density;
      (velocity).z = (momentum).z / density;
      speed_sqd = (((velocity).x * (velocity).x) + ((velocity).y * (velocity).y)) + ((velocity).z * (velocity).z);
      pressure = ((float)(_w2c_GAMMA0) + -1.0F) * (density_energy + (speed_sqd * (density * -5.0e-01F)));
      _temp___sqrt_arg2 = ((float)(_w2c_GAMMA0) * pressure) / density;
      _temp___save_sqrt3 = sqrtf(_temp___sqrt_arg2);
      speed_of_sound = _temp___save_sqrt3;
      _temp___sqrt_arg4 = *(areas + (unsigned long long)((unsigned long long) i));
      _temp___save_sqrt5 = sqrtf(_temp___sqrt_arg4);
      _temp___sqrt_arg6 = speed_sqd;
      _temp___save_sqrt7 = sqrtf(_temp___sqrt_arg6);
      * (step_factors + (unsigned long long)((unsigned long long) i)) = 5.0e-01F / (_temp___save_sqrt5 * (speed_of_sound + _temp___save_sqrt7));
    }
    i = i + (int)(__acc_tmp_0);
  }
} /* __accrg_compute_step_factor_1_1 */

