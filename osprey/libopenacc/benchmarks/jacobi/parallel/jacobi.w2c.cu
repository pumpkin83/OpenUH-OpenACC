
#include <hip/hip_runtime.h>
#define __nv50_blockIdx_x blockIdx.x
#define __nv50_blockIdx_y blockIdx.y
#define __nv50_blockIdx_z blockIdx.z
#define __nv50_threadIdx_x threadIdx.x
#define __nv50_threadIdx_y threadIdx.y
#define __nv50_threadIdx_z threadIdx.z
#define __nv50_blockdim_x blockDim.x
#define __nv50_blockdim_y blockDim.y
#define __nv50_blockdim_z blockDim.z
#define __nv50_griddim_x gridDim.x
#define __nv50_griddim_y gridDim.y
#define __nv50_griddim_z gridDim.z

extern "C" __global__ void __accrg_jacobi__1_1(
  int ny,
  int nx,
  double * w0,
  double c0,
  double c1,
  double c2,
  double * w1)
{
  
  int j;
  int i;
  int i00;
  int im10;
  int ip10;
  int i0m1;
  int i0p1;
  int im1m1;
  int im1p1;
  int ip1m1;
  int ip1p1;
  
  j = __nv50_blockIdx_x;
  while(j < (ny + -1))
  {
    i = __nv50_threadIdx_x;
    while(i < (nx + -1))
    {
      if((j >= 1) && (i >= 1))
      {
        i00 = i + (nx * j);
        im10 = (i + (nx * j)) + -1;
        ip10 = (i + (nx * j)) + 1;
        i0m1 = (i + (nx * j)) + -1;
        i0p1 = (i + (nx * j)) + 1;
        im1m1 = (i + (nx * j)) + -2;
        im1p1 = i + (nx * j);
        ip1m1 = i + (nx * j);
        ip1p1 = (i + (nx * j)) + 2;
        * (w1 + (unsigned long long)((unsigned long long) i00)) = ((*(w0 + (unsigned long long)((unsigned long long) i00)) * c0) + (c1 * (*(w0 + (unsigned long long)((unsigned long long) i0p1)) + (*(w0 + (unsigned long long)((unsigned long long) ip10)) + (*(w0 + (unsigned long long)((unsigned long long) im10)) + *(w0 + (unsigned long long)((unsigned long long) i0m1))))))) + (c2 * (*(w0 + (unsigned long long)((unsigned long long) ip1p1)) + (*(w0 + (unsigned long long)((unsigned long long) ip1m1)) + (*(w0 + (unsigned long long)((unsigned long long) im1m1)) + *(w0 + (unsigned long long)((unsigned long long) im1p1))))));
      }
      i = i + (int)(__nv50_blockdim_x);
    }
    j = j + (int)(__nv50_griddim_x);
  }
} /* __accrg_jacobi__1_1 */

