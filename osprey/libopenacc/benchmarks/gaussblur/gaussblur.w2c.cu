
#include <hip/hip_runtime.h>
#define __nv50_blockIdx_x blockIdx.x
#define __nv50_blockIdx_y blockIdx.y
#define __nv50_blockIdx_z blockIdx.z
#define __nv50_threadIdx_x threadIdx.x
#define __nv50_threadIdx_y threadIdx.y
#define __nv50_threadIdx_z threadIdx.z
#define __nv50_blockdim_x blockDim.x
#define __nv50_blockdim_y blockDim.y
#define __nv50_blockdim_z blockDim.z
#define __nv50_griddim_x gridDim.x
#define __nv50_griddim_y gridDim.y
#define __nv50_griddim_z gridDim.z

extern "C" __global__ void __accrg_gaussblur_1_1(
  int ny,
  int nx,
  double f,
  double * w0,
  const double s0,
  const double s1,
  const double s2,
  const double s4,
  const double s5,
  const double s8,
  double * w1)
{
  
  unsigned int j;
  unsigned int i;
  unsigned int __acc_tmp_0;
  
  j = __nv50_blockIdx_x * __nv50_blockdim_y;
  j = __nv50_threadIdx_y + j;
  __acc_tmp_0 = __nv50_blockdim_y * __nv50_griddim_x;
  while((int) j < (ny + -2))
  {
    i = __nv50_threadIdx_x;
    while((int) i < (nx + -2))
    {
      if((j >= (unsigned int)(2)) && (i >= (unsigned int)(2)))
      {
        * (w1 + (long long)((int) i + (nx * (int) j))) = f * ((((((*(w0 + (long long)((int) i + (nx * (int) j))) * s0) + (s1 * (*(w0 + (long long)((int) i + (nx * ((int) j + 1)))) + (*(w0 + (long long)((int) i + (nx * ((int) j + -1)))) + (*(w0 + (long long)(((int) i + (nx * (int) j)) + -1)) + *(w0 + (long long)(((int) i + (nx * (int) j)) + 1))))))) + (s2 * (*(w0 + (long long)(((int) i + (nx * ((int) j + 1))) + 1)) + (*(w0 + (long long)(((int) i + (nx * ((int) j + 1))) + -1)) + (*(w0 + (long long)(((int) i + (nx * ((int) j + -1))) + -1)) + *(w0 + (long long)(((int) i + (nx * ((int) j + -1))) + 1))))))) + (s4 * (*(w0 + (long long)((int) i + (nx * ((int) j + 2)))) + (*(w0 + (long long)((int) i + (nx * ((int) j + -2)))) + (*(w0 + (long long)(((int) i + (nx * (int) j)) + -2)) + *(w0 + (long long)(((int) i + (nx * (int) j)) + 2))))))) + (s5 * (*(w0 + (long long)(((int) i + (nx * ((int) j + 1))) + 2)) + (*(w0 + (long long)(((int) i + (nx * ((int) j + 2))) + 1)) + (*(w0 + (long long)(((int) i + (nx * ((int) j + 2))) + -1)) + (*(w0 + (long long)(((int) i + (nx * ((int) j + 1))) + -2)) + (*(w0 + (long long)(((int) i + (nx * ((int) j + -1))) + 2)) + (*(w0 + (long long)(((int) i + (nx * ((int) j + -2))) + 1)) + (*(w0 + (long long)(((int) i + (nx * ((int) j + -2))) + -1)) + *(w0 + (long long)(((int) i + (nx * ((int) j + -1))) + -2))))))))))) + (s8 * (*(w0 + (long long)(((int) i + (nx * ((int) j + 2))) + 2)) + (*(w0 + (long long)(((int) i + (nx * ((int) j + 2))) + -2)) + (*(w0 + (long long)(((int) i + (nx * ((int) j + -2))) + -2)) + *(w0 + (long long)(((int) i + (nx * ((int) j + -2))) + 2)))))));
      }
      i = __nv50_blockdim_x + i;
    }
    j = j + __acc_tmp_0;
  }
} /* __accrg_gaussblur_1_1 */

