
#include <hip/hip_runtime.h>
#define __nv50_blockIdx_x blockIdx.x
#define __nv50_blockIdx_y blockIdx.y
#define __nv50_blockIdx_z blockIdx.z
#define __nv50_threadIdx_x threadIdx.x
#define __nv50_threadIdx_y threadIdx.y
#define __nv50_threadIdx_z threadIdx.z
#define __nv50_blockdim_x blockDim.x
#define __nv50_blockdim_y blockDim.y
#define __nv50_blockdim_z blockDim.z
#define __nv50_griddim_x gridDim.x
#define __nv50_griddim_y gridDim.y
#define __nv50_griddim_z gridDim.z

extern "C" __global__ void __accrg_main_1_1(
  int N,
  int * A,
  int * B)
{
  
  unsigned int i;
  unsigned int __acc_tmp_0;
  
  i = __nv50_blockIdx_x * __nv50_blockdim_x;
  i = __nv50_threadIdx_x + i;
  __acc_tmp_0 = __nv50_blockdim_x * __nv50_griddim_x;
  while(N > (int) i)
  {
    if(1)
    {
      * (A + (unsigned long long)((unsigned long long) i)) = (int) i + 1;
      * (B + (unsigned long long)((unsigned long long) i)) = *(A + (unsigned long long)((unsigned long long) i));
    }
    i = i + __acc_tmp_0;
  }
} /* __accrg_main_1_1 */

