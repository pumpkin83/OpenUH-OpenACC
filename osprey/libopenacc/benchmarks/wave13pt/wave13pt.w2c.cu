
#include <hip/hip_runtime.h>
#define __nv50_blockIdx_x blockIdx.x
#define __nv50_blockIdx_y blockIdx.y
#define __nv50_blockIdx_z blockIdx.z
#define __nv50_threadIdx_x threadIdx.x
#define __nv50_threadIdx_y threadIdx.y
#define __nv50_threadIdx_z threadIdx.z
#define __nv50_blockdim_x blockDim.x
#define __nv50_blockdim_y blockDim.y
#define __nv50_blockdim_z blockDim.z
#define __nv50_griddim_x gridDim.x
#define __nv50_griddim_y gridDim.y
#define __nv50_griddim_z gridDim.z

extern "C" __global__ void __accrg_wave13pt_1_1(
  int ns,
  int ny,
  int nx,
  const double c1,
  double * w1,
  const double c0,
  double * w0,
  const double c2,
  double * w2)
{
  
  unsigned int k;
  unsigned int j;
  unsigned int i;
  unsigned int __acc_tmp_0;
  
  k = __nv50_blockIdx_x;
  __acc_tmp_0 = __nv50_blockdim_y * __nv50_griddim_y;
  while((int) k < (ns + -2))
  {
    j = __nv50_blockIdx_y * __nv50_blockdim_y;
    j = __nv50_threadIdx_y + j;
    while((int) j < (ny + -2))
    {
      i = __nv50_threadIdx_x;
      while((int) i < (nx + -2))
      {
        if((i >= (unsigned int)(2)) && ((k >= (unsigned int)(2)) && (j >= (unsigned int)(2))))
        {
          * (w2 + (long long)((int) i + ((nx * (int) j) + ((int) k * (ny * nx))))) = ((c1 * (*(w1 + (long long)((int) i + ((nx * (int) j) + (((int) k + -1) * (ny * nx))))) + (*(w1 + (long long)((int) i + ((nx * (int) j) + (((int) k + 1) * (ny * nx))))) + (*(w1 + (long long)((int) i + ((nx * ((int) j + -1)) + ((int) k * (ny * nx))))) + (*(w1 + (long long)((int) i + ((nx * ((int) j + 1)) + ((int) k * (ny * nx))))) + (*(w1 + (long long)(((int) i + ((nx * (int) j) + ((int) k * (ny * nx)))) + -1)) + *(w1 + (long long)(((int) i + ((nx * (int) j) + ((int) k * (ny * nx)))) + 1)))))))) + ((*(w1 + (long long)((int) i + ((nx * (int) j) + ((int) k * (ny * nx))))) * c0) - *(w0 + (long long)((int) i + ((nx * (int) j) + ((int) k * (ny * nx))))))) + (c2 * (*(w1 + (long long)((int) i + ((nx * (int) j) + (((int) k + -2) * (ny * nx))))) + (*(w1 + (long long)((int) i + ((nx * (int) j) + (((int) k + 2) * (ny * nx))))) + (*(w1 + (long long)((int) i + ((nx * ((int) j + -2)) + ((int) k * (ny * nx))))) + (*(w1 + (long long)((int) i + ((nx * ((int) j + 2)) + ((int) k * (ny * nx))))) + (*(w1 + (long long)(((int) i + ((nx * (int) j) + ((int) k * (ny * nx)))) + -2)) + *(w1 + (long long)(((int) i + ((nx * (int) j) + ((int) k * (ny * nx)))) + 2))))))));
        }
        i = __nv50_blockdim_x + i;
      }
      j = j + __acc_tmp_0;
    }
    k = __nv50_griddim_x + k;
  }
} /* __accrg_wave13pt_1_1 */

