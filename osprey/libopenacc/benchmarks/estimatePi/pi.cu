	 #include <stdlib.h>
     #include <stdio.h>
     #include <hip/hip_runtime.h>
     
     const int blockSize=16;
     const int gridSize=1024;
     
     const int totalSize=blockSize*gridSize;
     
     typedef unsigned char byte;
     
     __global__ void Calculate_Pi_On_GPU(byte isInsideCircle[])
     {
            const unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
            const unsigned idy = blockIdx.y*blockDim.y + threadIdx.y;
     
            const unsigned id=idx*totalSize+idy;
     
            double x=double(idx)/totalSize;
            double y=double(idy)/totalSize;
     
            x-=0.5;
            y-=0.5;
     
            isInsideCircle[id]=0;
            if (x*x + y*y<= 0.5*0.5)
                    isInsideCircle[id]=1;
     }
     
     int main()
     {
            int totalPoints=totalSize*totalSize;
     
            hipError_t hipError_t;
            byte* isInsideCircleOnDevice;
     
            hipError_t=hipMalloc((void**)&isInsideCircleOnDevice, totalPoints*sizeof(byte));
            if (hipError_t!=hipSuccess){
                    printf("Out of nvidia device memory\n");
                    return 0;
                    }
     
            /* Call the GPU kernel */
            dim3 block(blockSize,blockSize);
            dim3 grid(gridSize,gridSize);
            Calculate_Pi_On_GPU<<<grid, block>>>(isInsideCircleOnDevice);
     
            byte* isInsideCircle;
            isInsideCircle=(byte*) malloc(totalPoints*sizeof(byte));
            if (isInsideCircle==NULL) {
                    printf("Out of memory on host\n");
                    return 0;
                    }
     
            hipMemcpy( /* Desination:*/     isInsideCircle,
                               /* Source:    */     isInsideCircleOnDevice,
                               /* Size in bytes: */ totalPoints*sizeof(byte),
                               /* Direction   */    hipMemcpyDeviceToHost
                              );
     
            int insidePointCount=0;
            for (int i=0;i<totalPoints;i++)
                    insidePointCount+=isInsideCircle[i];
     
            double ratio=double(totalPoints)/insidePointCount;
            double pi=1/(ratio*0.5*0.5);
     
            printf("Total points : %d\n", totalPoints);
            printf("Inside points: %d\n", insidePointCount);
            printf("Ratio        : %f\n", ratio);
            printf("PI:            %f\n", pi);
     
     }
