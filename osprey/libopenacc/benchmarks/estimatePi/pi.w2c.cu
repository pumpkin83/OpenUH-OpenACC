
#include <hip/hip_runtime.h>
#define __nv50_blockIdx_x blockIdx.x
#define __nv50_blockIdx_y blockIdx.y
#define __nv50_blockIdx_z blockIdx.z
#define __nv50_threadIdx_x threadIdx.x
#define __nv50_threadIdx_y threadIdx.y
#define __nv50_threadIdx_z threadIdx.z
#define __nv50_blockdim_x blockDim.x
#define __nv50_blockdim_y blockDim.y
#define __nv50_blockdim_z blockDim.z
#define __nv50_griddim_x gridDim.x
#define __nv50_griddim_y gridDim.y
#define __nv50_griddim_z gridDim.z

extern "C" __global__ void __accrg_main_1_1(
  long long N,
  double * __reduction_pi)
{
  
  int i;
  double t;
  int __acc_tmp_0;
  int __acc_tmp_1;
  int __acc_tmp_2;
  
  __acc_tmp_0 = __nv50_blockdim_x * __nv50_griddim_x;
  __acc_tmp_0 = __acc_tmp_0 * (__nv50_threadIdx_y + __nv50_blockIdx_y);
  __acc_tmp_1 = __nv50_threadIdx_x + (__nv50_blockIdx_x * __nv50_blockdim_x);
  __acc_tmp_1 = __acc_tmp_0 + __acc_tmp_1;
  * (__reduction_pi + __acc_tmp_1) = 0.0;
  i = __nv50_blockIdx_x * __nv50_blockdim_x;
  i = __nv50_threadIdx_x + i;
  __acc_tmp_2 = __nv50_blockdim_x * __nv50_griddim_x;
  while(N > (long long) i)
  {
    if(i >= (int)(0LL))
    {
      t = ((double)((long long) i) + 5.0e-01) / (double)(N);
      * (__reduction_pi + __acc_tmp_1) = *(__reduction_pi + __acc_tmp_1) + (4.0 / ((t * t) + 1.0));
    }
    i = __acc_tmp_2 + i;
  }
} /* __accrg_main_1_1 */

