#include "hip/hip_runtime.h"
#define __nv50_blockIdx_x blockIdx.x
#define __nv50_blockIdx_y blockIdx.y
#define __nv50_blockIdx_z blockIdx.z
#define __nv50_threadIdx_x threadIdx.x
#define __nv50_threadIdx_y threadIdx.y
#define __nv50_threadIdx_z threadIdx.z
#define __nv50_blockdim_x blockDim.x
#define __nv50_blockdim_y blockDim.y
#define __nv50_blockdim_z blockDim.z
#define __nv50_griddim_x gridDim.x
#define __nv50_griddim_y gridDim.y
#define __nv50_griddim_z gridDim.z

extern "C" __global__ void __accrg_reduction_ADD_int32_1_1(
  int * g_in_sum,
  int * g_out_sum,
  unsigned int n_of_sum,
  unsigned int blocksize,
  unsigned int is_power2)
{
  
  extern __shared__ int __sdata_sum[];
  unsigned int tid;
  unsigned int i;
  unsigned int gridSize;
  int mySum;
  volatile int * __smem_sum;
  
  tid = __nv50_threadIdx_x;
  i = (__nv50_threadIdx_x * (blocksize * __nv50_blockIdx_x)) * 2U;
  gridSize = (blocksize * __nv50_griddim_x) * 2U;
  mySum = 0U;
  while(n_of_sum > i)
  {
    mySum = *((int *)((unsigned int)(g_in_sum) + (i * 4U))) + mySum;
    if((is_power2 == 1U) || (n_of_sum > (blocksize + i)))
    {
      mySum = *((int *)((unsigned int)(g_in_sum) + ((blocksize + i) * 4U))) + mySum;
    }
    i = i + gridSize;
  }
  * ((int *)((unsigned int)(__sdata_sum) + (tid * 4U))) = mySum;
  __syncthreads();
  if(blocksize >= 512U)
  {
    if(tid < 256U)
    {
      mySum = *(((int *)((unsigned int)(__sdata_sum) + (tid * 4U))) + 256LL) + mySum;
      * ((int *)((unsigned int)(__sdata_sum) + (tid * 4U))) = mySum;
    }
    __syncthreads();
  }
  if(blocksize >= 256U)
  {
    if(tid < 128U)
    {
      mySum = *(((int *)((unsigned int)(__sdata_sum) + (tid * 4U))) + 128LL) + mySum;
      * ((int *)((unsigned int)(__sdata_sum) + (tid * 4U))) = mySum;
    }
    __syncthreads();
  }
  if(blocksize >= 128U)
  {
    if(tid < 64U)
    {
      mySum = *(((int *)((unsigned int)(__sdata_sum) + (tid * 4U))) + 64LL) + mySum;
      * ((int *)((unsigned int)(__sdata_sum) + (tid * 4U))) = mySum;
    }
    __syncthreads();
  }
  if(tid < 32U)
  {
    __smem_sum = __sdata_sum;
    if(blocksize >= 64U)
    {
      if(tid < 32U)
      {
        mySum = mySum + *(((volatile int *)((unsigned int)(__smem_sum) + (tid * 4U))) + 32LL);
        * ((volatile int *)((unsigned int)(__smem_sum) + (tid * 4U))) = mySum;
      }
      __syncthreads();
    }
    if(blocksize >= 32U)
    {
      if(tid < 16U)
      {
        mySum = mySum + *(((volatile int *)((unsigned int)(__smem_sum) + (tid * 4U))) + 16LL);
        * ((volatile int *)((unsigned int)(__smem_sum) + (tid * 4U))) = mySum;
      }
      __syncthreads();
    }
    if(blocksize >= 16U)
    {
      if(tid < 8U)
      {
        mySum = mySum + *(((volatile int *)((unsigned int)(__smem_sum) + (tid * 4U))) + 8LL);
        * ((volatile int *)((unsigned int)(__smem_sum) + (tid * 4U))) = mySum;
      }
      __syncthreads();
    }
    if(blocksize >= 8U)
    {
      if(tid < 4U)
      {
        mySum = mySum + *(((volatile int *)((unsigned int)(__smem_sum) + (tid * 4U))) + 4LL);
        * ((volatile int *)((unsigned int)(__smem_sum) + (tid * 4U))) = mySum;
      }
      __syncthreads();
    }
    if(blocksize >= 4U)
    {
      if(tid < 2U)
      {
        mySum = mySum + *(((volatile int *)((unsigned int)(__smem_sum) + (tid * 4U))) + 2LL);
        * ((volatile int *)((unsigned int)(__smem_sum) + (tid * 4U))) = mySum;
      }
      __syncthreads();
    }
    if(blocksize >= 2U)
    {
      if(tid < 1U)
      {
        mySum = mySum + *(((volatile int *)((unsigned int)(__smem_sum) + (tid * 4U))) + 1LL);
        * ((volatile int *)((unsigned int)(__smem_sum) + (tid * 4U))) = mySum;
      }
      __syncthreads();
    }
  }
  if(tid == 0U)
  {
    * ((int *)((unsigned int)(g_out_sum) + (__nv50_blockIdx_x * 4U))) = *__sdata_sum;
  }
} /* __accrg_reduction_ADD_int32_1_1 */


extern "C" __global__ void __accrg_test1_1_2(
  int NJ,
  int NI,
  int * input,
  int sum,
  int * __reduction_sum)
{
  
  int j;
  int local_sum;
  int i;
  unsigned int __acc_tmp_0;
  unsigned int __acc_tmp_1;
  unsigned int __acc_tmp_2;
  
  __acc_tmp_0 = __nv50_blockdim_x * __nv50_griddim_x;
  __acc_tmp_0 = __acc_tmp_0 * (__nv50_threadIdx_y + __nv50_blockIdx_y);
  __acc_tmp_1 = __nv50_threadIdx_x + (__nv50_blockIdx_x * __nv50_blockdim_x);
  __acc_tmp_1 = __acc_tmp_0 + __acc_tmp_1;
  * (__reduction_sum + __acc_tmp_1) = 0;
  j = __nv50_blockIdx_x * __nv50_blockdim_x;
  j = (unsigned int)(j) + __nv50_threadIdx_x;
  __acc_tmp_2 = __nv50_blockdim_x * __nv50_griddim_x;
  while(NJ > j)
  {
    if(j >= 0)
    {
      local_sum = 0;
      i = 0;
      while(NI > i)
      {
        _2050 :;
        local_sum = *(input + (long long)(i + (NI * j))) + local_sum;
        i = i + 1;
        _1794 :;
      }
      _2306 :;
      sum = sum + local_sum;
    }
    j = j + (int)(__acc_tmp_2);
  }
} /* __accrg_test1_1_2 */

