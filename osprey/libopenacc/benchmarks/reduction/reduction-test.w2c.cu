
#include <hip/hip_runtime.h>
#define __nv50_blockIdx_x blockIdx.x
#define __nv50_blockIdx_y blockIdx.y
#define __nv50_blockIdx_z blockIdx.z
#define __nv50_threadIdx_x threadIdx.x
#define __nv50_threadIdx_y threadIdx.y
#define __nv50_threadIdx_z threadIdx.z
#define __nv50_blockdim_x blockDim.x
#define __nv50_blockdim_y blockDim.y
#define __nv50_blockdim_z blockDim.z
#define __nv50_griddim_x gridDim.x
#define __nv50_griddim_y gridDim.y
#define __nv50_griddim_z gridDim.z

extern "C" __global__ void __accrg_main_1_1(
  double * data,
  int N,
  double * __reduction_sum)
{
  
  int i;
  int __acc_tmp_0;
  int __acc_tmp_1;
  int __acc_tmp_2;
  
  __acc_tmp_0 = __nv50_blockdim_x * __nv50_griddim_x;
  __acc_tmp_0 = __acc_tmp_0 * (__nv50_threadIdx_y + __nv50_blockIdx_y);
  __acc_tmp_1 = __nv50_threadIdx_x + (__nv50_blockIdx_x * __nv50_blockdim_x);
  __acc_tmp_1 = __acc_tmp_0 + __acc_tmp_1;
  * (__reduction_sum + __acc_tmp_1) = 0.0;
  i = __nv50_blockIdx_x * __nv50_blockdim_x;
  i = __nv50_threadIdx_x + i;
  __acc_tmp_2 = __nv50_blockdim_x * __nv50_griddim_x;
  while(N > i)
  {
    * (__reduction_sum + __acc_tmp_1) = *(data + (unsigned int) i) + (double)(*(double *)((__reduction_sum + __acc_tmp_1)));
    i = i + __acc_tmp_2;
  }
} /* __accrg_main_1_1 */

