
#include <hip/hip_runtime.h>
#define __nv50_blockIdx_x blockIdx.x
#define __nv50_blockIdx_y blockIdx.y
#define __nv50_blockIdx_z blockIdx.z
#define __nv50_threadIdx_x threadIdx.x
#define __nv50_threadIdx_y threadIdx.y
#define __nv50_threadIdx_z threadIdx.z
#define __nv50_blockdim_x blockDim.x
#define __nv50_blockdim_y blockDim.y
#define __nv50_blockdim_z blockDim.z
#define __nv50_griddim_x gridDim.x
#define __nv50_griddim_y gridDim.y
#define __nv50_griddim_z gridDim.z

extern "C" __global__ void __accrg_main_1_1(
  int * __d_x,
  int * __d_y)
{
  
  int k;
  int j;
  int i;
  unsigned int __acc_tmp_0;
  
  k = __nv50_blockIdx_x;
  __acc_tmp_0 = __nv50_blockdim_y * __nv50_griddim_y;
  while(k <= 7)
  {
    j = (int)(__nv50_blockIdx_y) * (int)(__nv50_blockdim_y);
    j = j + (int)(__nv50_threadIdx_y);
    while(j <= 15)
    {
      i = __nv50_threadIdx_x;
      while(i <= 127)
      {
        if((i >= 0) && ((k >= 0) && (j >= 0)))
        {
          * (__d_y + i + ((k * 2048LL) + (j * 128))) = (*(__d_x + i + ((k * 2048LL) + (j * 128))) * 3) + 2;
        }
        i = i + (int)(__nv50_blockdim_x);
      }
      j = j + (int)(__acc_tmp_0);
    }
    k = k + (int)(__nv50_griddim_x);
  }
} /* __accrg_main_1_1 */

