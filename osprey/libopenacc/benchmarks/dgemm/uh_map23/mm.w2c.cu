
#include <hip/hip_runtime.h>
#define __nv50_blockIdx_x blockIdx.x
#define __nv50_blockIdx_y blockIdx.y
#define __nv50_blockIdx_z blockIdx.z
#define __nv50_threadIdx_x threadIdx.x
#define __nv50_threadIdx_y threadIdx.y
#define __nv50_threadIdx_z threadIdx.z
#define __nv50_blockdim_x blockDim.x
#define __nv50_blockdim_y blockDim.y
#define __nv50_blockdim_z blockDim.z
#define __nv50_griddim_x gridDim.x
#define __nv50_griddim_y gridDim.y
#define __nv50_griddim_z gridDim.z

extern "C" __global__ void __accrg_iter_matmul_1_1(
  int __d_n,
  double * __d_A,
  double * __d_B,
  double * __d_C)
{
  
  int i;
  double c;
  int j;
  int k;
  unsigned int __acc_tmp_0;
  
  i = __nv50_blockIdx_y;
  __acc_tmp_0 = __nv50_blockdim_x * __nv50_griddim_x;
  while(__d_n > i)
  {
    j = (int)(__nv50_blockIdx_x) * (int)(__nv50_blockdim_x);
    j = j + (int)(__nv50_threadIdx_x);
    while(__d_n > j)
    {
      if((i >= 0) && (j >= 0))
      {
        c = 0.0;
        k = 0;
        while(__d_n > k)
        {
          _514 :;
          c = c + (*(__d_A + (long long)(k + (__d_n * i))) ** (__d_B + (long long)(j + (__d_n * k))));
          k = k + 1;
          _258 :;
        }
        _770 :;
        * (__d_C + (long long)(j + (__d_n * i))) = c;
      }
      j = j + (int)(__acc_tmp_0);
    }
    i = i + (int)(__nv50_griddim_y);
  }
} /* __accrg_iter_matmul_1_1 */

