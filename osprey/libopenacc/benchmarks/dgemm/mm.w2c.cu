
#include <hip/hip_runtime.h>
#define __nv50_blockIdx_x blockIdx.x
#define __nv50_blockIdx_y blockIdx.y
#define __nv50_blockIdx_z blockIdx.z
#define __nv50_threadIdx_x threadIdx.x
#define __nv50_threadIdx_y threadIdx.y
#define __nv50_threadIdx_z threadIdx.z
#define __nv50_blockdim_x blockDim.x
#define __nv50_blockdim_y blockDim.y
#define __nv50_blockdim_z blockDim.z
#define __nv50_griddim_x gridDim.x
#define __nv50_griddim_y gridDim.y
#define __nv50_griddim_z gridDim.z

extern "C" __global__ void __accrg_iter_matmul_1_1(
  int n,
  double * A,
  double * B,
  double * C)
{
  
  int i;
  double c;
  int j;
  int k;
  unsigned int __acc_tmp_0;
  
  i = __nv50_blockIdx_y;
  __acc_tmp_0 = __nv50_blockdim_x * __nv50_griddim_x;
  while(n > i)
  {
    j = (int)(__nv50_blockIdx_x) * (int)(__nv50_blockdim_x);
    j = j + (int)(__nv50_threadIdx_x);
    while(n > j)
    {
      if((i >= 0) && (j >= 0))
      {
        c = 0.0;
        k = 0;
        while(n > k)
        {
          _514 :;
          c = c + (*(A + (long long)(k + (n * i))) ** (B + (long long)(j + (n * k))));
          k = k + 1;
          _258 :;
        }
        _770 :;
        * (C + (long long)(j + (n * i))) = c;
      }
      j = j + (int)(__acc_tmp_0);
    }
    i = i + (int)(__nv50_griddim_y);
  }
} /* __accrg_iter_matmul_1_1 */

