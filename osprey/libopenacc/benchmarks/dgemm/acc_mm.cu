
#include <hip/hip_runtime.h>
#define __nv50_blockIdx_x blockIdx.x
#define __nv50_blockIdx_y blockIdx.y
#define __nv50_blockIdx_z blockIdx.z
#define __nv50_threadIdx_x threadIdx.x
#define __nv50_threadIdx_y threadIdx.y
#define __nv50_threadIdx_z threadIdx.z
#define __nv50_blockdim_x blockDim.x
#define __nv50_blockdim_y blockDim.y
#define __nv50_blockdim_z blockDim.z
#define __nv50_griddim_x gridDim.x
#define __nv50_griddim_y gridDim.y
#define __nv50_griddim_z gridDim.z

extern "C" __global__ void __accrg_iter_matmul_1_1(
  double * A,
  double * B,
  double * C,
  int n)
{
  
  int i;
  int j;
  int __acc_tmp_0;
  int k;
  double c;
  
  i = __nv50_blockIdx_y;
  j = __nv50_blockIdx_x * __nv50_blockdim_x;
  j = __nv50_threadIdx_x + j;
  __acc_tmp_0 = __nv50_blockdim_x * __nv50_griddim_x;
  while(n > i)
  {
    j = __nv50_blockIdx_x * __nv50_blockdim_x;
    j = __nv50_threadIdx_x + j;
    while(n > j)
    {
      c = 0.0;
      k = 0;
      while(n > k)
      {
        c = c + (*(A + (unsigned int)((k + (n * i)))) ** (B + (unsigned int)((j + (n * k)))));
        k = k + 1;
      }
      * (C + (unsigned int)((j + (n * i)))) = c;
      j = j + __acc_tmp_0;
    }
    i = __nv50_griddim_y + i;
  }
} /* __accrg_iter_matmul_1_1 */

