
#include <hip/hip_runtime.h>
#define __nv50_blockIdx_x blockIdx.x
#define __nv50_blockIdx_y blockIdx.y
#define __nv50_blockIdx_z blockIdx.z
#define __nv50_threadIdx_x threadIdx.x
#define __nv50_threadIdx_y threadIdx.y
#define __nv50_threadIdx_z threadIdx.z
#define __nv50_blockdim_x blockDim.x
#define __nv50_blockdim_y blockDim.y
#define __nv50_blockdim_z blockDim.z
#define __nv50_griddim_x gridDim.x
#define __nv50_griddim_y gridDim.y
#define __nv50_griddim_z gridDim.z

extern "C" __global__ void __accrg_iter_matmul_1_1(
  int n,
  double * A,
  double * B,
  double * C)
{
  
  int i;
  int j;
  double c;
  int k;
  unsigned int __acc_tmp_0;
  unsigned int __acc_tmp_1;
  
  i = (int)(__nv50_blockIdx_y) * (int)(__nv50_blockdim_y);
  i = i + (int)(__nv50_threadIdx_y);
  __acc_tmp_0 = __nv50_blockdim_y * __nv50_griddim_y;
  __acc_tmp_1 = __nv50_blockdim_x * __nv50_griddim_x;
  while(n > i)
  {
    j = (int)(__nv50_blockIdx_x) * (int)(__nv50_blockdim_x);
    j = j + (int)(__nv50_threadIdx_x);
    while(n > j)
    {
      if((i >= 0) && (j >= 0))
      {
        c = 0.0;
        k = 0;
        while(n > k)
        {
          _514 :;
          c = c + (*(A + (long long)(k + (n * i))) ** (B + (long long)(j + (n * k))));
          k = k + 1;
          _258 :;
        }
        _770 :;
        * (C + (long long)(j + (n * i))) = c;
      }
      j = j + (int)(__acc_tmp_1);
    }
    i = i + (int)(__acc_tmp_0);
  }
} /* __accrg_iter_matmul_1_1 */

