
#include <hip/hip_runtime.h>
#define __nv50_blockIdx_x blockIdx.x
#define __nv50_blockIdx_y blockIdx.y
#define __nv50_blockIdx_z blockIdx.z
#define __nv50_threadIdx_x threadIdx.x
#define __nv50_threadIdx_y threadIdx.y
#define __nv50_threadIdx_z threadIdx.z
#define __nv50_blockdim_x blockDim.x
#define __nv50_blockdim_y blockDim.y
#define __nv50_blockdim_z blockDim.z
#define __nv50_griddim_x gridDim.x
#define __nv50_griddim_y gridDim.y
#define __nv50_griddim_z gridDim.z

extern "C" __global__ void __accrg_step_kernel_cpu_1_1(
  double * temp_in,
  double * temp_out,
  int ni,
  int nj,
  double tfac)
{
  
  int j;
  int i;
  int i00;
  int im10;
  int ip10;
  int i0m1;
  int i0p1;
  double d2tdx2;
  double d2tdy2;
  int __acc_tmp_0;
  int __acc_tmp_1;
  
  j = __nv50_blockIdx_y*__nv50_blockdim_y;
  j = j + __nv50_threadIdx_y;
  __acc_tmp_0 = __nv50_blockdim_x * __nv50_griddim_x;
  __acc_tmp_1 = __nv50_blockdim_y * __nv50_griddim_y;
  while(nj > j)
  {
    i = __nv50_blockIdx_x * __nv50_blockdim_x;
    i = __nv50_threadIdx_x + i;
    while(ni > i)
    {
	  if(i>0 && i<ni-1 && j>0 && j<nj-1)
	  {
      	i00 = i + (ni * j);
      	im10 = (i + (ni * j)) + (-1);
      	ip10 = (i + (ni * j)) + 1;
      	i0m1 = i + (ni * (j + (-1)));
      	i0p1 = i + (ni * (j + 1));
      	d2tdx2 = *(temp_in + (unsigned int) ip10) + (*(temp_in + (unsigned int) im10) + (*(temp_in + (unsigned int) i00) * (-2.0)));
      	d2tdy2 = *(temp_in + (unsigned int) i0p1) + (*(temp_in + (unsigned int) i0m1) + (*(temp_in + (unsigned int) i00) * (-2.0)));
      	* (temp_out + (unsigned int) i00) = *(temp_in + (unsigned int) i00) + (tfac * (d2tdx2 + d2tdy2));
	  }
      i = i + __acc_tmp_0;
    }
    //j = __nv50_griddim_y + j;
	  j = j + __acc_tmp_1;
  }
} /* __accrg_step_kernel_cpu_1_1 */

