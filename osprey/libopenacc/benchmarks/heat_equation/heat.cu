
#include <hip/hip_runtime.h>

extern "C" __global__ void step_kernel_gpu(
  double* temp_in,
  double* temp_out,
  int ni,
  int nj,
  double tfac)
  {
	int i, j, ti, tj, i00, im10, ip10, i0m1, i0p1;
	double d2tdx2, d2tdy2;

	ti = threadIdx.x;
	tj = threadIdx.y;
	i = blockIdx.x*blockDim.x + ti;
	j = blockIdx.y*blockDim.y + tj;

	i00 = j*ni + i;
	im10 = j*ni + (i-1);
	ip10 = j*ni + (i+1);
	i0m1 = (j-1)*ni + i;
	i0p1 = (j+1)*ni + i;

	if(i>0 && i<ni-1 && j>0 && j<nj-1)
	{
		// evaluate derivatives 
		d2tdx2 = temp_in[im10] - 2*temp_in[i00] + temp_in[ip10];
		d2tdy2 = temp_in[i0m1] - 2*temp_in[i00] + temp_in[i0p1];
										    	             
		// update temperature
		temp_out[i00] = temp_in[i00] + tfac*(d2tdx2 + d2tdy2);
	}

  }
