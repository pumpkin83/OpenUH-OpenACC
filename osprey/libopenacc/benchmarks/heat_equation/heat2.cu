
#include <hip/hip_runtime.h>

extern "C" __global__ void step_kernel_gpu(
  double* temp_in,
  double* temp_out,
  int ni,
  int nj,
  double tfac)
  {
	int i, j, ti, tj, i00, im10, ip10, i0m1, i0p1;
	double d2tdx2, d2tdy2;


/*
	if(i>0 && i<ni-1 && j>0 && j<nj-1)
	{
		// evaluate derivatives 
		d2tdx2 = temp_in[im10] - 2*temp_in[i00] + temp_in[ip10];
		d2tdy2 = temp_in[i0m1] - 2*temp_in[i00] + temp_in[i0p1];
										    	             
		// update temperature
		temp_out[i00] = temp_in[i00] + tfac*(d2tdx2 + d2tdy2);
	}
*/
	tj = threadIdx.y;
	j = blockIdx.y*blockDim.y + tj;
   while(j < nj)
   {
		ti = threadIdx.x;
		i = blockIdx.x*blockDim.x + ti;
		while(i < ni){
			if(i>0 && i<ni-1 && j>0 && j<nj-1)
			{
				i00 = j*ni + i;
				im10 = j*ni + (i-1);
				ip10 = j*ni + (i+1);
				i0m1 = (j-1)*ni + i;
				i0p1 = (j+1)*ni + i;

				d2tdx2 = temp_in[im10] - 2*temp_in[i00] + temp_in[ip10];
				d2tdy2 = temp_in[i0m1] - 2*temp_in[i00] + temp_in[i0p1];
				temp_out[i00] = temp_in[i00] + tfac*(d2tdx2 + d2tdy2);
			}
			i += blockDim.x * gridDim.x;
		}
		j += blockDim.y*gridDim.y;
	}
  }