
#include <hip/hip_runtime.h>
#define __nv50_blockIdx_x blockIdx.x
#define __nv50_blockIdx_y blockIdx.y
#define __nv50_blockIdx_z blockIdx.z
#define __nv50_threadIdx_x threadIdx.x
#define __nv50_threadIdx_y threadIdx.y
#define __nv50_threadIdx_z threadIdx.z
#define __nv50_blockdim_x blockDim.x
#define __nv50_blockdim_y blockDim.y
#define __nv50_blockdim_z blockDim.z
#define __nv50_griddim_x gridDim.x
#define __nv50_griddim_y gridDim.y
#define __nv50_griddim_z gridDim.z

extern "C" __global__ void __accrg_cpu_stencil_1_1(
  const int nx,
  const int ny,
  const int nz,
  float c1,
  float * A0,
  float c0,
  float * Anext)
{
  
  unsigned int i;
  unsigned int j;
  unsigned int k;
  unsigned int __acc_tmp_0;
  unsigned int __acc_tmp_1;
  
  i = __nv50_threadIdx_z;
  __acc_tmp_0 = __nv50_blockdim_y * __nv50_griddim_y;
  __acc_tmp_1 = __nv50_blockdim_x * __nv50_griddim_x;
  while((int) i < (nx + -1))
  {
    j = __nv50_blockIdx_y * __nv50_blockdim_y;
    j = __nv50_threadIdx_y + j;
    while((int) j < (ny + -1))
    {
      k = __nv50_blockIdx_x * __nv50_blockdim_x;
      k = __nv50_threadIdx_x + k;
      while((int) k < (nz + -1))
      {
        if((k >= (unsigned int)(1)) && ((i >= (unsigned int)(1)) && (j >= (unsigned int)(1))))
        {
          * (Anext + (long long)((int) i + (nx * ((int) j + (ny * (int) k))))) = (c1 * (*(A0 + (long long)(((int) i + (nx * ((int) j + (ny * (int) k)))) + -1)) + (*(A0 + (long long)(((int) i + (nx * ((int) j + (ny * (int) k)))) + 1)) + (*(A0 + (long long)((int) i + (nx * (((int) j + (ny * (int) k)) + -1)))) + (*(A0 + (long long)((int) i + (nx * (((int) j + (ny * (int) k)) + 1)))) + (*(A0 + (long long)((int) i + (nx * ((int) j + (ny * ((int) k + -1)))))) + *(A0 + (long long)((int) i + (nx * ((int) j + (ny * ((int) k + 1)))))))))))) - (*(A0 + (long long)((int) i + (nx * ((int) j + (ny * (int) k))))) * c0);
        }
        k = k + __acc_tmp_1;
      }
      j = j + __acc_tmp_0;
    }
    i = __nv50_blockdim_z + i;
  }
} /* __accrg_cpu_stencil_1_1 */

