
#include <hip/hip_runtime.h>

#define __nv50_blockIdx_x blockIdx.x
#define __nv50_blockdim_x blockDim.x
#define __nv50_threadIdx_x threadIdx.x
#define __nv50_griddim_x gridDim.x

/*
__global__ void __accrg_main_1_1(a, b, c, n)
  double * a;
  double * b;
  double * c;
  int n;
*/
__global__ void __accrg_main_1_1(double* a, double* b, double* c, int n)
{
  
  int i;
  int __acc_tmp_0;
  
  i = __nv50_blockIdx_x * __nv50_blockdim_x;
  i = __nv50_threadIdx_x + i;
  __acc_tmp_0 = __nv50_blockdim_x * __nv50_griddim_x;
  while(n > i)
  {
    * (c + (unsigned int) i) = *(a + (unsigned int) i) + *(b + (unsigned int) i);
    i = i + __acc_tmp_0;
  }
} /* __accrg_main_1_1 */
